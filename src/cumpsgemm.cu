#include <iostream>
#include <cassert>
#include <hipblas.h>
#include <cumpsgemm/cumpsgemm.hpp>

#include "cumpsgemm_internal.hpp"

// cuMpGEMM implementation

namespace {
template <
	class T,
	unsigned SMEM_M,
	unsigned SMEM_N,
	unsigned SMEM_K,
	unsigned FRAG_M,
	unsigned FRAG_N,
	unsigned FRAG_K,
	unsigned BLOCK_SIZE,
	class TC_T,
	class EC
>
void layout_selector (
			const hipblasOperation_t op_A,
			const hipblasOperation_t op_B,
			const std::size_t m,
			const std::size_t n,
			const std::size_t k,
			const T alpha,
			const T* const a_ptr, const std::size_t lda,
			const T* const b_ptr, const std::size_t ldb,
			const T beta,
			T* const c_ptr, const std::size_t ldc,
			hipStream_t hip_stream
		) {
#define CASE(A, a, B, b) \
	if (op_A == a && op_B == b) {cumpsgemm::launch_kernel<T, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, A, B, TC_T, EC>(m, n, k, alpha, a_ptr, lda, b_ptr, ldb, beta, c_ptr, ldc, hip_stream);return;}

	CASE(cumpsgemm::col_major, HIPBLAS_OP_N, cumpsgemm::col_major, HIPBLAS_OP_N);
	CASE(cumpsgemm::row_major, HIPBLAS_OP_T, cumpsgemm::col_major, HIPBLAS_OP_N);
	CASE(cumpsgemm::conjugate, HIPBLAS_OP_C, cumpsgemm::col_major, HIPBLAS_OP_N);
	CASE(cumpsgemm::col_major, HIPBLAS_OP_N, cumpsgemm::row_major, HIPBLAS_OP_T);
	CASE(cumpsgemm::row_major, HIPBLAS_OP_T, cumpsgemm::row_major, HIPBLAS_OP_T);
	CASE(cumpsgemm::conjugate, HIPBLAS_OP_C, cumpsgemm::row_major, HIPBLAS_OP_T);
	CASE(cumpsgemm::col_major, HIPBLAS_OP_N, cumpsgemm::conjugate, HIPBLAS_OP_C);
	CASE(cumpsgemm::row_major, HIPBLAS_OP_T, cumpsgemm::conjugate, HIPBLAS_OP_C);
	CASE(cumpsgemm::conjugate, HIPBLAS_OP_C, cumpsgemm::conjugate, HIPBLAS_OP_C);
#undef CASE
}

template <
	class T,
	unsigned SMEM_M,
	unsigned SMEM_N,
	unsigned SMEM_K,
	unsigned FRAG_M,
	unsigned FRAG_N,
	unsigned FRAG_K,
	unsigned BLOCK_SIZE,
	class TC_T,
	class EC
>
void stridedBatch_layout_selector (
			const hipblasOperation_t op_A,
			const hipblasOperation_t op_B,
			const std::size_t m,
			const std::size_t n,
			const std::size_t k,
			const T alpha,
			const T* const a_ptr, const std::size_t lda, const std::size_t strideda,
			const T* const b_ptr, const std::size_t ldb, const std::size_t stridedb,
			const T beta,
			T* const c_ptr, const std::size_t ldc, const std::size_t stridedc,
			const std::size_t batch_count,
			hipStream_t hip_stream
		) {
#define CASE(A, a, B, b) \
	if (op_A == a && op_B == b) {cumpsgemm::launch_stridedBatch_kernel<T, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, A, B, TC_T, EC>(m, n, k, alpha, a_ptr, lda, strideda, b_ptr, ldb, stridedb, beta, c_ptr, ldc, stridedc, batch_count, hip_stream);return;}

	CASE(cumpsgemm::col_major, HIPBLAS_OP_N, cumpsgemm::col_major, HIPBLAS_OP_N);
	CASE(cumpsgemm::row_major, HIPBLAS_OP_T, cumpsgemm::col_major, HIPBLAS_OP_N);
	CASE(cumpsgemm::conjugate, HIPBLAS_OP_C, cumpsgemm::col_major, HIPBLAS_OP_N);
	CASE(cumpsgemm::col_major, HIPBLAS_OP_N, cumpsgemm::row_major, HIPBLAS_OP_T);
	CASE(cumpsgemm::row_major, HIPBLAS_OP_T, cumpsgemm::row_major, HIPBLAS_OP_T);
	CASE(cumpsgemm::conjugate, HIPBLAS_OP_C, cumpsgemm::row_major, HIPBLAS_OP_T);
	CASE(cumpsgemm::col_major, HIPBLAS_OP_N, cumpsgemm::conjugate, HIPBLAS_OP_C);
	CASE(cumpsgemm::row_major, HIPBLAS_OP_T, cumpsgemm::conjugate, HIPBLAS_OP_C);
	CASE(cumpsgemm::conjugate, HIPBLAS_OP_C, cumpsgemm::conjugate, HIPBLAS_OP_C);
#undef CASE
}
} // unnamed namespace

template <class T>
hipblasStatus_t cumpsgemm::gemm(
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const T* alpha,
		const T* const a_dmem_ptr, const uint64_t lda,
		const T* const b_dmem_ptr, const uint64_t ldb,
		const T* beta,
		T* const c_dmem_ptr, const uint64_t ldc,
		const cuMpSGEMM_compute_mode_t compute_mode,
		hipStream_t hip_stream
		) {
	switch (compute_mode) {
	case CUMPSGEMM_FP16TC:   layout_selector<T, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, half                         , mtk::wmma::tcec::without_ec>(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, *beta, c_dmem_ptr, ldc, hip_stream);break;
	case CUMPSGEMM_FP16TCEC: layout_selector<T, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, half                         , mtk::wmma::tcec::with_ec   >(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, *beta, c_dmem_ptr, ldc, hip_stream);break;
	case CUMPSGEMM_TF32TC:   layout_selector<T, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, nvcuda::wmma::precision::tf32, mtk::wmma::tcec::without_ec>(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, *beta, c_dmem_ptr, ldc, hip_stream);break;
	case CUMPSGEMM_TF32TCEC: layout_selector<T, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, nvcuda::wmma::precision::tf32, mtk::wmma::tcec::with_ec   >(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, *beta, c_dmem_ptr, ldc, hip_stream);break;
	default:break;
	}

	return HIPBLAS_STATUS_SUCCESS;
}


template <class T>
hipblasStatus_t cumpsgemm::gemm_stridedBatch(
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const T* alpha,
		const T* const a_dmem_ptr, const uint64_t lda, const uint64_t stridea,
		const T* const b_dmem_ptr, const uint64_t ldb, const uint64_t strideb,
		const T* beta,
		T* const c_dmem_ptr, const uint64_t ldc, const uint64_t stridec,
		const uint64_t batch_count,
		const cuMpSGEMM_compute_mode_t compute_mode,
		hipStream_t hip_stream
		) {
	switch (compute_mode) {
	case CUMPSGEMM_FP16TC:   stridedBatch_layout_selector<T, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, half                         , mtk::wmma::tcec::without_ec>(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, stridea, b_dmem_ptr, ldb, strideb, *beta, c_dmem_ptr, ldc, stridec, batch_count, hip_stream);break;
	case CUMPSGEMM_FP16TCEC: stridedBatch_layout_selector<T, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, half                         , mtk::wmma::tcec::with_ec   >(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, stridea, b_dmem_ptr, ldb, strideb, *beta, c_dmem_ptr, ldc, stridec, batch_count, hip_stream);break;
	case CUMPSGEMM_TF32TC:   stridedBatch_layout_selector<T, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, nvcuda::wmma::precision::tf32, mtk::wmma::tcec::without_ec>(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, stridea, b_dmem_ptr, ldb, strideb, *beta, c_dmem_ptr, ldc, stridec, batch_count, hip_stream);break;
	case CUMPSGEMM_TF32TCEC: stridedBatch_layout_selector<T, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, nvcuda::wmma::precision::tf32, mtk::wmma::tcec::with_ec   >(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, stridea, b_dmem_ptr, ldb, strideb, *beta, c_dmem_ptr, ldc, stridec, batch_count, hip_stream);break;
	default:break;
	}

	return HIPBLAS_STATUS_SUCCESS;
}

extern "C" {
hipblasStatus_t cuMpSGEMM_sgemm(
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const float* alpha,
		const float* const a_dmem_ptr, const uint64_t lda,
		const float* const b_dmem_ptr, const uint64_t ldb,
		const float* beta,
		float* const c_dmem_ptr, const uint64_t ldc,
		const cuMpSGEMM_compute_mode_t compute_mode,
		hipStream_t hip_stream
		) {
	assert(op_A != HIPBLAS_OP_C);
	assert(op_B != HIPBLAS_OP_C);
	return cumpsgemm::gemm<float>(
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda,
			b_dmem_ptr, ldb,
			beta,
			c_dmem_ptr, ldc,
			compute_mode,
			hip_stream
			);
}

hipblasStatus_t cuMpSGEMM_cgemm(
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const hipComplex* alpha,
		const hipComplex* const a_dmem_ptr, const uint64_t lda,
		const hipComplex* const b_dmem_ptr, const uint64_t ldb,
		const hipComplex* beta,
		hipComplex* const c_dmem_ptr, const uint64_t ldc,
		const cuMpSGEMM_compute_mode_t compute_mode,
		hipStream_t hip_stream
		) {
	return cumpsgemm::gemm<hipComplex>(
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda,
			b_dmem_ptr, ldb,
			beta,
			c_dmem_ptr, ldc,
			compute_mode,
			hip_stream
			);
}

hipblasStatus_t cuMpSGEMM_sgemm_strided_batch(
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const float* alpha,
		const float* const a_dmem_ptr, const uint64_t lda, const uint64_t stridea,
		const float* const b_dmem_ptr, const uint64_t ldb, const uint64_t strideb,
		const float* beta,
		float* const c_dmem_ptr, const uint64_t ldc, const uint64_t stridec,
		const uint64_t batch_count,
		const cuMpSGEMM_compute_mode_t compute_mode,
		hipStream_t hip_stream
		) {
	assert(op_A != HIPBLAS_OP_C);
	assert(op_B != HIPBLAS_OP_C);
	return cumpsgemm::gemm_stridedBatch<float>(
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda, stridea,
			b_dmem_ptr, ldb, strideb,
			beta,
			c_dmem_ptr, ldc, stridec,
			batch_count,
			compute_mode,
			hip_stream
			);
}

hipblasStatus_t cuMpSGEMM_cgemm_strided_batch(
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const hipComplex* alpha,
		const hipComplex* const a_dmem_ptr, const uint64_t lda, const uint64_t stridea,
		const hipComplex* const b_dmem_ptr, const uint64_t ldb, const uint64_t strideb,
		const hipComplex* beta,
		hipComplex* const c_dmem_ptr, const uint64_t ldc, const uint64_t stridec,
		const uint64_t batch_count,
		const cuMpSGEMM_compute_mode_t compute_mode,
		hipStream_t hip_stream
		) {
	return cumpsgemm::gemm_stridedBatch<hipComplex>(
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda, stridea,
			b_dmem_ptr, ldb, strideb,
			beta,
			c_dmem_ptr, ldc, stridec,
			batch_count,
			compute_mode,
			hip_stream
			);
}
}
