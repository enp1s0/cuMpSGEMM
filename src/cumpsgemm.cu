#include <iostream>
#include <cassert>
#include <type_traits>
#include <hipblas.h>
#include <cumpsgemm/cumpsgemm.hpp>

#include "handle.hpp"

namespace {
template <class T>
cumpsgemm::kernel_module_code::code_t gen_module_code(
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const cuMpSGEMM_compute_mode_t compute_mode
		) {
	cumpsgemm::kernel_module_code::code_t code = 0;
	switch (compute_mode) {
	case CUMPSGEMM_FP16TC:   code |= cumpsgemm::kernel_module_code::half | cumpsgemm::kernel_module_code::without_ec;break;
	case CUMPSGEMM_FP16TCEC: code |= cumpsgemm::kernel_module_code::half | cumpsgemm::kernel_module_code::with_ec   ;break;
	case CUMPSGEMM_TF32TC:   code |= cumpsgemm::kernel_module_code::tf32 | cumpsgemm::kernel_module_code::without_ec;break;
	case CUMPSGEMM_TF32TCEC: code |= cumpsgemm::kernel_module_code::tf32 | cumpsgemm::kernel_module_code::with_ec   ;break;
	default:break;
	}
	switch (op_A) {
	case HIPBLAS_OP_N: code |= cumpsgemm::kernel_module_code::op_a_col_major;break;
	case HIPBLAS_OP_T: code |= cumpsgemm::kernel_module_code::op_a_row_major;break;
	case HIPBLAS_OP_C: code |= cumpsgemm::kernel_module_code::op_a_conjugate;break;
	default:break;
	}
	switch (op_B) {
	case HIPBLAS_OP_N: code |= cumpsgemm::kernel_module_code::op_b_col_major;break;
	case HIPBLAS_OP_T: code |= cumpsgemm::kernel_module_code::op_b_row_major;break;
	case HIPBLAS_OP_C: code |= cumpsgemm::kernel_module_code::op_b_conjugate;break;
	default:break;
	}
	if (std::is_same<T, float>::value) {
		code |= cumpsgemm::kernel_module_code::s;
	} else if (std::is_same<T, hipComplex>::value) {
		code |= cumpsgemm::kernel_module_code::c;
	}

	assert(code <= cumpsgemm::kernel_module_code::max_code);

	return code;
}

template <class T>
void launch_kernel (
			const cumpsgemm::gemm_module gemm_module,
			const std::size_t m,
			const std::size_t n,
			const std::size_t k,
			const T alpha,
			const T* const a_ptr, const std::size_t lda,
			const T* const b_ptr, const std::size_t ldb,
			const T beta,
			T* const c_ptr, const std::size_t ldc,
			hipStream_t hip_stream
		) {
	const auto kernel_ptr = reinterpret_cast<cumpsgemm::gemm_kernel_func_t<T>>(gemm_module.kernel_func);
	const dim3 block_size(gemm_module.block_size);
	const dim3 grid_size(
			((m + gemm_module.smem_m - 1) / gemm_module.smem_m) * ((n + gemm_module.smem_n - 1) / gemm_module.smem_n)
			);

	kernel_ptr<<<grid_size, block_size, gemm_module.smem_size, hip_stream>>>(
			m, n, k,
			alpha,
			a_ptr, lda,
			b_ptr, ldb,
			beta,
			c_ptr, ldc
			);
}

template <class T>
void launch_kernel (
			const cumpsgemm::gemm_module gemm_module,
			const std::size_t m,
			const std::size_t n,
			const std::size_t k,
			const T alpha,
			const T* const a_ptr, const std::size_t lda, const uint64_t stridea,
			const T* const b_ptr, const std::size_t ldb, const uint64_t strideb,
			const T beta,
			T* const c_ptr, const std::size_t ldc, const uint64_t stridec,
			const uint64_t batch_count,
			hipStream_t hip_stream
		) {
	const auto kernel_ptr = reinterpret_cast<cumpsgemm::gemm_stridedBatch_kernel_func_t<T>>(gemm_module.kernel_func);
	const dim3 block_size(gemm_module.block_size);
	const auto num_blocks_per_gemm = (m + gemm_module.smem_m - 1) / gemm_module.smem_m * (n + gemm_module.smem_n - 1) / gemm_module.smem_n;
	const dim3 grid_size(
			num_blocks_per_gemm * batch_count
			);

	kernel_ptr<<<grid_size, block_size, gemm_module.smem_size, hip_stream>>>(
			m, n, k,
			alpha,
			a_ptr, lda, stridea,
			b_ptr, ldb, strideb,
			beta,
			c_ptr, ldc, stridec,
			num_blocks_per_gemm
			);
}
} // unnamed namespace

template <class T>
hipblasStatus_t cumpsgemm::gemm(
		cuMpSGEMM_handle_t handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const T* alpha,
		const T* const a_dmem_ptr, const uint64_t lda,
		const T* const b_dmem_ptr, const uint64_t ldb,
		const T* beta,
		T* const c_dmem_ptr, const uint64_t ldc,
		const cuMpSGEMM_compute_mode_t compute_mode,
		int* const used_kernel_modeule_id
		) {
	const auto code = gen_module_code<T>(op_A, op_B, compute_mode);

	const auto kernel_module_candidate_list = handle->gemm_module[code];

	unsigned module_id;
	auto gemm_module = kernel_module_candidate_list[0];
	for (module_id = 0; module_id < handle->num_kernel_candidates - 1; module_id++) {

	}

	if (used_kernel_modeule_id != nullptr) {
		*used_kernel_modeule_id = module_id;
	}

	launch_kernel<T>(
			gemm_module,
			m, n, k,
			*alpha,
			a_dmem_ptr, lda,
			b_dmem_ptr, ldb,
			*beta,
			c_dmem_ptr, ldc,
			handle->hip_stream
			);

	return HIPBLAS_STATUS_SUCCESS;
}


template <class T>
hipblasStatus_t cumpsgemm::gemm_stridedBatch(
		cuMpSGEMM_handle_t handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const T* alpha,
		const T* const a_dmem_ptr, const uint64_t lda, const uint64_t stridea,
		const T* const b_dmem_ptr, const uint64_t ldb, const uint64_t strideb,
		const T* beta,
		T* const c_dmem_ptr, const uint64_t ldc, const uint64_t stridec,
		const uint64_t batch_count,
		const cuMpSGEMM_compute_mode_t compute_mode,
		int* const used_kernel_modeule_id
		) {
	const auto code = gen_module_code<T>(op_A, op_B, compute_mode);

	const auto kernel_module_candidate_list = handle->gemm_stridedBatch_module[code];

	unsigned module_id;
	auto gemm_module = kernel_module_candidate_list[0];
	for (module_id = 0; module_id < handle->num_kernel_candidates - 1; module_id++) {

	}

	if (used_kernel_modeule_id != nullptr) {
		*used_kernel_modeule_id = module_id;
	}

	launch_kernel<T>(
			gemm_module,
			m, n, k,
			*alpha,
			a_dmem_ptr, lda, stridea,
			b_dmem_ptr, ldb, strideb,
			*beta,
			c_dmem_ptr, ldc, stridec,
			batch_count,
			handle->hip_stream
			);

	return HIPBLAS_STATUS_SUCCESS;
}

extern "C" {
hipblasStatus_t cuMpSGEMM_sgemm(
		cuMpSGEMM_handle_t handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const float* alpha,
		const float* const a_dmem_ptr, const uint64_t lda,
		const float* const b_dmem_ptr, const uint64_t ldb,
		const float* beta,
		float* const c_dmem_ptr, const uint64_t ldc,
		const cuMpSGEMM_compute_mode_t compute_mode
		) {
	assert(op_A != HIPBLAS_OP_C);
	assert(op_B != HIPBLAS_OP_C);
	return cumpsgemm::gemm<float>(
			handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda,
			b_dmem_ptr, ldb,
			beta,
			c_dmem_ptr, ldc,
			compute_mode
			);
}

hipblasStatus_t cuMpSGEMM_cgemm(
		cuMpSGEMM_handle_t handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const hipComplex* alpha,
		const hipComplex* const a_dmem_ptr, const uint64_t lda,
		const hipComplex* const b_dmem_ptr, const uint64_t ldb,
		const hipComplex* beta,
		hipComplex* const c_dmem_ptr, const uint64_t ldc,
		const cuMpSGEMM_compute_mode_t compute_mode
		) {
	return cumpsgemm::gemm<hipComplex>(
			handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda,
			b_dmem_ptr, ldb,
			beta,
			c_dmem_ptr, ldc,
			compute_mode
			);
}

hipblasStatus_t cuMpSGEMM_sgemm_strided_batch(
		cuMpSGEMM_handle_t handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const float* alpha,
		const float* const a_dmem_ptr, const uint64_t lda, const uint64_t stridea,
		const float* const b_dmem_ptr, const uint64_t ldb, const uint64_t strideb,
		const float* beta,
		float* const c_dmem_ptr, const uint64_t ldc, const uint64_t stridec,
		const uint64_t batch_count,
		const cuMpSGEMM_compute_mode_t compute_mode
		) {
	assert(op_A != HIPBLAS_OP_C);
	assert(op_B != HIPBLAS_OP_C);
	return cumpsgemm::gemm_stridedBatch<float>(
			handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda, stridea,
			b_dmem_ptr, ldb, strideb,
			beta,
			c_dmem_ptr, ldc, stridec,
			batch_count,
			compute_mode
			);
}

hipblasStatus_t cuMpSGEMM_cgemm_strided_batch(
		cuMpSGEMM_handle_t handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const hipComplex* alpha,
		const hipComplex* const a_dmem_ptr, const uint64_t lda, const uint64_t stridea,
		const hipComplex* const b_dmem_ptr, const uint64_t ldb, const uint64_t strideb,
		const hipComplex* beta,
		hipComplex* const c_dmem_ptr, const uint64_t ldc, const uint64_t stridec,
		const uint64_t batch_count,
		const cuMpSGEMM_compute_mode_t compute_mode
		) {
	return cumpsgemm::gemm_stridedBatch<hipComplex>(
			handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda, stridea,
			b_dmem_ptr, ldb, strideb,
			beta,
			c_dmem_ptr, ldc, stridec,
			batch_count,
			compute_mode
			);
}
} // extern "C"
