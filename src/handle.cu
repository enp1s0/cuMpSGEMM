#include <cstddef>
#include <cumpsgemm/cumpsgemm.h>
#include "handle.hpp"
#include "cumpsgemm_kernel.cuh"

#define SET_GEMM_KERNEL_MODULE(module_list, io_t, tc_t, ec, op_a, op_b, smem_m, smem_n, smem_k, frag_m, frag_n, frag_k, block_size, num_unrollings, num_stages, pipelined, gemm_type, stage) \
	module_list[cumpsgemm::kernel_module_code::tc_t | cumpsgemm::kernel_module_code::ec | cumpsgemm::kernel_module_code::op_a_##op_a | cumpsgemm::kernel_module_code::op_b_##op_b | cumpsgemm::kernel_module_code::gemm_type][stage] =\
	cumpsgemm::generate_gemm_module<io_t,smem_m,smem_n,smem_k,frag_m,frag_n,frag_k,block_size,num_unrollings,num_stages,cumpsgemm::op_a,cumpsgemm::op_b,tc_t,mtk::wmma::tcec::ec, pipelined>();

#define SET_GEMM_STRIDEDBATCH_KERNEL_MODULE(module_list, io_t, tc_t, ec, op_a, op_b, smem_m, smem_n, smem_k, frag_m, frag_n, frag_k, block_size, num_unrollings, num_stages, pipelined, gemm_type, stage) \
	module_list[cumpsgemm::kernel_module_code::tc_t | cumpsgemm::kernel_module_code::ec | cumpsgemm::kernel_module_code::op_a_##op_a | cumpsgemm::kernel_module_code::op_b_##op_b | cumpsgemm::kernel_module_code::gemm_type][stage] =\
	cumpsgemm::generate_gemm_stridedBatch_module<io_t,smem_m,smem_n,smem_k,frag_m,frag_n,frag_k,block_size,num_unrollings,num_stages,cumpsgemm::op_a,cumpsgemm::op_b,tc_t,mtk::wmma::tcec::ec, pipelined>();

#define COMPILE_SGEMM_KERNEL
#define COMPILE_CGEMM_KERNEL
#define COMPILE_SGEMM_STRIDEDBATCH_KERNEL
#define COMPILE_CGEMM_STRIDEDBATCH_KERNEL

#define ENABLE_A100_OPTIMAZED_PARAMETERS

extern "C" {
hipblasStatus_t cuMpSGEMM_create(cuMpSGEMM_handle_t* const handle) {
	if ((*handle = new cuMpSGEMM_handle) == nullptr) {
		return HIPBLAS_STATUS_INTERNAL_ERROR;
	}

	int num_sms;
	hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
	(*handle)->num_sms = num_sms;

	using tf32 = nvcuda::wmma::precision::tf32;

#ifdef ENABLE_A100_OPTIMAZED_PARAMETERS
	// set kernel modules
#ifdef COMPILE_SGEMM_KERNEL
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, col_major,  64, 128,  32,  32,  64,  32, 128,   2,   2, false, s, 0); // N=  16384, p= 46.50 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, col_major, 128,  64,  32,  32,  64,  32, 128,   2,   2, false, s, 1); // N=   4096, p= 42.38 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, col_major,  64,  64,  32,  32,  32,  32, 128,   1,   2, false, s, 2); // N=   1024, p= 21.66 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, col_major,  64, 128,  32,  32,  64,  16, 128,   1,   2, false, s, 0); // N=  16384, p= 29.85 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, col_major,  64, 128,  32,  32,  64,  16, 128,   1,   2, false, s, 1); // N=   4096, p= 29.70 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, col_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, s, 2); // N=   1024, p= 19.80 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, col_major, 128, 128,  32,  64,  64,  16, 128,   1,   2, false, s, 0); // N=  16384, p= 75.47 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, col_major, 128, 128,  32,  64,  64,  32, 128,   1,   2, false, s, 1); // N=   4096, p= 88.33 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, col_major, 128, 128,  32,  64,  64,  32, 128,   1,   2, false, s, 2); // N=   1024, p= 43.91 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, col_major,  64, 128,  32,  32,  64,  16, 128,   1,   2, false, s, 0); // N=  16384, p= 64.92 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, col_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 1); // N=   4096, p= 69.45 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, col_major, 128, 128,  32,  64,  64,  32, 128,   1,   2, false, s, 2); // N=   1024, p= 37.51 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, row_major, 128,  64,  32,  64,  32,  32, 128,   1,   2, false, s, 0); // N=  16384, p= 39.40 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, row_major, 128,  64,  32,  64,  32,  32, 128,   1,   2, false, s, 1); // N=   4096, p= 40.31 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, row_major, 128,  32,  32,  32,  32,  16, 128,   2,   2, false, s, 2); // N=   1024, p= 24.61 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, row_major,  64, 128,  32,  32,  64,  16, 128,   1,   2, false, s, 0); // N=  16384, p= 30.48 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, row_major,  64, 128,  32,  32,  64,  16, 128,   1,   2, false, s, 1); // N=   4096, p= 30.52 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, row_major,  64,  64,  32,  32,  32,  32, 128,   1,   2, false, s, 2); // N=   1024, p= 18.63 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, row_major, 128, 128,  32,  64,  64,  16, 128,   1,   2, false, s, 0); // N=  16384, p= 68.42 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, row_major, 128, 128,  32,  64,  64,  32, 128,   1,   2, false, s, 1); // N=   4096, p= 76.96 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, row_major, 128, 128,  32,  64,  64,  32, 128,   1,   2, false, s, 2); // N=   1024, p= 39.29 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, row_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 0); // N=  16384, p= 65.25 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, row_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 1); // N=   4096, p= 70.62 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, row_major, 128, 128,  32,  64,  64,  32, 128,   1,   2, false, s, 2); // N=   1024, p= 37.84 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, col_major,  64, 128,  32,  32,  64,  32, 128,   1,   2, false, s, 0); // N=  16384, p= 48.65 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, col_major,  64, 128,  32,  64,  32,  32, 128,   1,   2, false, s, 1); // N=   4096, p= 50.54 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, col_major, 128,  32,  32,  32,  32,  32, 128,   1,   2, false, s, 2); // N=   1024, p= 27.59 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, col_major,  64, 128,  32,  64,  32,  16, 128,   1,   2, false, s, 0); // N=  16384, p= 29.85 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, col_major,  64, 128,  32,  64,  32,  16, 128,   1,   2, false, s, 1); // N=   4096, p= 29.90 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, col_major,  32,  64,  32,  32,  16,  16, 128,   2,   2, false, s, 2); // N=   1024, p= 18.07 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, col_major,  64, 128,  32,  32,  64,  16, 128,   1,   2, false, s, 0); // N=  16384, p= 73.16 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, col_major, 128, 128,  32,  64,  64,  32, 128,   1,   2, false, s, 1); // N=   4096, p= 96.84 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, col_major, 128, 128,  32,  64,  64,  32, 128,   1,   2, false, s, 2); // N=   1024, p= 49.44 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, col_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 0); // N=  16384, p= 60.76 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, col_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 1); // N=   4096, p= 65.98 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, col_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 2); // N=   1024, p= 34.41 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, row_major,  64, 128,  32,  64,  32,  32, 128,   2,   2, false, s, 0); // N=  16384, p= 47.28 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, row_major,  64, 128,  32,  64,  32,  32, 128,   2,   2, false, s, 1); // N=   4096, p= 49.13 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, row_major, 128,  32,  32,  32,  32,  16, 128,   1,   2, false, s, 2); // N=   1024, p= 26.19 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, row_major,  64, 128,  32,  32,  64,  16, 128,   1,   2, false, s, 0); // N=  16384, p= 30.71 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, row_major,  64, 128,  32,  32,  64,  16, 128,   1,   2, false, s, 1); // N=   4096, p= 30.80 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, row_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, s, 2); // N=   1024, p= 19.26 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, row_major,  64, 128,  32,  64,  32,  32, 128,   1,   2, false, s, 0); // N=  16384, p= 70.62 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, row_major, 128, 128,  32,  64,  64,  32, 128,   1,   2, false, s, 1); // N=   4096, p= 83.73 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, row_major, 128, 128,  32,  64,  64,  32, 128,   1,   2, false, s, 2); // N=   1024, p= 42.66 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, row_major, 128, 128,  32,  32,  64,  32, 256,   2,   2, false, s, 0); // N=  16384, p= 65.69 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, row_major,  64, 128,  32,  32,  64,  16, 128,   2,   2, false, s, 1); // N=   4096, p= 68.57 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, row_major, 128, 128,  32,  64,  64,  16, 128,   2,   2, false, s, 2); // N=   1024, p= 37.41 [TFlop/s]
#endif
#ifdef COMPILE_CGEMM_KERNEL
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, col_major, 128,  64,  32,  16,  64,  16, 256,   2,   2, false, c, 0); // N=   8192, p= 44.72 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, col_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=   2048, p= 33.01 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, col_major,  32,  32,  32,  16,  16,  32, 128,   2,   2, false, c, 2); // N=    512, p= 17.99 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, col_major, 128,  64,  32,  16,  64,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 31.25 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, col_major,  64,  64,  32,  16,  64,  16, 128,   2,   2, false, c, 1); // N=   2048, p= 30.15 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, col_major,  32,  32,  32,  16,  16,  32, 128,   1,   2, false, c, 2); // N=    512, p= 18.80 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, col_major, 128,  64,  32,  32,  64,  16, 128,   2,   2, false, c, 0); // N=   8192, p= 96.91 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, col_major, 128,  64,  32,  32,  64,  16, 128,   2,   2, false, c, 1); // N=   2048, p= 89.74 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, col_major,  32, 128,  32,  16,  64,  16, 128,   1,   2, false, c, 2); // N=    512, p= 34.26 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, col_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 66.97 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, col_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=   2048, p= 63.16 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, col_major,  32,  32,  32,  16,  16,  32, 128,   1,   2, false, c, 2); // N=    512, p= 28.00 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, row_major, 128,  64,  32,  32,  32,  16, 256,   2,   2, false, c, 0); // N=   8192, p= 43.02 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, row_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 40.02 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, row_major,  32, 128,  32,  32,  32,  16, 128,   1,   2, false, c, 2); // N=    512, p= 19.03 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, row_major, 128,  64,  32,  16,  64,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 31.33 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, row_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 29.82 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, row_major,  32,  32,  32,  16,  16,  32, 128,   1,   2, false, c, 2); // N=    512, p= 17.35 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, row_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, c, 0); // N=   8192, p= 72.90 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, row_major,  64, 128,  32,  32,  64,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 67.69 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, row_major, 128,  32,  32,  32,  32,  16, 128,   1,   2, false, c, 2); // N=    512, p= 26.09 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, row_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 64.71 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, row_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=   2048, p= 59.59 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, row_major,  32,  32,  32,  16,  16,  32, 128,   1,   2, false, c, 2); // N=    512, p= 26.68 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   8192, p= 39.55 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 35.67 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 17.21 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   8192, p= 28.65 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, conjugate,  64,  32,  32,  16,  32,  32, 128,   1,   2, false, c, 1); // N=   2048, p= 27.00 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 15.22 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, conjugate, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, c, 0); // N=   8192, p= 58.96 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, conjugate, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 49.92 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, conjugate, 128,  32,  32,  32,  32,  16, 128,   1,   2, false, c, 2); // N=    512, p= 20.33 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, conjugate,  64,  64,  32,  32,  32,  32, 128,   1,   2, false, c, 0); // N=   8192, p= 52.34 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, conjugate,  64,  64,  32,  32,  32,  32, 128,   1,   2, false, c, 1); // N=   2048, p= 49.19 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, conjugate,  32,  32,  32,  16,  16,  16, 128,   2,   2, false, c, 2); // N=    512, p= 22.15 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, col_major,  64, 128,  32,  32,  32,  16, 256,   2,   2, false, c, 0); // N=   8192, p= 53.08 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, col_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=   2048, p= 51.37 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, col_major,  32,  32,  32,  16,  16,  32, 128,   1,   2, false, c, 2); // N=    512, p= 26.95 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, col_major, 128,  64,  32,  16,  64,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 31.35 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, col_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 29.93 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, col_major,  32,  32,  32,  16,  16,  32, 128,   1,   2, false, c, 2); // N=    512, p= 18.22 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, col_major,  64, 128,  32,  32,  64,  16, 128,   1,   2, false, c, 0); // N=   8192, p=119.63 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, col_major, 128,  64,  32,  64,  32,  16, 128,   1,   2, false, c, 1); // N=   2048, p=120.24 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, col_major,  64,  64,  32,  64,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 43.38 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, col_major,  64, 128,  32,  32,  32,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 64.00 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, col_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=   2048, p= 59.82 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, col_major,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 27.91 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, row_major,  64, 128,  32,  32,  32,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 51.04 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, row_major,  64, 128,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=   2048, p= 48.62 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, row_major,  32, 128,  32,  32,  32,  16, 128,   1,   2, false, c, 2); // N=    512, p= 21.84 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, row_major,  64, 128,  32,  16,  64,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 30.84 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, row_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 30.09 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, row_major,  32,  32,  32,  16,  16,  32, 128,   1,   2, false, c, 2); // N=    512, p= 17.75 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, row_major,  64, 128,  32,  64,  32,  16, 128,   1,   2, false, c, 0); // N=   8192, p=102.02 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, row_major, 128,  64,  32,  64,  32,  16, 128,   2,   2, false, c, 1); // N=   2048, p= 95.26 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, row_major, 128,  32,  32,  64,  16,  16, 128,   2,   2, false, c, 2); // N=    512, p= 35.24 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, row_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 64.55 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, row_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=   2048, p= 59.86 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, row_major,  32,  32,  32,  16,  16,  32, 128,   1,   2, false, c, 2); // N=    512, p= 26.39 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, conjugate, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 43.86 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 1); // N=   2048, p= 40.50 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 19.01 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   8192, p= 28.14 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, conjugate,  64,  32,  32,  16,  32,  32, 128,   1,   2, false, c, 1); // N=   2048, p= 27.00 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 15.12 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, conjugate, 128,  64,  32,  64,  32,  16, 128,   1,   2, false, c, 0); // N=   8192, p= 72.22 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, conjugate,  64,  64,  32,  64,  16,  32, 128,   2,   2, false, c, 1); // N=   2048, p= 63.47 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, conjugate, 128,  32,  32,  32,  16,  16, 256,   2,   2, false, c, 2); // N=    512, p= 26.11 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, conjugate,  64,  64,  32,  32,  32,  32, 128,   1,   2, false, c, 0); // N=   8192, p= 51.91 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 1); // N=   2048, p= 48.95 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, conjugate,  32,  32,  32,  16,  16,  16, 128,   2,   2, false, c, 2); // N=    512, p= 22.22 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, col_major,  64,  64,  32,  64,  16,  16, 128,   2,   2, false, c, 0); // N=   8192, p= 47.33 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, col_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 43.46 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, col_major,  32,  32,  32,  16,  16,  32, 128,   2,   2, false, c, 2); // N=    512, p= 21.75 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, col_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   8192, p= 28.64 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, col_major,  32,  64,  32,  16,  32,  32, 128,   1,   2, false, c, 1); // N=   2048, p= 27.08 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, col_major,  32,  32,  32,  16,  16,  16, 128,   2,   2, false, c, 2); // N=    512, p= 16.00 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, col_major,  64, 128,  32,  64,  32,  16, 128,   1,   2, false, c, 0); // N=   8192, p= 86.71 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, col_major,  64,  64,  32,  32,  16,  16, 256,   2,   2, false, c, 1); // N=   2048, p= 81.54 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, col_major,  32, 128,  32,  32,  16,  16, 256,   1,   2, false, c, 2); // N=    512, p= 31.99 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, col_major,  64,  64,  32,  32,  16,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 52.83 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, col_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 50.00 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, col_major,  32,  32,  32,  16,  16,  16, 128,   2,   2, false, c, 2); // N=    512, p= 23.00 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, row_major,  64,  64,  32,  64,  16,  16, 128,   2,   2, false, c, 0); // N=   8192, p= 45.84 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, row_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 42.46 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, row_major,  32,  32,  32,  16,  16,  16, 128,   2,   2, false, c, 2); // N=    512, p= 19.69 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, row_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   8192, p= 28.75 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, row_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 1); // N=   2048, p= 26.71 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, row_major,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 15.81 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, row_major,  64, 128,  32,  64,  32,  16, 128,   1,   2, false, c, 0); // N=   8192, p= 74.84 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, row_major,  64,  64,  32,  64,  16,  32, 128,   1,   2, false, c, 1); // N=   2048, p= 66.78 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, row_major,  32, 128,  32,  32,  16,  16, 256,   1,   2, false, c, 2); // N=    512, p= 26.72 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, row_major,  64,  64,  32,  32,  16,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 52.14 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, row_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 49.44 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, row_major,  32,  32,  32,  16,  16,  16, 128,   2,   2, false, c, 2); // N=    512, p= 22.97 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, conjugate,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 0); // N=   8192, p= 41.75 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, conjugate,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 39.53 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 16.93 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, conjugate,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 0); // N=   8192, p= 26.38 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, conjugate,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 24.12 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 14.00 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, conjugate,  64,  64,  32,  64,  16,  32, 128,   2,   2, false, c, 0); // N=   8192, p= 61.38 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, conjugate,  64,  64,  32,  64,  16,  32, 128,   2,   2, false, c, 1); // N=   2048, p= 57.45 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 21.49 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, conjugate,  64,  64,  32,  32,  16,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 49.81 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, conjugate,  64,  64,  32,  32,  16,  16, 256,   1,   2, false, c, 1); // N=   2048, p= 45.85 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 19.08 [TFlop/s]
#endif

#ifdef COMPILE_SGEMM_STRIDEDBATCH_KERNEL
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, col_major, 128,  64,  32,  32,  64,  32, 128,   1,   2, false, s, 0); // N=   1024, p= 44.68 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, col_major, 128,  64,  32,  32,  64,  32, 128,   1,   2, false, s, 1); // N=    256, p= 24.63 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, col_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  6.38 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, col_major, 128,  64,  32,  64,  32,  16, 128,   1,   2, false, s, 0); // N=   1024, p= 27.86 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, col_major, 128,  64,  32,  64,  32,  16, 128,   1,   2, false, s, 1); // N=    256, p= 21.60 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, col_major,  64,  64,  64,  32,  32,  32, 128,   1,   2, false, s, 2); // N=     64, p=  7.33 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, col_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 0); // N=   1024, p= 85.03 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, col_major, 128,  64,  32,  32,  64,  32, 128,   1,   2, false, s, 1); // N=    256, p= 41.12 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, col_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  9.22 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, col_major, 128, 128,  32,  64,  64,  16, 128,   2,   2, false, s, 0); // N=   1024, p= 67.49 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, col_major, 128,  64,  32,  32,  64,  32, 128,   1,   2, false, s, 1); // N=    256, p= 39.24 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, col_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  9.17 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, row_major, 128,  64,  32,  64,  32,  32, 128,   1,   2, false, s, 0); // N=   1024, p= 38.41 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, row_major, 128,  64,  32,  64,  32,  32, 128,   1,   2, false, s, 1); // N=    256, p= 26.91 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, row_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  8.14 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, row_major, 128,  64,  32,  32,  64,  16, 128,   2,   2, false, s, 0); // N=   1024, p= 28.10 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, row_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, s, 1); // N=    256, p= 21.80 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, row_major,  64,  64,  64,  32,  32,  32, 128,   1,   2, false, s, 2); // N=     64, p=  7.39 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, row_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 0); // N=   1024, p= 76.12 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, row_major, 128, 128,  32,  64,  64,  16, 128,   1,   2, false, s, 1); // N=    256, p= 39.12 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, row_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  9.02 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, row_major, 128, 128,  32,  64,  64,  32, 128,   1,   2, false, s, 0); // N=   1024, p= 68.45 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, row_major, 128,  64,  32,  64,  32,  32, 128,   1,   2, false, s, 1); // N=    256, p= 40.95 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, row_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  9.17 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, col_major, 128,  64,  32,  64,  32,  32, 128,   1,   2, false, s, 0); // N=   1024, p= 47.68 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, col_major, 128,  64,  32,  64,  32,  32, 128,   1,   2, false, s, 1); // N=    256, p= 33.01 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, col_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  8.47 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, col_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, s, 0); // N=   1024, p= 26.66 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, col_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, s, 1); // N=    256, p= 21.10 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, col_major,  64,  64,  64,  32,  32,  32, 128,   1,   2, false, s, 2); // N=     64, p=  7.17 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, col_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 0); // N=   1024, p= 90.90 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, col_major, 128,  64,  32,  32,  32,  32, 256,   1,   2, false, s, 1); // N=    256, p= 41.68 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, col_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  9.64 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, col_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 0); // N=   1024, p= 62.73 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, col_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, s, 1); // N=    256, p= 38.33 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, col_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  8.78 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, row_major, 128,  64,  32,  64,  32,  32, 128,   2,   2, false, s, 0); // N=   1024, p= 45.04 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, row_major, 128,  64,  32,  64,  32,  32, 128,   1,   2, false, s, 1); // N=    256, p= 30.68 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, row_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  8.18 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, row_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, s, 0); // N=   1024, p= 27.84 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, row_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, s, 1); // N=    256, p= 21.88 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, row_major,  64,  64,  64,  32,  32,  32, 128,   1,   2, false, s, 2); // N=     64, p=  7.36 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, row_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 0); // N=   1024, p= 83.69 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, row_major, 128,  64,  32,  64,  32,  32, 128,   2,   2, false, s, 1); // N=    256, p= 40.49 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, row_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  9.37 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, row_major, 128, 128,  32,  64,  64,  16, 128,   1,   2, false, s, 0); // N=   1024, p= 65.96 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, row_major, 128,  64,  32,  32,  64,  32, 128,   1,   2, false, s, 1); // N=    256, p= 39.27 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, row_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  9.17 [TFlop/s]
#endif
#ifdef COMPILE_CGEMM_STRIDEDBATCH_KERNEL
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, col_major,  64,  64,  32,  16,  64,  32, 128,   1,   2, false, c, 0); // N=   1024, p= 44.93 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, col_major,  64,  64,  32,  16,  64,  32, 128,   1,   2, false, c, 1); // N=    256, p= 30.66 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, col_major,  64,  32,  32,  16,  32,  32, 128,   2,   2, false, c, 2); // N=     64, p= 14.31 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, col_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 30.79 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, col_major,  64,  32,  32,  16,  32,  32, 128,   1,   2, false, c, 1); // N=    256, p= 24.96 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, col_major,  64,  32,  32,  16,  32,  16, 128,   2,   2, false, c, 2); // N=     64, p= 15.43 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, col_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 88.76 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, col_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, c, 1); // N=    256, p= 64.39 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, col_major,  64,  64,  64,  16,  64,  32, 128,   1,   2, false, c, 2); // N=     64, p= 24.05 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, col_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 0); // N=   1024, p= 61.04 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, col_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=    256, p= 46.78 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, col_major,  64,  64,  64,  32,  16,  32, 256,   1,   2, false, c, 2); // N=     64, p= 20.34 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, row_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   1024, p= 41.29 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, row_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 1); // N=    256, p= 32.49 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, row_major,  64,  64,  64,  32,  32,  16, 128,   1,   2, false, c, 2); // N=     64, p= 17.57 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, row_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 28.77 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, row_major,  64,  32,  32,  16,  32,  32, 128,   2,   2, false, c, 1); // N=    256, p= 24.71 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, row_major,  64,  32,  32,  16,  32,  16, 128,   1,   2, false, c, 2); // N=     64, p= 15.32 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, row_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 67.23 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, row_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, c, 1); // N=    256, p= 50.76 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, row_major,  64,  64,  64,  32,  32,  16, 128,   1,   2, false, c, 2); // N=     64, p= 21.30 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, row_major,  64,  64,  32,  32,  32,  32, 128,   2,   2, false, c, 0); // N=   1024, p= 57.31 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, row_major,  64,  64,  32,  32,  32,  32, 128,   2,   2, false, c, 1); // N=    256, p= 43.23 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, row_major,  64,  64,  64,  32,  16,  32, 256,   1,   2, false, c, 2); // N=     64, p= 20.34 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   1024, p= 38.97 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 1); // N=    256, p= 27.73 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, conjugate,  64,  32,  32,  32,  16,  16, 128,   1,   2, false, c, 2); // N=     64, p= 15.50 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, conjugate,  64,  32,  32,  16,  32,  32, 128,   1,   2, false, c, 0); // N=   1024, p= 27.11 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, conjugate,  64,  32,  32,  16,  32,  32, 128,   1,   2, false, c, 1); // N=    256, p= 22.45 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, conjugate,  64,  64,  64,  16,  32,  16, 256,   2,   2, false, c, 2); // N=     64, p= 14.28 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, conjugate, 128, 128,  32,  32,  64,  16, 256,   1,   2, false, c, 0); // N=   1024, p= 59.50 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, conjugate, 128, 128,  32,  32,  64,  16, 256,   1,   2, false, c, 1); // N=    256, p= 36.40 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, conjugate,  64,  64,  64,  32,  32,  16, 128,   1,   2, false, c, 2); // N=     64, p= 17.21 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, conjugate,  64,  64,  32,  32,  32,  32, 128,   1,   2, false, c, 0); // N=   1024, p= 52.08 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, conjugate,  64,  64,  32,  32,  16,  16, 256,   2,   2, false, c, 1); // N=    256, p= 37.48 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, conjugate,  64,  64,  64,  32,  16,  32, 256,   2,   2, false, c, 2); // N=     64, p= 18.75 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, col_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   1024, p= 51.34 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, col_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=    256, p= 39.90 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, col_major,  64,  32,  32,  32,  16,  32, 128,   2,   2, false, c, 2); // N=     64, p= 19.74 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, col_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 31.33 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, col_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 1); // N=    256, p= 25.08 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, col_major,  64,  64,  64,  16,  32,  16, 256,   2,   2, false, c, 2); // N=     64, p= 15.43 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, col_major, 128,  64,  32,  64,  32,  16, 128,   1,   2, false, c, 0); // N=   1024, p=119.62 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, col_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, c, 1); // N=    256, p= 77.28 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, col_major,  64,  64,  64,  64,  16,  32, 128,   2,   2, false, c, 2); // N=     64, p= 25.91 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, col_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 0); // N=   1024, p= 59.02 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, col_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=    256, p= 45.77 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, col_major,  64,  32,  32,  32,  16,  16, 128,   1,   2, false, c, 2); // N=     64, p= 20.46 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, row_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   1024, p= 47.92 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, row_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=    256, p= 38.19 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, row_major,  64,  64,  64,  32,  32,  16, 128,   2,   2, false, c, 2); // N=     64, p= 19.17 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, row_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 31.18 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, row_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 1); // N=    256, p= 24.90 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, row_major,  64,  32,  32,  16,  32,  16, 128,   2,   2, false, c, 2); // N=     64, p= 15.43 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, row_major, 128,  64,  32,  64,  32,  16, 128,   2,   2, false, c, 0); // N=   1024, p= 90.58 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, row_major, 128,  64,  32,  64,  32,  32, 128,   1,   2, false, c, 1); // N=    256, p= 65.40 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, row_major,  64,  64,  64,  64,  16,  32, 128,   2,   2, false, c, 2); // N=     64, p= 24.67 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, row_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 0); // N=   1024, p= 58.91 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, row_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=    256, p= 45.61 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, row_major,  64,  32,  32,  32,  16,  16, 128,   1,   2, false, c, 2); // N=     64, p= 20.46 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 44.86 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, conjugate, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=    256, p= 31.60 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, conjugate,  64,  32,  32,  32,  16,  32, 128,   1,   2, false, c, 2); // N=     64, p= 17.03 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   1024, p= 27.46 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, conjugate,  64,  32,  32,  16,  32,  32, 128,   1,   2, false, c, 1); // N=    256, p= 22.66 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, conjugate,  64,  64,  64,  16,  32,  16, 256,   2,   2, false, c, 2); // N=     64, p= 13.89 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, conjugate, 128, 128,  32,  64,  32,  16, 256,   1,   2, false, c, 0); // N=   1024, p= 76.47 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, conjugate, 128,  64,  32,  64,  16,  16, 256,   1,   2, false, c, 1); // N=    256, p= 46.43 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, conjugate,  64,  64,  64,  32,  16,  16, 256,   2,   2, false, c, 2); // N=     64, p= 20.84 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, conjugate,  64,  64,  32,  32,  32,  32, 128,   1,   2, false, c, 0); // N=   1024, p= 52.63 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, conjugate, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=    256, p= 36.62 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, conjugate,  64,  64,  64,  32,  16,  64, 256,   1,   2, false, c, 2); // N=     64, p= 18.44 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, col_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 47.84 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, col_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=    256, p= 32.18 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, col_major,  64,  64,  64,  32,  16,  16, 256,   2,   2, false, c, 2); // N=     64, p= 17.61 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, col_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 29.00 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, col_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 1); // N=    256, p= 21.77 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, col_major,  64,  64,  64,  16,  32,  16, 256,   2,   2, false, c, 2); // N=     64, p= 14.19 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, col_major, 128, 128,  32,  64,  32,  16, 256,   2,   2, false, c, 0); // N=   1024, p= 95.88 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, col_major, 128, 128,  32,  64,  32,  16, 256,   1,   2, false, c, 1); // N=    256, p= 47.06 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, col_major,  64,  64,  64,  32,  16,  16, 256,   2,   2, false, c, 2); // N=     64, p= 22.98 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, col_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 53.66 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, col_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=    256, p= 34.70 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, col_major,  64,  64,  64,  32,  16,  64, 256,   1,   2, false, c, 2); // N=     64, p= 18.34 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, row_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   1024, p= 45.18 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, row_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 1); // N=    256, p= 30.69 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, row_major,  64,  64,  64,  16,  32,  16, 256,   2,   2, false, c, 2); // N=     64, p= 17.16 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, row_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   1024, p= 27.39 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, row_major,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 1); // N=    256, p= 20.94 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, row_major,  64,  64,  64,  16,  32,  16, 256,   1,   2, false, c, 2); // N=     64, p= 14.28 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, row_major, 128, 128,  32,  64,  32,  16, 256,   1,   2, false, c, 0); // N=   1024, p= 78.89 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, row_major, 128, 128,  32,  64,  32,  16, 256,   1,   2, false, c, 1); // N=    256, p= 41.69 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, row_major,  64,  64,  64,  32,  16,  16, 256,   2,   2, false, c, 2); // N=     64, p= 20.84 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, row_major,  64,  64,  32,  32,  32,  32, 128,   1,   2, false, c, 0); // N=   1024, p= 53.13 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, row_major,  64,  64,  32,  32,  32,  32, 128,   1,   2, false, c, 1); // N=    256, p= 34.32 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, row_major,  64,  64,  64,  32,  16,  64, 256,   2,   2, false, c, 2); // N=     64, p= 18.49 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, conjugate,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 41.35 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, conjugate,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=    256, p= 26.40 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, conjugate,  64,  64,  64,  32,  16,  32, 256,   2,   2, false, c, 2); // N=     64, p= 15.90 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, conjugate,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 25.68 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 1); // N=    256, p= 18.68 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, conjugate,  64,  64,  64,  16,  32,  16, 256,   1,   2, false, c, 2); // N=     64, p= 13.29 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, conjugate, 128, 128,  32,  64,  32,  16, 256,   1,   2, false, c, 0); // N=   1024, p= 70.08 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, conjugate, 128, 128,  32,  64,  32,  16, 256,   1,   2, false, c, 1); // N=    256, p= 37.41 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, conjugate,  64,  64,  64,  32,  16,  16, 256,   2,   2, false, c, 2); // N=     64, p= 19.07 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, conjugate,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   1024, p= 47.19 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 1); // N=    256, p= 28.63 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, conjugate,  64,  32,  32,  16,  16,  16, 256,   1,   2, false, c, 2); // N=     64, p= 17.48 [TFlop/s]
#endif

#else // ENABLE_A100_OPTIMAZED_PARAMETERS
#ifdef COMPILE_SGEMM_KERNEL
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
#endif
#ifdef COMPILE_CGEMM_KERNEL
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
#endif

#ifdef COMPILE_SGEMM_STRIDEDBATCH_KERNEL
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
#endif
#ifdef COMPILE_CGEMM_STRIDEDBATCH_KERNEL
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
#endif
#endif

	(*handle)->exp_stats_enabled = false;
	(*handle)->buffer_length = 10000;
	(*handle)->ignore_threshold = 0;
	(*handle)->lost_threshold = 0;
	(*handle)->current_buffer_id = 1;
	(*handle)->counter_init_disabled = false;
	CUTF_CHECK_ERROR(hipMalloc    (&((*handle)->dev_lost_counter_buffer ), sizeof(cumpsgemm::counter_t) * (*handle)->buffer_length));
	CUTF_CHECK_ERROR(hipMalloc    (&((*handle)->dev_total_counter_buffer), sizeof(cumpsgemm::counter_t) * (*handle)->buffer_length));
	CUTF_CHECK_ERROR(hipHostMalloc(&((*handle)->host_lost_counter_buffer ), sizeof(cumpsgemm::counter_t) * (*handle)->buffer_length));
	CUTF_CHECK_ERROR(hipHostMalloc(&((*handle)->host_total_counter_buffer), sizeof(cumpsgemm::counter_t) * (*handle)->buffer_length));

	(*handle)->host_lost_counter_buffer [0] = 1;
	(*handle)->host_total_counter_buffer[0] = 1;

	return HIPBLAS_STATUS_SUCCESS;
}

hipblasStatus_t cuMpSGEMM_destroy(cuMpSGEMM_handle_t handle) {
	CUTF_CHECK_ERROR(hipFree    (handle->dev_lost_counter_buffer ));
	CUTF_CHECK_ERROR(hipFree    (handle->dev_total_counter_buffer));
	CUTF_CHECK_ERROR(hipHostFree(handle->host_lost_counter_buffer ));
	CUTF_CHECK_ERROR(hipHostFree(handle->host_total_counter_buffer));

	delete handle;
	return HIPBLAS_STATUS_SUCCESS;
}

hipblasStatus_t cuMpSGEMM_set_stream(cuMpSGEMM_handle_t handle, const hipStream_t hip_stream) {
	handle->hip_stream = hip_stream;
	return HIPBLAS_STATUS_SUCCESS;
}
}
