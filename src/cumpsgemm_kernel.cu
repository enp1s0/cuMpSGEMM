#include "hip/hip_runtime.h"
#include <cassert>
#include <type_traits>
#include <wmma_extension/utils.hpp>
#include <cutf/cuda.hpp>
#include <cutf/error.hpp>

#include <cumpsgemm/cumpsgemm.h>
#include "device_tcec_wrapper.hpp"

namespace {
constexpr unsigned smem_A_skew = 8;
constexpr unsigned smem_B_skew = 8;
constexpr unsigned smem_C_skew = 8;
constexpr unsigned warp_size = 32;

// smem size
template <unsigned SMEM_M, unsigned SMEM_N, unsigned SKEW, class Layout>
struct get_smem_size                                             {static constexpr unsigned value = (SMEM_M + SKEW) * SMEM_N;};
template <unsigned SMEM_M, unsigned SMEM_N, unsigned SKEW>
struct get_smem_size<SMEM_M, SMEM_N, SKEW, cumpsgemm::row_major> {static constexpr unsigned value = (SMEM_N + SKEW) * SMEM_M;};

// leading dimension
template <unsigned SMEM_M, unsigned SMEM_N, unsigned SKEW, class Layout>
struct get_smem_ld                                             {static constexpr unsigned value = SMEM_M + SKEW;};
template <unsigned SMEM_M, unsigned SMEM_N, unsigned SKEW>
struct get_smem_ld<SMEM_M, SMEM_N, SKEW, cumpsgemm::row_major> {static constexpr unsigned value = SMEM_N + SKEW;};

// smem index
template <unsigned SMEM_M, unsigned SMEM_N, unsigned SKEW, class Layout>
struct get_smem_index                                             {__device__ unsigned operator() (const unsigned m, const unsigned n) {return (m + n * (SMEM_M + SKEW));}};
template <unsigned SMEM_M, unsigned SMEM_N, unsigned SKEW>
struct get_smem_index<SMEM_M, SMEM_N, SKEW, cumpsgemm::row_major> {__device__ unsigned operator() (const unsigned m, const unsigned n) {return (n + m * (SMEM_N + SKEW));}};

// zero
template <class T>
__device__ T zero() {return 0;}
template <> __device__ hipComplex zero<hipComplex>() {return make_hipComplex(0, 0);}

// Dmem loader
template <class T, unsigned SMEM_M, unsigned SMEM_N, unsigned SKEW, unsigned BLOCK_SIZE>
struct dmem_loader_core {
	__device__ dmem_loader_core(){}
	__device__ void operator() (
			T* const smem_ptr,
			const T* const dmem_ptr,
			const std::size_t ld,
			const std::size_t start_m,
			const std::size_t start_n,
			const std::size_t size_m,
			const std::size_t size_n
			) {
		if (start_m + SMEM_M < size_m && start_n + SMEM_N < size_n) {
			for (unsigned offset = 0; offset < SMEM_M * SMEM_N; offset += BLOCK_SIZE) {
				const auto index = offset + threadIdx.x;
				const auto m = index % SMEM_M;
				const auto n = index / SMEM_M;
				const auto smem_index = m + n * (SMEM_M + SKEW);
				const auto dmem_index = (start_m + m) + (start_n + n) * ld;
				smem_ptr[smem_index] = dmem_ptr[dmem_index];
			}
		} else {
			for (unsigned offset = 0; offset < SMEM_M * SMEM_N; offset += BLOCK_SIZE) {
				const auto index = offset + threadIdx.x;
				const auto m = index % SMEM_M;
				const auto n = index / SMEM_M;
				const auto smem_index = m + n * (SMEM_M + SKEW);
				const auto dmem_index = (start_m + m) + (start_n + n) * ld;

				T v = zero<T>();
				if ((start_m + m) < size_m && (start_n + n) < size_n) {
					v = dmem_ptr[dmem_index];
				}
				__syncwarp();
				smem_ptr[smem_index] = v;
			}
		}
	}
};

template <class _Layout, class T, unsigned SMEM_M, unsigned SMEM_N, unsigned SKEW, unsigned BLOCK_SIZE>
struct dmem_loader {
	using Layout = _Layout;
	__device__ dmem_loader(){}
	__device__ void operator() (
			T* const smem_ptr,
			const T* const dmem_ptr,
			const std::size_t ld,
			const std::size_t start_m,
			const std::size_t start_n,
			const std::size_t size_m,
			const std::size_t size_n
			) {
		dmem_loader_core<T, SMEM_M, SMEM_N, SKEW, BLOCK_SIZE>{}(
				smem_ptr,
				dmem_ptr,
				ld,
				start_m, start_n,
				size_m, size_n
				);
	}
};

template <class T, unsigned SMEM_M, unsigned SMEM_N, unsigned SKEW, unsigned BLOCK_SIZE>
struct dmem_loader<cumpsgemm::row_major, T, SMEM_M, SMEM_N, SKEW, BLOCK_SIZE> {
	using Layout = cumpsgemm::row_major;
	__device__ dmem_loader(){}
	__device__ void operator() (
			T* const smem_ptr,
			const T* const dmem_ptr,
			const std::size_t ld,
			const std::size_t start_m,
			const std::size_t start_n,
			const std::size_t size_m,
			const std::size_t size_n
			) {
		dmem_loader_core<T, SMEM_N, SMEM_M, SKEW, BLOCK_SIZE>{}(
				smem_ptr,
				dmem_ptr,
				ld,
				start_n, start_m,
				size_n, size_m
				);
	}
};

template <unsigned SMEM_M, unsigned SMEM_N, unsigned SKEW, unsigned BLOCK_SIZE>
struct dmem_loader<cumpsgemm::conjugate, hipComplex, SMEM_M, SMEM_N, SKEW, BLOCK_SIZE> {
	using Layout = cumpsgemm::col_major;
	__device__ dmem_loader(){}
	__device__ void operator() (
			hipComplex* const smem_ptr,
			const hipComplex* const dmem_ptr,
			const std::size_t ld,
			const std::size_t start_m,
			const std::size_t start_n,
			const std::size_t size_m,
			const std::size_t size_n
			) {
		if (start_m + SMEM_M < size_m && start_n + SMEM_N < size_n) {
			for (unsigned offset = 0; offset < SMEM_M * SMEM_N; offset += BLOCK_SIZE) {
				const auto index = offset + threadIdx.x;
				const auto m = index % SMEM_M;
				const auto n = index / SMEM_M;
				const auto smem_index = m + n * (SMEM_M + SKEW);
				const auto dmem_index = (start_m + m) + (start_n + n) * ld;
				const auto v = dmem_ptr[dmem_index];
				smem_ptr[smem_index] = make_hipComplex(v.x, -v.y);
			}
		} else {
			for (unsigned offset = 0; offset < SMEM_M * SMEM_N; offset += BLOCK_SIZE) {
				const auto index = offset + threadIdx.x;
				const auto m = index % SMEM_M;
				const auto n = index / SMEM_M;
				const auto smem_index = m + n * (SMEM_M + SKEW);
				const auto dmem_index = (start_m + m) + (start_n + n) * ld;

				auto v = zero<hipComplex>();
				if ((start_m + m) < size_m && (start_n + n) < size_n) {
					const auto w = dmem_ptr[dmem_index];
					v = make_hipComplex(w.x, -w.y);
				}
				__syncwarp();
				smem_ptr[smem_index] = v;
			}
		}
	}
};

template <unsigned SMEM_M, unsigned SMEM_N, unsigned SKEW, unsigned BLOCK_SIZE>
struct dmem_loader<cumpsgemm::conjugate, float, SMEM_M, SMEM_N, SKEW, BLOCK_SIZE> {
	using Layout = cumpsgemm::col_major;
	__device__ dmem_loader(){}
	__device__ void operator() (
			float* const,
			const float* const,
			const std::size_t,
			const std::size_t,
			const std::size_t,
			const std::size_t,
			const std::size_t
			) {
		// Do nothing, only for suppressing compilation error.
	}
};

template <class T>
__device__ T mul(const T a, const T alpha) {
	return a * alpha;
}
template <>
__device__ hipComplex mul<hipComplex>(const hipComplex a, const hipComplex alpha) {
	return make_hipComplex(a.x * alpha.x - a.y * alpha.y, a.y * alpha.x + a.x * alpha.y);
}

template <class T>
__device__ T mad(const T a, const T alpha, const T b) {
	return a * alpha + b;
}
template <>
__device__ hipComplex mad<hipComplex>(const hipComplex a, const hipComplex alpha, const hipComplex b) {
	return make_hipComplex(
			a.x * alpha.x - a.y * alpha.y + b.x,
			a.y * alpha.x + a.x * alpha.y + b.y
			);
}

template<class T>
__device__ bool is_zero(const T& v) {
	return v == 0;
}
template <>
__device__ bool is_zero(const hipComplex& v) {
	return v.x == 0 && v.y == 0;
}

template <class T, unsigned SMEM_M, unsigned SMEM_N, unsigned SKEW, unsigned BLOCK_SIZE>
struct dmem_storer {
	__device__ dmem_storer(){}
	__device__ void operator() (
			T* const dmem_ptr,
			const std::size_t ld,
			const std::size_t start_m,
			const std::size_t start_n,
			const std::size_t size_m,
			const std::size_t size_n,
			const T* const smem_ptr,
			const T alpha, const T beta
			) {
		if (is_zero(beta)) {
			if (start_m + SMEM_M < size_m && start_n + SMEM_N < size_n) {
				for (unsigned offset = 0; offset < SMEM_M * SMEM_N; offset += BLOCK_SIZE) {
					const auto index = offset + threadIdx.x;
					const auto m = index % SMEM_M;
					const auto n = index / SMEM_M;
					const auto smem_index = m + n * (SMEM_M + SKEW);
					const auto dmem_index = (start_m + m) + (start_n + n) * ld;
					dmem_ptr[dmem_index] = mul(smem_ptr[smem_index], alpha);
				}
			} else {
				for (unsigned offset = 0; offset < SMEM_M * SMEM_N; offset += BLOCK_SIZE) {
					const auto index = offset + threadIdx.x;
					const auto m = index % SMEM_M;
					const auto n = index / SMEM_M;
					const auto smem_index = m + n * (SMEM_M + SKEW);
					const auto dmem_index = (start_m + m) + (start_n + n) * ld;

					if ((start_m + m) < size_m && (start_n + n) < size_n) {
						dmem_ptr[dmem_index] = mul(smem_ptr[smem_index], alpha);
					}
					__syncwarp();
				}
			}
		} else {
			if (start_m + SMEM_M < size_m && start_n + SMEM_N < size_n) {
				for (unsigned offset = 0; offset < SMEM_M * SMEM_N; offset += BLOCK_SIZE) {
					const auto index = offset + threadIdx.x;
					const auto m = index % SMEM_M;
					const auto n = index / SMEM_M;
					const auto smem_index = m + n * (SMEM_M + SKEW);
					const auto dmem_index = (start_m + m) + (start_n + n) * ld;
					dmem_ptr[dmem_index] = mad(smem_ptr[smem_index], alpha, mul(dmem_ptr[dmem_index], beta));
				}
			} else {
				for (unsigned offset = 0; offset < SMEM_M * SMEM_N; offset += BLOCK_SIZE) {
					const auto index = offset + threadIdx.x;
					const auto m = index % SMEM_M;
					const auto n = index / SMEM_M;
					const auto smem_index = m + n * (SMEM_M + SKEW);
					const auto dmem_index = (start_m + m) + (start_n + n) * ld;

					if ((start_m + m) < size_m && (start_n + n) < size_n) {
						dmem_ptr[dmem_index] = mad(smem_ptr[smem_index], alpha, mul(dmem_ptr[dmem_index], beta));
					}
					__syncwarp();
				}
			}
		}
	}
};

template <
	class T,
	unsigned SMEM_M,
	unsigned SMEM_N,
	unsigned SMEM_K,
	unsigned FRAG_M,
	unsigned FRAG_N,
	unsigned FRAG_K,
	unsigned BLOCK_SIZE,
	class OP_A,
	class OP_B,
	class TC_T,
	class EC
>
__device__ void mma_smem (
		cumpsgemm::device::tc_fragment<T, nvcuda::wmma::accumulator, FRAG_M, FRAG_N, FRAG_K, void, TC_T, EC> frag_c[SMEM_M * SMEM_N / (FRAG_M * FRAG_N) / (BLOCK_SIZE / warp_size)],
		const T* const a_smem_ptr,
		const T* const b_smem_ptr
		) {
	static_assert((SMEM_M / FRAG_M) * (SMEM_N / FRAG_N) >= (BLOCK_SIZE / warp_size));
	for (unsigned i = threadIdx.x / warp_size; i < (SMEM_M / FRAG_M) * (SMEM_N / FRAG_N); i += BLOCK_SIZE / warp_size) {
		const unsigned bm = i % (SMEM_M / FRAG_M);
		const unsigned bn = i / (SMEM_M / FRAG_M);

		for (unsigned k = 0; k < SMEM_K; k += FRAG_K) {
			cumpsgemm::device::tc_fragment<T, nvcuda::wmma::matrix_a, FRAG_M, FRAG_N, FRAG_K, OP_A, TC_T, EC> frag_a;
			cumpsgemm::device::load_matrix<OP_A>(frag_a, a_smem_ptr + get_smem_index<SMEM_M, SMEM_K, smem_A_skew, OP_A>{}(bm * FRAG_M, k), get_smem_ld<SMEM_M, SMEM_K, smem_A_skew, OP_A>::value);

			cumpsgemm::device::tc_fragment<T, nvcuda::wmma::matrix_b, FRAG_M, FRAG_N, FRAG_K, OP_B, TC_T, EC> frag_b;
			cumpsgemm::device::load_matrix<OP_B>(frag_b, b_smem_ptr + get_smem_index<SMEM_K, SMEM_N, smem_B_skew, OP_B>{}(k, bn * FRAG_N), get_smem_ld<SMEM_K, SMEM_N, smem_B_skew, OP_B>::value);

			cumpsgemm::device::mma(frag_c[i], frag_a, frag_b, frag_c[i]);
		}
	}
}

template <
	class T,
	unsigned SMEM_M,
	unsigned SMEM_N,
	unsigned SMEM_K,
	unsigned FRAG_M,
	unsigned FRAG_N,
	unsigned FRAG_K,
	unsigned BLOCK_SIZE,
	class A_DMEM_LOADER,
	class B_DMEM_LOADER,
	class C_DMEM_STORER,
	class TC_T,
	class EC
>
__global__ void gemm_kernel(
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const T alpha,
		const T* const a_dmem_ptr, const uint64_t lda,
		const T* const b_dmem_ptr, const uint64_t ldb,
		const T beta,
		T* const c_dmem_ptr, const uint64_t ldc
		) {
	extern __shared__ uint8_t smem_base[];
	T* smem = reinterpret_cast<T*>(smem_base);
	T* const a_smem_ptr = smem;
	T* const b_smem_ptr = smem + get_smem_size<SMEM_M, SMEM_K, smem_A_skew, typename A_DMEM_LOADER::Layout>::value * 2;

	A_DMEM_LOADER a_dmem_loader;
	B_DMEM_LOADER b_dmem_loader;

	constexpr unsigned frag_c_array_size = SMEM_M * SMEM_N / (FRAG_M * FRAG_N) / (BLOCK_SIZE / warp_size);
	cumpsgemm::device::tc_fragment<T, nvcuda::wmma::accumulator, FRAG_M, FRAG_N, FRAG_K, void, TC_T, EC> frag_c[frag_c_array_size];
	for (unsigned i = 0; i < frag_c_array_size; i++) {
		cumpsgemm::device::fill_zero(frag_c[i]);
	}

	unsigned smem_buffer_id = 0;
	a_dmem_loader(
			a_smem_ptr + get_smem_size<SMEM_M, SMEM_K, smem_A_skew, typename A_DMEM_LOADER::Layout>::value * smem_buffer_id,
			a_dmem_ptr,
			lda,
			blockIdx.x * SMEM_M, 0,
			m, k
			);
	b_dmem_loader(
			b_smem_ptr + get_smem_size<SMEM_K, SMEM_N, smem_B_skew, typename B_DMEM_LOADER::Layout>::value * smem_buffer_id,
			b_dmem_ptr,
			ldb,
			0, blockIdx.y * SMEM_N,
			k, n
			);
	__syncthreads();
	for (uint64_t bk = SMEM_K; bk < k; bk += SMEM_K) {
		smem_buffer_id = (bk / SMEM_K) % 2;
		a_dmem_loader(
				a_smem_ptr + get_smem_size<SMEM_M, SMEM_K, smem_A_skew, typename A_DMEM_LOADER::Layout>::value * smem_buffer_id,
				a_dmem_ptr,
				lda,
				blockIdx.x * SMEM_M, bk,
				m, k
				);
		b_dmem_loader(
				b_smem_ptr + get_smem_size<SMEM_K, SMEM_N, smem_B_skew, typename B_DMEM_LOADER::Layout>::value * smem_buffer_id,
				b_dmem_ptr,
				ldb,
				bk, blockIdx.y * SMEM_N,
				k, n
				);
		mma_smem<
			T,
			SMEM_M, SMEM_N, SMEM_K,
			FRAG_M, FRAG_N, FRAG_K,
			BLOCK_SIZE,
			typename A_DMEM_LOADER::Layout,
			typename B_DMEM_LOADER::Layout,
			TC_T,
			EC>(
					frag_c,
					a_smem_ptr + get_smem_size<SMEM_M, SMEM_K, smem_A_skew, typename A_DMEM_LOADER::Layout>::value * (1 - smem_buffer_id),
					b_smem_ptr + get_smem_size<SMEM_K, SMEM_N, smem_B_skew, typename B_DMEM_LOADER::Layout>::value * (1 - smem_buffer_id)
				 );
		__syncthreads();
	}
	mma_smem<
		T,
		SMEM_M, SMEM_N, SMEM_K,
		FRAG_M, FRAG_N, FRAG_K,
		BLOCK_SIZE,
		typename A_DMEM_LOADER::Layout,
		typename B_DMEM_LOADER::Layout,
		TC_T,
		EC>(
				frag_c,
				a_smem_ptr + get_smem_size<SMEM_M, SMEM_K, smem_A_skew, typename A_DMEM_LOADER::Layout>::value * smem_buffer_id,
				b_smem_ptr + get_smem_size<SMEM_K, SMEM_N, smem_B_skew, typename B_DMEM_LOADER::Layout>::value * smem_buffer_id
			 );
	__syncthreads();

	// register to smem
	for (unsigned i = threadIdx.x / warp_size; i < (SMEM_M / FRAG_M) * (SMEM_N / FRAG_N); i += BLOCK_SIZE / warp_size) {
		const unsigned bm = i % (SMEM_M / FRAG_M);
		const unsigned bn = i / (SMEM_M / FRAG_M);
		cumpsgemm::device::store_matrix(
				smem + get_smem_index<SMEM_M, SMEM_N, smem_C_skew, cumpsgemm::col_major>{}(
					bm * FRAG_M, bn * FRAG_N
					),
				frag_c[i],
				SMEM_M + smem_C_skew
				);
	}
	__syncthreads();

	C_DMEM_STORER c_dmem_storer;
	c_dmem_storer(
			c_dmem_ptr, ldc,
			blockIdx.x * SMEM_M, blockIdx.y * SMEM_N,
			m, n,
			smem,
			alpha, beta
			);
}

template <class T>
using kernel_func_t = void (*)(
			const uint64_t,
			const uint64_t,
			const uint64_t,
			const T,
			const T* const, const uint64_t,
			const T* const, const uint64_t,
			const T,
			T* const, const uint64_t
			);


template <
	class T,
	unsigned SMEM_M,
	unsigned SMEM_N,
	unsigned SMEM_K,
	unsigned FRAG_M,
	unsigned FRAG_N,
	unsigned FRAG_K,
	unsigned BLOCK_SIZE,
	class OP_A,
	class OP_B,
	class TC_T,
	class EC
>
kernel_func_t<T> get_kernel_func_ptr() {
	constexpr kernel_func_t<T> func_ptr = &(gemm_kernel<
		T,
		SMEM_M, SMEM_N, SMEM_K,
		FRAG_M, FRAG_N, FRAG_K,
		BLOCK_SIZE,
		dmem_loader<OP_A, T, SMEM_M, SMEM_K, smem_A_skew, BLOCK_SIZE>,
		dmem_loader<OP_B, T, SMEM_K, SMEM_N, smem_B_skew, BLOCK_SIZE>,
		dmem_storer<T, SMEM_M, SMEM_N, smem_C_skew, BLOCK_SIZE>,
		TC_T,
		EC
	>);
	return func_ptr;
}

template <
	class T,
	unsigned SMEM_M,
	unsigned SMEM_N,
	unsigned SMEM_K,
	class OP_A,
	class OP_B
	>
unsigned get_total_smem_size() {
	return sizeof(T) * std::max<unsigned>(
			(SMEM_M + smem_C_skew) * SMEM_N,
			2 * (
				get_smem_size<SMEM_M, SMEM_K, smem_A_skew, OP_A>::value +
				get_smem_size<SMEM_K, SMEM_N, smem_B_skew, OP_B>::value
				));
}

template <
	class T,
	unsigned SMEM_M,
	unsigned SMEM_N,
	unsigned SMEM_K,
	unsigned FRAG_M,
	unsigned FRAG_N,
	unsigned FRAG_K,
	unsigned BLOCK_SIZE,
	class OP_A,
	class OP_B,
	class TC_T,
	class EC
>
void launch_kernel (
			const std::size_t m,
			const std::size_t n,
			const std::size_t k,
			const T alpha,
			const T* const a_ptr, const std::size_t lda,
			const T* const b_ptr, const std::size_t ldb,
			const T beta,
			T* const c_ptr, const std::size_t ldc,
			hipStream_t hip_stream
		) {
	const auto smem_size_in_byte = get_total_smem_size<
		T,
		SMEM_M, SMEM_N, SMEM_K,
		OP_A, OP_B>();
	const auto kernel_ptr = get_kernel_func_ptr<
		T,
		SMEM_M, SMEM_N, SMEM_K,
		FRAG_M, FRAG_N, FRAG_K,
		BLOCK_SIZE,
		OP_A, OP_B,
		TC_T, EC>();
	CUTF_CHECK_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_ptr), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size_in_byte));

	const dim3 block_size(BLOCK_SIZE);
	const dim3 grid_size(
			(m + SMEM_M - 1) / SMEM_M,
			(n + SMEM_N - 1) / SMEM_N
			);

	kernel_ptr<<<grid_size, block_size, smem_size_in_byte, hip_stream>>>(
			m, n, k,
			alpha,
			a_ptr, lda,
			b_ptr, ldb,
			beta,
			c_ptr, ldc
			);
}

template <
	class T,
	unsigned SMEM_M,
	unsigned SMEM_N,
	unsigned SMEM_K,
	unsigned FRAG_M,
	unsigned FRAG_N,
	unsigned FRAG_K,
	unsigned BLOCK_SIZE,
	class TC_T,
	class EC
>
void layout_selector (
			const hipblasOperation_t op_A,
			const hipblasOperation_t op_B,
			const std::size_t m,
			const std::size_t n,
			const std::size_t k,
			const T alpha,
			const T* const a_ptr, const std::size_t lda,
			const T* const b_ptr, const std::size_t ldb,
			const T beta,
			T* const c_ptr, const std::size_t ldc,
			hipStream_t hip_stream
		) {
	assert((std::is_same<T, float>::value && !(op_A == HIPBLAS_OP_C || op_B == HIPBLAS_OP_C)));
#define CASE(A, a, B, b) \
	if (op_A == a && op_B == b) {launch_kernel<T, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, A, B, TC_T, EC>(m, n, k, alpha, a_ptr, lda, b_ptr, ldb, beta, c_ptr, ldc, hip_stream);return;}

	CASE(cumpsgemm::col_major, HIPBLAS_OP_N, cumpsgemm::col_major, HIPBLAS_OP_N);
	CASE(cumpsgemm::row_major, HIPBLAS_OP_T, cumpsgemm::col_major, HIPBLAS_OP_N);
	CASE(cumpsgemm::conjugate, HIPBLAS_OP_C, cumpsgemm::col_major, HIPBLAS_OP_N);
	CASE(cumpsgemm::col_major, HIPBLAS_OP_N, cumpsgemm::row_major, HIPBLAS_OP_T);
	CASE(cumpsgemm::row_major, HIPBLAS_OP_T, cumpsgemm::row_major, HIPBLAS_OP_T);
	CASE(cumpsgemm::conjugate, HIPBLAS_OP_C, cumpsgemm::row_major, HIPBLAS_OP_T);
	CASE(cumpsgemm::col_major, HIPBLAS_OP_N, cumpsgemm::conjugate, HIPBLAS_OP_C);
	CASE(cumpsgemm::row_major, HIPBLAS_OP_T, cumpsgemm::conjugate, HIPBLAS_OP_C);
	CASE(cumpsgemm::conjugate, HIPBLAS_OP_C, cumpsgemm::conjugate, HIPBLAS_OP_C);
}
} // noname namespace

extern "C" hipblasStatus_t cuMpSGEMM_sgemm(
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const float* alpha,
		const float* const a_dmem_ptr, const uint64_t lda,
		const float* const b_dmem_ptr, const uint64_t ldb,
		const float* beta,
		float* const c_dmem_ptr, const uint64_t ldc,
		const cuMpSGEMM_compute_mode_t compute_mode,
		hipStream_t hip_stream
		) {
	constexpr unsigned SMEM_M = 64;
	constexpr unsigned SMEM_N = 64;
	constexpr unsigned SMEM_K = 64;
	constexpr unsigned FRAG_M = 32;
	constexpr unsigned FRAG_N = 32;
	constexpr unsigned FRAG_K = 32;
	constexpr unsigned BLOCK_SIZE = 128;
	switch (compute_mode) {
	case CUMPSGEMM_FP16TC:   layout_selector<float, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, half                         , mtk::wmma::tcec::without_ec>(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, *beta, c_dmem_ptr, ldc, hip_stream);break;
	case CUMPSGEMM_FP16TCEC: layout_selector<float, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, half                         , mtk::wmma::tcec::with_ec   >(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, *beta, c_dmem_ptr, ldc, hip_stream);break;
	case CUMPSGEMM_TF32TC:   layout_selector<float, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, nvcuda::wmma::precision::tf32, mtk::wmma::tcec::without_ec>(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, *beta, c_dmem_ptr, ldc, hip_stream);break;
	case CUMPSGEMM_TF32TCEC: layout_selector<float, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, nvcuda::wmma::precision::tf32, mtk::wmma::tcec::with_ec   >(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, *beta, c_dmem_ptr, ldc, hip_stream);break;
	default:break;
	}

	return HIPBLAS_STATUS_SUCCESS;
}

extern "C" hipblasStatus_t cuMpSGEMM_cgemm(
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const hipComplex* alpha,
		const hipComplex* const a_dmem_ptr, const uint64_t lda,
		const hipComplex* const b_dmem_ptr, const uint64_t ldb,
		const hipComplex* beta,
		hipComplex* const c_dmem_ptr, const uint64_t ldc,
		const cuMpSGEMM_compute_mode_t compute_mode,
		hipStream_t hip_stream
		) {
	constexpr unsigned SMEM_M = 64;
	constexpr unsigned SMEM_N = 64;
	constexpr unsigned SMEM_K = 64;
	constexpr unsigned FRAG_M = 32;
	constexpr unsigned FRAG_N = 32;
	constexpr unsigned FRAG_K = 32;
	constexpr unsigned BLOCK_SIZE = 128;
	switch (compute_mode) {
	case CUMPSGEMM_FP16TC:   layout_selector<hipComplex, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, half                         , mtk::wmma::tcec::without_ec>(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, *beta, c_dmem_ptr, ldc, hip_stream);break;
	case CUMPSGEMM_FP16TCEC: layout_selector<hipComplex, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, half                         , mtk::wmma::tcec::with_ec   >(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, *beta, c_dmem_ptr, ldc, hip_stream);break;
	case CUMPSGEMM_TF32TC:   layout_selector<hipComplex, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, nvcuda::wmma::precision::tf32, mtk::wmma::tcec::without_ec>(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, *beta, c_dmem_ptr, ldc, hip_stream);break;
	case CUMPSGEMM_TF32TCEC: layout_selector<hipComplex, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, nvcuda::wmma::precision::tf32, mtk::wmma::tcec::with_ec   >(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, *beta, c_dmem_ptr, ldc, hip_stream);break;
	default:break;
	}

	return HIPBLAS_STATUS_SUCCESS;
}
