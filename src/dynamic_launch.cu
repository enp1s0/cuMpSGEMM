#include "hip/hip_runtime.h"
#include <cutf/memory.hpp>
#include <cumpsgemm/hijack_control.hpp>
#include "dynamic_launch.hpp"

namespace {
__global__ void init_flag_buffer(
		int* const ptr
		) {
	ptr[0] = CUMPSGEMM_TF32TCEC;
	ptr[1] = CUMPSGEMM_FP16TCEC;
}
} // unnamed namespace

void init_dynamic_launch_flag_buffer(
		cuMpSGEMM_handle* handle
		) {
	handle->dynamic_launch_handle = new cumpsgemm::dynamic_launch::dynamic_launch_handle;

	handle->dynamic_launch_handle->flag_buffer_length = 10000;
	handle->dynamic_launch_handle->current_buffer_id = 1;
	handle->dynamic_launch_handle->enabled = false;
	handle->dynamic_launch_handle->enabled_id = 0;
	handle->dynamic_launch_handle->mode_A = CUMPSGEMM_TF32TCEC;
	handle->dynamic_launch_handle->mode_B = CUMPSGEMM_FP16TCEC;

	CUTF_CHECK_ERROR(hipMalloc(&handle->dynamic_launch_handle->frag_buffer, sizeof(int) * handle->dynamic_launch_handle->flag_buffer_length));
	init_flag_buffer<<<1, 1, 0, handle->hip_stream>>>(handle->dynamic_launch_handle->frag_buffer);
}

void destroy_launch_flag_buffer(
		cuMpSGEMM_handle* handle
		) {
	CUTF_CHECK_ERROR(hipFree(handle->dynamic_launch_handle->frag_buffer));
	delete handle->dynamic_launch_handle;
}

unsigned cumpsgemm::dynamic_launch::get_next_dynamic_launch_flag_buffer_id(
		cuMpSGEMM_handle* handle
		) {
	const auto next = ++(handle->dynamic_launch_handle->current_buffer_id);
	if (next < handle->dynamic_launch_handle->flag_buffer_length) {
		return next;
	}
	handle->dynamic_launch_handle->flag_buffer_length = 2;
	return 2;
}

void cumpsgemm::dynamic_launch::set_dynamic_launch_flag_buffer_id(
		cuMpSGEMM_handle* handle,
		const unsigned id
		) {
	handle->dynamic_launch_handle->enabled_id = id;
	handle->dynamic_launch_handle->enabled = 1;
}

void cumpsgemm::dynamic_launch::unset_dynamic_launch_flag_buffer_id(
		cuMpSGEMM_handle* handle
		) {
	handle->dynamic_launch_handle->enabled_id = 0;
	handle->dynamic_launch_handle->enabled = 0;
}

void cumpsgemm::dynamic_launch::set_compute_mode_AB(
		cuMpSGEMM_handle* handle,
		const cuMpSGEMM_compute_mode_t mode_A,
		const cuMpSGEMM_compute_mode_t mode_B) {
	switch (mode_A) {
	case CUMPSGEMM_TF32TCEC:
	case CUMPSGEMM_FP16TCEC:
	case CUMPSGEMM_FP16TC:
	case CUMPSGEMM_TF32TC:
		break;
	default:
		throw std::runtime_error(std::string("mode A must be an SGEMM emulation method. @") + __func__);
		break;
	}
	switch (mode_B) {
	case CUMPSGEMM_TF32TCEC:
	case CUMPSGEMM_FP16TCEC:
	case CUMPSGEMM_FP16TC:
	case CUMPSGEMM_TF32TC:
		break;
	default:
		throw std::runtime_error(std::string("mode B must be an SGEMM emulation method. @") + __func__);
		break;
	}
	handle->dynamic_launch_handle->mode_A = mode_A;
	handle->dynamic_launch_handle->mode_B = mode_B;
}
