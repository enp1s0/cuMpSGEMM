#include <iostream>
#include <cassert>
#include <type_traits>
#include <hipblas.h>
#include <cutf/cuda.hpp>
#include <cumpsgemm/cumpsgemm.hpp>

#include "handle.hpp"
#include "exp_stats.hpp"
#include "dynamic_launch.hpp"

// For debug
//#define CUMPSGEMM_CHECK_KERNEL_ERROR

namespace {
template <class T>
cumpsgemm::kernel_module_code::code_t gen_module_code(
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const cuMpSGEMM_compute_mode_t compute_mode
		) {
	cumpsgemm::kernel_module_code::code_t code = 0;
	switch (compute_mode) {
	case CUMPSGEMM_FP16TC:   code |= cumpsgemm::kernel_module_code::half | cumpsgemm::kernel_module_code::without_ec;break;
	case CUMPSGEMM_FP16TCEC: code |= cumpsgemm::kernel_module_code::half | cumpsgemm::kernel_module_code::with_ec   ;break;
	case CUMPSGEMM_TF32TC:   code |= cumpsgemm::kernel_module_code::tf32 | cumpsgemm::kernel_module_code::without_ec;break;
	case CUMPSGEMM_TF32TCEC: code |= cumpsgemm::kernel_module_code::tf32 | cumpsgemm::kernel_module_code::with_ec   ;break;
	default:break;
	}
	switch (op_A) {
	case HIPBLAS_OP_N: code |= cumpsgemm::kernel_module_code::op_a_col_major;break;
	case HIPBLAS_OP_T: code |= cumpsgemm::kernel_module_code::op_a_row_major;break;
	case HIPBLAS_OP_C: code |= cumpsgemm::kernel_module_code::op_a_conjugate;break;
	default:break;
	}
	switch (op_B) {
	case HIPBLAS_OP_N: code |= cumpsgemm::kernel_module_code::op_b_col_major;break;
	case HIPBLAS_OP_T: code |= cumpsgemm::kernel_module_code::op_b_row_major;break;
	case HIPBLAS_OP_C: code |= cumpsgemm::kernel_module_code::op_b_conjugate;break;
	default:break;
	}
	if (std::is_same<T, float>::value) {
		code |= cumpsgemm::kernel_module_code::s;
	} else if (std::is_same<T, hipComplex>::value) {
		code |= cumpsgemm::kernel_module_code::c;
	}

	assert(code <= cumpsgemm::kernel_module_code::max_code);

	return code;
}

template <class T>
void launch_kernel (
			const cumpsgemm::gemm_module gemm_module,
			const int* const dynamic_launch_buffer_ptr,
			const std::size_t m,
			const std::size_t n,
			const std::size_t k,
			const T alpha,
			const T* const a_ptr, const std::size_t lda,
			const T* const b_ptr, const std::size_t ldb,
			const T beta,
			T* const c_ptr, const std::size_t ldc,
			hipStream_t hip_stream
		) {
	const auto kernel_ptr = reinterpret_cast<cumpsgemm::gemm_kernel_func_t<T>>(gemm_module.kernel_func);
	const dim3 block_size(gemm_module.block_size);
	const dim3 grid_size(
			((m + gemm_module.smem_m - 1) / gemm_module.smem_m) * ((n + gemm_module.smem_n - 1) / gemm_module.smem_n)
			);

	kernel_ptr<<<grid_size, block_size, gemm_module.smem_size, hip_stream>>>(
			dynamic_launch_buffer_ptr,
			m, n, k,
			alpha,
			a_ptr, lda,
			b_ptr, ldb,
			beta,
			c_ptr, ldc
			);
#ifdef CUMPSGEMM_CHECK_KERNEL_ERROR
	CUTF_CHECK_ERROR(hipStreamSynchronize(hip_stream));
#endif
}

template <class T>
void launch_kernel (
			const cumpsgemm::gemm_module gemm_module,
			const int* const dynamic_launch_buffer_ptr,
			const std::size_t m,
			const std::size_t n,
			const std::size_t k,
			const T alpha,
			const T* const a_ptr, const std::size_t lda, const uint64_t stridea,
			const T* const b_ptr, const std::size_t ldb, const uint64_t strideb,
			const T beta,
			T* const c_ptr, const std::size_t ldc, const uint64_t stridec,
			const uint64_t batch_count,
			hipStream_t hip_stream
		) {
	const auto kernel_ptr = reinterpret_cast<cumpsgemm::gemm_stridedBatch_kernel_func_t<T>>(gemm_module.kernel_func);
	const dim3 block_size(gemm_module.block_size);
	const auto num_blocks_per_gemm = (m + gemm_module.smem_m - 1) / gemm_module.smem_m * (n + gemm_module.smem_n - 1) / gemm_module.smem_n;
	const dim3 grid_size(
			num_blocks_per_gemm * batch_count
			);

	kernel_ptr<<<grid_size, block_size, gemm_module.smem_size, hip_stream>>>(
			dynamic_launch_buffer_ptr,
			m, n, k,
			alpha,
			a_ptr, lda, stridea,
			b_ptr, ldb, strideb,
			beta,
			c_ptr, ldc, stridec,
			num_blocks_per_gemm
			);
#ifdef CUMPSGEMM_CHECK_KERNEL_ERROR
	CUTF_CHECK_ERROR(hipStreamSynchronize(hip_stream));
#endif
}
} // unnamed namespace

template <class T>
hipblasStatus_t cumpsgemm::gemm(
		cuMpSGEMM_handle_t handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const T* alpha,
		const T* const a_dmem_ptr, const uint64_t lda,
		const T* const b_dmem_ptr, const uint64_t ldb,
		const T* beta,
		T* const c_dmem_ptr, const uint64_t ldc,
		const cuMpSGEMM_compute_mode_t compute_mode,
		unsigned* const used_kernel_modeule_id
		) {

	if (compute_mode != CUMPSGEMM_AUTO) {
		const auto code = gen_module_code<T>(op_A, op_B, compute_mode);

		const auto kernel_module_candidate_list = handle->gemm_module[code];

		unsigned module_id;
		auto gemm_module = kernel_module_candidate_list[handle->num_kernel_candidates - 1];
		for (module_id = 0; module_id < handle->num_kernel_candidates - 1; module_id++) {
			const auto module = kernel_module_candidate_list[module_id];
			if (m * n / (module.smem_m * module.smem_n) > handle->num_sms * 2 /*A magic number :) */) {
				gemm_module = module;
				break;
			}
		}

		if (used_kernel_modeule_id != nullptr) {
			*used_kernel_modeule_id = module_id;
		}

		launch_kernel<T>(
				gemm_module,
				nullptr,
				m, n, k,
				*alpha,
				a_dmem_ptr, lda,
				b_dmem_ptr, ldb,
				*beta,
				c_dmem_ptr, ldc,
				handle->hip_stream
				);
	} else {
		const auto code_A = gen_module_code<T>(op_A, op_B, handle->dynamic_launch_handle->mode_A);
		const auto code_B = gen_module_code<T>(op_A, op_B, handle->dynamic_launch_handle->mode_B);

		const auto kernel_module_candidate_list_A = handle->gemm_module[code_A];
		const auto kernel_module_candidate_list_B = handle->gemm_module[code_B];

		unsigned module_id;
		auto gemm_module_A = kernel_module_candidate_list_A[handle->num_kernel_candidates - 1];
		auto gemm_module_B = kernel_module_candidate_list_B[handle->num_kernel_candidates - 1];

		for (module_id = 0; module_id < handle->num_kernel_candidates - 1; module_id++) {
			const auto module = kernel_module_candidate_list_A[module_id];
			if (m * n / (module.smem_m * module.smem_n) > handle->num_sms * 2 /*A magic number :) */) {
				gemm_module_A = module;
				break;
			}
		}

		for (module_id = 0; module_id < handle->num_kernel_candidates - 1; module_id++) {
			const auto module = kernel_module_candidate_list_B[module_id];
			if (m * n / (module.smem_m * module.smem_n) > handle->num_sms * 2 /*A magic number :) */) {
				gemm_module_B = module;
				break;
			}
		}

		if (used_kernel_modeule_id != nullptr) {
			*used_kernel_modeule_id = ~0u;
		}

		launch_kernel<T>(
				gemm_module_A,
				handle->dynamic_launch_handle->frag_buffer + handle->dynamic_launch_handle->enabled_id,
				m, n, k,
				*alpha,
				a_dmem_ptr, lda,
				b_dmem_ptr, ldb,
				*beta,
				c_dmem_ptr, ldc,
				handle->hip_stream
				);
		launch_kernel<T>(
				gemm_module_B,
				handle->dynamic_launch_handle->frag_buffer + handle->dynamic_launch_handle->enabled_id,
				m, n, k,
				*alpha,
				a_dmem_ptr, lda,
				b_dmem_ptr, ldb,
				*beta,
				c_dmem_ptr, ldc,
				handle->hip_stream
				);

	}
	if (handle->exp_stats_handle->enabled) {
		cumpsgemm::exp_stats::exp_stats_ext(
				handle,
				m, n,
				c_dmem_ptr, ldc,
				1,
				0	
				);
		}

	return HIPBLAS_STATUS_SUCCESS;
}


template <class T>
hipblasStatus_t cumpsgemm::gemm_stridedBatch(
		cuMpSGEMM_handle_t handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const T* alpha,
		const T* const a_dmem_ptr, const uint64_t lda, const uint64_t stridea,
		const T* const b_dmem_ptr, const uint64_t ldb, const uint64_t strideb,
		const T* beta,
		T* const c_dmem_ptr, const uint64_t ldc, const uint64_t stridec,
		const uint64_t batch_count,
		const cuMpSGEMM_compute_mode_t compute_mode,
		unsigned* const used_kernel_modeule_id
		) {
	if (m * n > (1lu << 24)) {
		const auto orig_exp_stats_enabled = handle->exp_stats_handle->enabled;
		handle->exp_stats_handle->enabled = 0;
		for (std::uint64_t i = 0; i < batch_count; i++) {
			cumpsgemm::gemm(
					handle,
					op_A, op_B,
					m, n, k,
					alpha,
					a_dmem_ptr + i * stridea, lda,
					b_dmem_ptr + i * strideb, ldb,
					beta,
					c_dmem_ptr + i * stridec, ldc,
					compute_mode,
					used_kernel_modeule_id
					);
		}
		handle->exp_stats_handle->enabled = orig_exp_stats_enabled;
		if (handle->exp_stats_handle->enabled) {
			cumpsgemm::exp_stats::exp_stats_ext(
					handle,
					m, n,
					c_dmem_ptr, ldc,
					batch_count,
					stridec
					);
		}
		return HIPBLAS_STATUS_SUCCESS;
	}

	if (compute_mode != CUMPSGEMM_AUTO) {
		const auto code = gen_module_code<T>(op_A, op_B, compute_mode);

		const auto kernel_module_candidate_list = handle->gemm_stridedBatch_module[code];

		unsigned module_id;
		auto gemm_module = kernel_module_candidate_list[handle->num_kernel_candidates - 1];
		for (module_id = 0; module_id < handle->num_kernel_candidates - 1; module_id++) {
			const auto module = kernel_module_candidate_list[module_id];
			if (m * n / (module.smem_m * module.smem_n) * batch_count > handle->num_sms * 2 /*A magic number :) */) {
				gemm_module = module;
				break;
			}
		}

		if (used_kernel_modeule_id != nullptr) {
			*used_kernel_modeule_id = module_id;
		}

		launch_kernel<T>(
				gemm_module,
				nullptr,
				m, n, k,
				*alpha,
				a_dmem_ptr, lda, stridea,
				b_dmem_ptr, ldb, strideb,
				*beta,
				c_dmem_ptr, ldc, stridec,
				batch_count,
				handle->hip_stream
				);
	} else {
		const auto code_A = gen_module_code<T>(op_A, op_B, handle->dynamic_launch_handle->mode_A);
		const auto code_B = gen_module_code<T>(op_A, op_B, handle->dynamic_launch_handle->mode_B);

		const auto kernel_module_candidate_list_A = handle->gemm_stridedBatch_module[code_A];
		const auto kernel_module_candidate_list_B = handle->gemm_stridedBatch_module[code_B];

		unsigned module_id;
		auto gemm_module_A = kernel_module_candidate_list_A[handle->num_kernel_candidates - 1];
		auto gemm_module_B = kernel_module_candidate_list_B[handle->num_kernel_candidates - 1];

		for (module_id = 0; module_id < handle->num_kernel_candidates - 1; module_id++) {
			const auto module = kernel_module_candidate_list_A[module_id];
			if (m * n / (module.smem_m * module.smem_n) * batch_count > handle->num_sms * 2 /*A magic number :) */) {
				gemm_module_A = module;
				break;
			}
		}

		for (module_id = 0; module_id < handle->num_kernel_candidates - 1; module_id++) {
			const auto module = kernel_module_candidate_list_B[module_id];
			if (m * n / (module.smem_m * module.smem_n) * batch_count > handle->num_sms * 2 /*A magic number :) */) {
				gemm_module_B = module;
				break;
			}
		}

		if (used_kernel_modeule_id != nullptr) {
			*used_kernel_modeule_id = ~0u;
		}

		launch_kernel<T>(
				gemm_module_A,
				handle->dynamic_launch_handle->frag_buffer + handle->dynamic_launch_handle->enabled_id,
				m, n, k,
				*alpha,
				a_dmem_ptr, lda, stridea,
				b_dmem_ptr, ldb, strideb,
				*beta,
				c_dmem_ptr, ldc, stridec,
				batch_count,
				handle->hip_stream
				);
		launch_kernel<T>(
				gemm_module_B,
				handle->dynamic_launch_handle->frag_buffer + handle->dynamic_launch_handle->enabled_id,
				m, n, k,
				*alpha,
				a_dmem_ptr, lda, stridea,
				b_dmem_ptr, ldb, strideb,
				*beta,
				c_dmem_ptr, ldc, stridec,
				batch_count,
				handle->hip_stream
				);
	}

	if (handle->exp_stats_handle->enabled) {
		cumpsgemm::exp_stats::exp_stats_ext(
				handle,
				m, n,
				c_dmem_ptr, ldc,
				batch_count,
				stridec
				);
	}

	return HIPBLAS_STATUS_SUCCESS;
}

extern "C" {
hipblasStatus_t cuMpSGEMM_sgemm(
		cuMpSGEMM_handle_t handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const float* alpha,
		const float* const a_dmem_ptr, const uint64_t lda,
		const float* const b_dmem_ptr, const uint64_t ldb,
		const float* beta,
		float* const c_dmem_ptr, const uint64_t ldc,
		const cuMpSGEMM_compute_mode_t compute_mode
		) {
	assert(op_A != HIPBLAS_OP_C);
	assert(op_B != HIPBLAS_OP_C);
	return cumpsgemm::gemm<float>(
			handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda,
			b_dmem_ptr, ldb,
			beta,
			c_dmem_ptr, ldc,
			compute_mode
			);
}

hipblasStatus_t cuMpSGEMM_cgemm(
		cuMpSGEMM_handle_t handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const hipComplex* alpha,
		const hipComplex* const a_dmem_ptr, const uint64_t lda,
		const hipComplex* const b_dmem_ptr, const uint64_t ldb,
		const hipComplex* beta,
		hipComplex* const c_dmem_ptr, const uint64_t ldc,
		const cuMpSGEMM_compute_mode_t compute_mode
		) {
	return cumpsgemm::gemm<hipComplex>(
			handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda,
			b_dmem_ptr, ldb,
			beta,
			c_dmem_ptr, ldc,
			compute_mode
			);
}

hipblasStatus_t cuMpSGEMM_sgemm_strided_batch(
		cuMpSGEMM_handle_t handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const float* alpha,
		const float* const a_dmem_ptr, const uint64_t lda, const uint64_t stridea,
		const float* const b_dmem_ptr, const uint64_t ldb, const uint64_t strideb,
		const float* beta,
		float* const c_dmem_ptr, const uint64_t ldc, const uint64_t stridec,
		const uint64_t batch_count,
		const cuMpSGEMM_compute_mode_t compute_mode
		) {
	assert(op_A != HIPBLAS_OP_C);
	assert(op_B != HIPBLAS_OP_C);
	return cumpsgemm::gemm_stridedBatch<float>(
			handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda, stridea,
			b_dmem_ptr, ldb, strideb,
			beta,
			c_dmem_ptr, ldc, stridec,
			batch_count,
			compute_mode
			);
}

hipblasStatus_t cuMpSGEMM_cgemm_strided_batch(
		cuMpSGEMM_handle_t handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const hipComplex* alpha,
		const hipComplex* const a_dmem_ptr, const uint64_t lda, const uint64_t stridea,
		const hipComplex* const b_dmem_ptr, const uint64_t ldb, const uint64_t strideb,
		const hipComplex* beta,
		hipComplex* const c_dmem_ptr, const uint64_t ldc, const uint64_t stridec,
		const uint64_t batch_count,
		const cuMpSGEMM_compute_mode_t compute_mode
		) {
	return cumpsgemm::gemm_stridedBatch<hipComplex>(
			handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda, stridea,
			b_dmem_ptr, ldb, strideb,
			beta,
			c_dmem_ptr, ldc, stridec,
			batch_count,
			compute_mode
			);
}
} // extern "C"


std::pair<std::size_t, std::size_t> cumpsgemm::get_exp_stats(
		cuMpSGEMM_handle_t handle,
		const unsigned buffer_id
		) {
	return cumpsgemm::exp_stats::get_exp_stats(handle, buffer_id);
}

unsigned cumpsgemm::get_current_buffer_id(
		cuMpSGEMM_handle_t handle
		) {
	return cumpsgemm::exp_stats::get_current_buffer_id(handle);
}

void cumpsgemm::reset_buffer_id(
		cuMpSGEMM_handle_t handle
		) {
	cumpsgemm::exp_stats::reset_buffer_id(handle);
}
