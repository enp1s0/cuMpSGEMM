#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <chrono>
#include <regex>
#include <cutf/hiprand.hpp>
#include <cutf/memory.hpp>
#include <cutf/cublas.hpp>
#include <cumpsgemm/cumpsgemm.hpp>

double error_threshold(
		const cuMpSGEMM_compute_mode_t compute_mode,
		const std::size_t N
		) {
	if (compute_mode == CUMPSGEMM_FP16TC ||
			compute_mode == CUMPSGEMM_TF32TC) {
		return 1. / (1 << 10) * std::sqrt(N);
	}
	return 1. / (1 << 23) * std::sqrt(N);
}

__device__ double mad(
		const float a,
		const float b,
		const double c
		) {
	return static_cast<double>(a) * static_cast<double>(b) + c;
}

__device__ double2 mad(
		const float2 a,
		const float2 b,
		const double2 c
		) {
	const auto dbl_a = hipComplexFloatToDouble(a);
	const auto dbl_b = hipComplexFloatToDouble(b);
	return hipCadd(hipCmul(dbl_a, dbl_b), c);
}

template <class T>
struct doubled_t {using type = double;};
template <> struct doubled_t<hipComplex> {using type = hipDoubleComplex;};

template <class T>
__device__ T load_with_op(
		const T* const ptr,
		hipblasOperation_t op
		) {
	return *ptr;
}

template <>
__device__ hipComplex load_with_op<hipComplex>(
		const hipComplex* const ptr,
		hipblasOperation_t op
		) {
	if (op == HIPBLAS_OP_C) {
		const auto v = *ptr;
		return hipConjf(v);
	}
	return *ptr;
}

__device__ double diff2(
		const hipDoubleComplex ab,
		const hipComplex c
		) {
	const auto real_diff = ab.x - c.x;
	const auto imag_diff = ab.y - c.y;
	return real_diff * real_diff + imag_diff * imag_diff;
}
__device__ double diff2(
		const double ab,
		const float c
		) {
	const auto diff = ab - c;
	return diff * diff;
}
__device__ double norm2(
		const hipDoubleComplex a
		) {
	return a.x * a.x + a.y * a.y;
}
__device__ double norm2(
		const double a
		) {
	return a * a;
}


template <class T>
__host__ __device__ T one() {return 1;}
template <> __host__ __device__ hipComplex one() {return make_hipComplex(1, 0);}
template <class T>
__host__ __device__ T zero() {return 0;}
template <> __host__ __device__ hipComplex zero() {return make_hipComplex(0, 0);}
template <> __host__ __device__ hipDoubleComplex zero() {return make_hipDoubleComplex(0, 0);}


template <class T>
__global__ void calc_matmul_residual_kernel(
		double* const base_norm2_ptr,
		double* const diff_norm2_ptr,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const unsigned m,
		const unsigned n,
		const unsigned k,
		const T* const a_ptr, const unsigned lda,
		const T* const b_ptr, const unsigned ldb,
		const T* const c_ptr, const unsigned ldc
		) {
	const auto tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid >= m * n) return;

	const auto c_m = tid % m;
	const auto c_n = tid / m;

	auto c = zero<typename doubled_t<T>::type>();
	for (std::size_t ik = 0; ik < k; ik++) {
		std::size_t a_index = 0;
		if (op_A == HIPBLAS_OP_N) {
			a_index = c_m + ik * lda;
		} else {
			a_index = ik + c_m * lda;
		}

		std::size_t b_index = 0;
		if (op_B == HIPBLAS_OP_N) {
			b_index = ik + c_n * ldb;
		} else {
			b_index = c_n + ik * ldb;
		}

		c = mad(
				load_with_op(a_ptr + a_index, op_A),
				load_with_op(b_ptr + b_index, op_B),
				c
				);
		const auto aa = load_with_op(a_ptr + a_index, op_A);
		const auto bb = load_with_op(b_ptr + b_index, op_B);
	}
	const auto base_norm2 = norm2(c);
	const auto diff_norm2 = diff2(c, c_ptr[c_m + c_n * ldc]);

	atomicAdd(base_norm2_ptr, base_norm2);
	atomicAdd(diff_norm2_ptr, diff_norm2);
}

template <class T>
double calc_matmul_residual(
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const unsigned m,
		const unsigned n,
		const unsigned k,
		const T* const a_ptr, const unsigned lda,
		const T* const b_ptr, const unsigned ldb,
		const T* const c_ptr, const unsigned ldc
		) {
	auto base_norm2_ptr = cutf::memory::malloc_managed<double>(1);
	auto diff_norm2_ptr = cutf::memory::malloc_managed<double>(1);

	*base_norm2_ptr = 0;
	*diff_norm2_ptr = 0;

	constexpr unsigned block_size = 256;
	const auto num_threads = m * n;
	const auto grid_size = (num_threads + block_size - 1) / block_size;

	hipDeviceSynchronize();
	calc_matmul_residual_kernel<<<grid_size, block_size>>>(
			base_norm2_ptr, diff_norm2_ptr,
			op_A, op_B,
			m, n, k,
			a_ptr, lda,
			b_ptr, ldb,
			c_ptr, ldc
			);
	hipDeviceSynchronize();

	const auto residual = std::sqrt(*diff_norm2_ptr / *base_norm2_ptr);

	cutf::memory::free(base_norm2_ptr);
	cutf::memory::free(diff_norm2_ptr);

	return residual;
}

void cublas_gemm(
		hipblasHandle_t const cublas_handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const unsigned m,
		const unsigned n,
		const unsigned k,
		const float* const alpha,
		const float* const a_ptr, const unsigned lda,
		const float* const b_ptr, const unsigned ldb,
		const float* const beta,
		float* const c_ptr, const unsigned ldc
		) {
		CUTF_CHECK_ERROR(hipblasSgemm(
					cublas_handle,
					op_A, op_B,
					m, n, k,
					alpha,
					a_ptr, lda,
					b_ptr, ldb,
					beta,
					c_ptr, ldc
					));
}

void cublas_gemm(
		hipblasHandle_t const cublas_handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const unsigned m,
		const unsigned n,
		const unsigned k,
		const hipComplex* const alpha,
		const hipComplex* const a_ptr, const unsigned lda,
		const hipComplex* const b_ptr, const unsigned ldb,
		const hipComplex* const beta,
		hipComplex* const c_ptr, const unsigned ldc
		) {
		CUTF_CHECK_ERROR(hipblasCgemm(
					cublas_handle,
					op_A, op_B,
					m, n, k,
					alpha,
					a_ptr, lda,
					b_ptr, ldb,
					beta,
					c_ptr, ldc
					));
}

void cublas_gemm_strided_batch(
		hipblasHandle_t const cublas_handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const unsigned m,
		const unsigned n,
		const unsigned k,
		const float* const alpha,
		const float* const a_ptr, const unsigned lda, const long long int stride_a,
		const float* const b_ptr, const unsigned ldb, const long long int stride_b,
		const float* const beta,
		float* const c_ptr, const unsigned ldc, const long long int stride_c,
		const long long int batch_count
		) {
		CUTF_CHECK_ERROR(hipblasSgemmStridedBatched(
					cublas_handle,
					op_A, op_B,
					m, n, k,
					alpha,
					a_ptr, lda, stride_a,
					b_ptr, ldb, stride_b,
					beta,
					c_ptr, ldc, stride_c,
					batch_count
					));
}

void cublas_gemm_strided_batch(
		hipblasHandle_t const cublas_handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const unsigned m,
		const unsigned n,
		const unsigned k,
		const hipComplex* const alpha,
		const hipComplex* const a_ptr, const unsigned lda, const long long int stride_a,
		const hipComplex* const b_ptr, const unsigned ldb, const long long int stride_b,
		const hipComplex* const beta,
		hipComplex* const c_ptr, const unsigned ldc, const long long int stride_c,
		const long long int batch_count
		) {
		CUTF_CHECK_ERROR(hipblasCgemmStridedBatched(
					cublas_handle,
					op_A, op_B,
					m, n, k,
					alpha,
					a_ptr, lda, stride_a,
					b_ptr, ldb, stride_b,
					beta,
					c_ptr, ldc, stride_c,
					batch_count
					));
}

template <class T>
int sgemm_test_core(
		hipblasHandle_t const cublas_handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const unsigned m,
		const unsigned n,
		const unsigned k,
		T* const a_ptr, const unsigned lda,
		T* const b_ptr, const unsigned ldb,
		T* const c_ptr, const unsigned ldc,
		const cuMpSGEMM_compute_mode_t compute_mode
		) {
	const auto alpha = one<T>(), beta = zero<T>();

	auto gemm_func = [&]() {
		if (compute_mode == CUMPSGEMM_CUBLAS) {
			cublas_gemm(
					cublas_handle,
					op_A, op_B,
					m, n, k,
					&alpha,
					a_ptr, lda,
					b_ptr, ldb,
					&beta,
					c_ptr, ldc
					);
		} else {
			cumpsgemm::gemm(
					op_A, op_B,
					m, n, k,
					&alpha,
					a_ptr, lda,
					b_ptr, ldb,
					&beta,
					c_ptr, ldc,
					compute_mode
					);
		}
	};

	gemm_func();

	CUTF_CHECK_ERROR(hipDeviceSynchronize());

	const auto residual = calc_matmul_residual(
					op_A, op_B,
					m, n, k,
					a_ptr, lda,
					b_ptr, ldb,
					c_ptr, ldc
			);
	const auto check = residual < error_threshold(compute_mode, m);

	// Throughput
	constexpr unsigned test_count = 16;
	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto start_clock = std::chrono::system_clock::now();
	for (unsigned i = 0; i < test_count; i++) {
		gemm_func();
	}
	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto end_clock = std::chrono::system_clock::now();
	const auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count() * 1e-6;
	const auto throughput = 2lu * m * n * k * (std::is_same<float, T>::value ? 1 : 4) / (elapsed_time / test_count);

	std::printf("%s,%s,%s,%s,%u,%u,%u,%e,%e,%s\n",
			(std::is_same<float, T>::value ? "sgemm" : "cgemm"),
			cuMpSGEMM_get_compute_mode_string(compute_mode),
			(op_A == HIPBLAS_OP_N) ? "N" : ((op_A == HIPBLAS_OP_T) ? "T" : "C"),
			(op_B == HIPBLAS_OP_N) ? "N" : ((op_B == HIPBLAS_OP_T) ? "T" : "C"),
			m, n, k,
			throughput * 1e-12,
			residual,
			(check ? "OK" : "NG")
			);
	std::fflush(stdout);

	if (check) {
		return 0;
	} else {
		return 1;
	}
}

template <class T>
int sgemm_strided_batch_test_core(
		hipblasHandle_t const cublas_handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const unsigned m,
		const unsigned n,
		const unsigned k,
		T* const a_ptr, const unsigned lda, const long long int stride_a,
		T* const b_ptr, const unsigned ldb, const long long int stride_b,
		T* const c_ptr, const unsigned ldc, const long long int stride_c,
		const long long int batch_count,
		const cuMpSGEMM_compute_mode_t compute_mode
		) {
	const auto alpha = one<T>(), beta = zero<T>();

	auto gemm_func = [&]() {
		if (compute_mode == CUMPSGEMM_CUBLAS) {
			cublas_gemm_strided_batch(
					cublas_handle,
					op_A, op_B,
					m, n, k,
					&alpha,
					a_ptr, lda, stride_a,
					b_ptr, ldb, stride_b,
					&beta,
					c_ptr, ldc, stride_c,
					batch_count
					);
		} else {
			cumpsgemm::gemm_stridedBatch(
					op_A, op_B,
					m, n, k,
					&alpha,
					a_ptr, lda, stride_a,
					b_ptr, ldb, stride_b,
					&beta,
					c_ptr, ldc, stride_c,
					batch_count,
					compute_mode
					);
		}
	};

	gemm_func();

	CUTF_CHECK_ERROR(hipDeviceSynchronize());

	double residual = 0;
	for (unsigned long long int b = 0; b < batch_count; b++) {
	 	residual += calc_matmul_residual(
					op_A, op_B,
					m, n, k,
					a_ptr + stride_a, lda,
					b_ptr + stride_b, ldb,
					c_ptr + stride_c, ldc
			);
	}
	residual /= batch_count;
	const auto check = residual < error_threshold(compute_mode, m);

	// Throughput
	constexpr unsigned test_count = 16;
	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto start_clock = std::chrono::system_clock::now();
	for (unsigned i = 0; i < test_count; i++) {
		gemm_func();
	}
	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto end_clock = std::chrono::system_clock::now();
	const auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count() * 1e-6;
	const auto throughput = 2lu * m * n * k * batch_count * (std::is_same<float, T>::value ? 1 : 4) / (elapsed_time / test_count);

	std::printf("%s,%s,%s,%s,%u,%u,%u,%lld,%e,%e,%s\n",
			(std::is_same<float, T>::value ? "sgemm" : "cgemm"),
			cuMpSGEMM_get_compute_mode_string(compute_mode),
			(op_A == HIPBLAS_OP_N) ? "N" : ((op_A == HIPBLAS_OP_T) ? "T" : "C"),
			(op_B == HIPBLAS_OP_N) ? "N" : ((op_B == HIPBLAS_OP_T) ? "T" : "C"),
			m, n, k,
			batch_count,
			throughput * 1e-12,
			residual,
			(check ? "OK" : "NG")
			);
	std::fflush(stdout);

	if (check) {
		return 0;
	} else {
		return 1;
	}
}

void sgemm_test(const std::size_t min_N, const std::size_t max_N, const std::size_t interval, const bool only_cublas) {
	constexpr uint64_t seed = 0;
	const std::size_t max_num_elements = max_N * max_N * 2;
	float* a_ptr = cutf::memory::malloc<float>(max_num_elements);
	float* b_ptr = cutf::memory::malloc<float>(max_num_elements);
	float* c_ptr = cutf::memory::malloc<float>(max_num_elements);

	auto curand_gen = cutf::hiprand::get_curand_unique_ptr(HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	CUTF_CHECK_ERROR(hiprandSetPseudoRandomGeneratorSeed(*curand_gen.get(), seed));
	CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*curand_gen.get(), a_ptr, max_num_elements));
	CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*curand_gen.get(), b_ptr, max_num_elements));

	std::vector<cuMpSGEMM_compute_mode_t> modes = {
		CUMPSGEMM_CUBLAS,
	};

	if (!only_cublas) {
		modes.push_back(CUMPSGEMM_FP16TCEC);
		modes.push_back(CUMPSGEMM_FP16TC);
		modes.push_back(CUMPSGEMM_TF32TCEC);
		modes.push_back(CUMPSGEMM_TF32TC);
	}

	std::vector<hipblasOperation_t> sgemm_ops = {
		HIPBLAS_OP_N,
		HIPBLAS_OP_T
	};
	std::vector<hipblasOperation_t> cgemm_ops = {
		HIPBLAS_OP_N,
		HIPBLAS_OP_T,
		HIPBLAS_OP_C
	};

	std::printf("## %s\n", __func__);
	std::printf("type,mode,op_A,op_B,m,n,k,throughput_in_tflops,residual,check\n");
	unsigned num_tests = 0;
	unsigned num_passed = 0;
	auto cublas_handle_uptr = cutf::cublas::get_cublas_unique_ptr();
	for (const auto mode : modes) {
		for (const auto op_A : sgemm_ops) {
			for (const auto op_B : sgemm_ops) {
				for (unsigned N = min_N; N <= max_N; N += interval) {
					const auto res = sgemm_test_core(
							*cublas_handle_uptr.get(),
							op_A,
							op_B,
							N, N, N,
							a_ptr, N,
							b_ptr, N,
							c_ptr, N,
							mode
							);
					num_tests++;
					if (res == 0) {
						num_passed++;
					}
				}
			}
		}
	}
	for (const auto mode : modes) {
		for (const auto op_A : cgemm_ops) {
			for (const auto op_B : cgemm_ops) {
				for (unsigned N = min_N; N <= max_N; N += interval) {
					const auto res = sgemm_test_core(
							*cublas_handle_uptr.get(),
							op_A,
							op_B,
							N, N, N,
							reinterpret_cast<hipComplex*>(a_ptr), N,
							reinterpret_cast<hipComplex*>(b_ptr), N,
							reinterpret_cast<hipComplex*>(c_ptr), N,
							mode
							);
					num_tests++;
					if (res == 0) {
						num_passed++;
					}
				}
			}
		}
	}
	CUTF_CHECK_ERROR(hipDeviceSynchronize());

	std::printf("Result : %u / %u passed\n",
			num_passed,
			num_tests
			);

	cutf::memory::free(a_ptr);
	cutf::memory::free(b_ptr);
	cutf::memory::free(c_ptr);
}

void sgemm_strided_batch_test(const std::size_t min_N, const std::size_t max_N, const std::size_t interval, const std::size_t batch_count, const bool only_cublas) {
	constexpr uint64_t seed = 0;
	const std::size_t max_num_elements = max_N * max_N * batch_count * 2;
	float* a_ptr = cutf::memory::malloc<float>(max_num_elements);
	float* b_ptr = cutf::memory::malloc<float>(max_num_elements);
	float* c_ptr = cutf::memory::malloc<float>(max_num_elements);

	auto curand_gen = cutf::hiprand::get_curand_unique_ptr(HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	CUTF_CHECK_ERROR(hiprandSetPseudoRandomGeneratorSeed(*curand_gen.get(), seed));
	CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*curand_gen.get(), a_ptr, max_num_elements));
	CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*curand_gen.get(), b_ptr, max_num_elements));


	std::vector<cuMpSGEMM_compute_mode_t> modes = {
		CUMPSGEMM_CUBLAS,
	};

	if (!only_cublas) {
		modes.push_back(CUMPSGEMM_FP16TCEC);
		modes.push_back(CUMPSGEMM_FP16TC);
		modes.push_back(CUMPSGEMM_TF32TCEC);
		modes.push_back(CUMPSGEMM_TF32TC);
	}

	std::vector<hipblasOperation_t> sgemm_ops = {
		HIPBLAS_OP_N,
		HIPBLAS_OP_T
	};
	std::vector<hipblasOperation_t> cgemm_ops = {
		HIPBLAS_OP_N,
		HIPBLAS_OP_T,
		HIPBLAS_OP_C
	};

	std::printf("## %s\n", __func__);
	std::printf("type,mode,op_A,op_B,m,n,k,batch_count,throughput_in_tflops,residual,check\n");
	unsigned num_tests = 0;
	unsigned num_passed = 0;
	auto cublas_handle_uptr = cutf::cublas::get_cublas_unique_ptr();
	for (const auto mode : modes) {
		for (const auto op_A : sgemm_ops) {
			for (const auto op_B : sgemm_ops) {
				for (unsigned N = min_N; N <= max_N; N += interval) {
					const auto res = sgemm_strided_batch_test_core(
							*cublas_handle_uptr.get(),
							op_A,
							op_B,
							N, N, N,
							a_ptr, N, max_N * max_N,
							b_ptr, N, max_N * max_N,
							c_ptr, N, max_N * max_N,
							batch_count,
							mode
							);
					num_tests++;
					if (res == 0) {
						num_passed++;
					}
				}
			}
		}
	}
	for (const auto mode : modes) {
		for (const auto op_A : cgemm_ops) {
			for (const auto op_B : cgemm_ops) {
				for (unsigned N = min_N; N <= max_N; N += interval) {
					const auto res = sgemm_strided_batch_test_core(
							*cublas_handle_uptr.get(),
							op_A,
							op_B,
							N, N, N,
							reinterpret_cast<hipComplex*>(a_ptr), N, max_N * max_N,
							reinterpret_cast<hipComplex*>(b_ptr), N, max_N * max_N,
							reinterpret_cast<hipComplex*>(c_ptr), N, max_N * max_N,
							batch_count,
							mode
							);
					num_tests++;
					if (res == 0) {
						num_passed++;
					}
				}
			}
		}
	}
	CUTF_CHECK_ERROR(hipDeviceSynchronize());

	std::printf("Result : %u / %u passed\n",
			num_passed,
			num_tests
			);

	cutf::memory::free(a_ptr);
	cutf::memory::free(b_ptr);
	cutf::memory::free(c_ptr);
}

// [cuMpSGEMM LOG] hipblasCgemm op=(N, T), shape=(4, 128, 65536), mode=TF32TCEC
void test_logged_shape(
		const std::string log_path
		) {
	std::ifstream ifs(log_path);
	if (!ifs) {
		throw std::runtime_error("No such file : " + log_path);
	}

	auto cublas_handle_uptr = cutf::cublas::get_cublas_unique_ptr();
	std::size_t num_passed = 0;
	std::size_t num_tested = 0;
	std::printf("## %s\n", __func__);
	const std::string log_prefix = "[cuMpSGEMM LOG] ";
	std::string buffer;
	while (std::getline(ifs, buffer)) {
		if (buffer.find(log_prefix) == std::string::npos) {
			continue;
		}
		buffer = buffer.substr(log_prefix.length());
		std::regex base_regex(R"((\w+) (.+), mode=(.+))");
		std::smatch base_match;

		std::string func = "";
		std::string params = "";
		std::string mode = "";
		if (std::regex_match(buffer, base_match, base_regex)) {
			func = base_match[1].str();
			params = base_match[2].str();
			mode = base_match[3].str();
		}

		if (func.length() * params.length() * mode.length() == 0) {
			continue;
		}

		cuMpSGEMM_compute_mode_t compute_mode = CUMPSGEMM_CUBLAS;
		if (mode == "FP16TC") {
			compute_mode = CUMPSGEMM_FP16TC;
		} else if (mode == "FP16TCEC") {
			compute_mode = CUMPSGEMM_FP16TCEC;
		} else if (mode == "TF32TC") {
			compute_mode = CUMPSGEMM_TF32TC;
		} else if (mode == "TF32TCEC") {
			compute_mode = CUMPSGEMM_TF32TCEC;
		} else {
			throw std::runtime_error("Unknown compute mode : " + mode);
		}

		if (func == "hipblasCgemm" || func == "hipblasSgemm") {
			std::regex param_regex(R"(op=\((.), (.)\), shape=\((\d+), (\d+), (\d+)\))");
			std::smatch param_match;

			std::size_t m = 0, n = 0, k = 0;
			hipblasOperation_t op_A, op_B;
			if (std::regex_match(params, param_match, param_regex) && param_match.size() > 1) {
				op_A = param_match[1].str() == "N" ? HIPBLAS_OP_N : (param_match[1].str() == "T" ? HIPBLAS_OP_T : HIPBLAS_OP_C);
				op_B = param_match[2].str() == "N" ? HIPBLAS_OP_N : (param_match[2].str() == "T" ? HIPBLAS_OP_T : HIPBLAS_OP_C);
				m = std::stoul(param_match[3].str());
				n = std::stoul(param_match[4].str());
				k = std::stoul(param_match[5].str());
			} else {
				throw std::runtime_error("Failed to parse parameters : " + params);
			}

			if (m * n * k == 0) {
				throw std::runtime_error("Invalid shape : (" + std::to_string(m) + ", " + std::to_string(n) + ", " + std::to_string(k) + ")");
			}
			constexpr uint64_t seed = 0;

			const std::size_t num_e = (func == "hipblasSgemm" ? 1 : 2);
			float* a_ptr = cutf::memory::malloc<float>(m * k * num_e);
			float* b_ptr = cutf::memory::malloc<float>(k * n * num_e);
			float* c_ptr = cutf::memory::malloc<float>(m * n * num_e);

			auto curand_gen = cutf::hiprand::get_curand_unique_ptr(HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
			CUTF_CHECK_ERROR(hiprandSetPseudoRandomGeneratorSeed(*curand_gen.get(), seed));
			CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*curand_gen.get(), a_ptr, m * k * num_e));
			CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*curand_gen.get(), b_ptr, k * n * num_e));
			int res;
			if (func == "hipblasSgemm") {
				res = sgemm_test_core(
						*cublas_handle_uptr.get(),
						op_A,
						op_B,
						m, n, k,
						a_ptr, (op_A == HIPBLAS_OP_N ? m : k),
						b_ptr, (op_B == HIPBLAS_OP_N ? k : n),
						c_ptr, m,
						compute_mode
						);
			} else {
				res = sgemm_test_core(
						*cublas_handle_uptr.get(),
						op_A,
						op_B,
						m, n, k,
						reinterpret_cast<hipComplex*>(a_ptr), (op_A == HIPBLAS_OP_N ? m : k),
						reinterpret_cast<hipComplex*>(b_ptr), (op_B == HIPBLAS_OP_N ? k : n),
						reinterpret_cast<hipComplex*>(c_ptr), m,
						compute_mode
						);
			}
			if (res == 0) {
				num_passed++;
			}
			num_tested++;

			cutf::memory::free(a_ptr);
			cutf::memory::free(b_ptr);
			cutf::memory::free(c_ptr);
		}
	}
	ifs.close();
	std::printf("%lu / %lu passed\n", num_passed, num_tested);
}

void print_usage(const char* program_name) {
	std::fprintf(stderr,
			"Usage : %s gemm [min_N] [max_N] [interval]\n"
			"      : %s gemm_strided_batch [min_N] [max_N] [interval] [batch_count]\n"
			"      : %s cublas_gemm [min_N] [max_N] [interval]\n"
			"      : %s cublas_gemm_strided_batch [min_N] [max_N] [interval] [batch_count]\n"
			"      : %s log [/path/to/log]\n",
			program_name, program_name, program_name, program_name, program_name
			);
	std::fflush(stderr);
}

int main(int argc, char** argv) {
	if (argc < 2) {
		print_usage(argv[0]);
		return 1;
	}

	const std::string command = argv[1];

	if (command == "gemm") {
		if (argc < 1 + 1 + 3) {
			print_usage(argv[0]);
			return 1;
		}
		sgemm_test(std::stoi(argv[2]), std::stoi(argv[3]), std::stoi(argv[4]), false);
	} else if (command == "gemm_strided_batch") {
		if (argc < 1 + 1 + 3 + 1) {
			print_usage(argv[0]);
			return 1;
		}
		sgemm_strided_batch_test(std::stoi(argv[2]), std::stoi(argv[3]), std::stoi(argv[4]), std::stoi(argv[5]), false);
	} else if (command == "cublas_gemm") {
		if (argc < 1 + 1 + 3) {
			print_usage(argv[0]);
			return 1;
		}
		sgemm_test(std::stoi(argv[2]), std::stoi(argv[3]), std::stoi(argv[4]), true);
	} else if (command == "cublas_gemm_strided_batch") {
		if (argc < 1 + 1 + 3 + 1) {
			print_usage(argv[0]);
			return 1;
		}
		sgemm_strided_batch_test(std::stoi(argv[2]), std::stoi(argv[3]), std::stoi(argv[4]), std::stoi(argv[5]), true);
	} else if (command == "log") {
		if (argc < 1 + 1 + 1) {
			print_usage(argv[0]);
			return 1;
		}
		test_logged_shape(argv[2]);
	} else {
		print_usage(argv[0]);
		return 1;
	}
}
