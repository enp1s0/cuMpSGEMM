#include <cumpsgemm/cumpsgemm.h>
#include "handle.hpp"

extern "C" {
hipblasStatus_t cuMpSGEMM_create(cuMpSGEMM_handle_t* const handle) {
	if ((*handle = new cuMpSGEMM_handle) == nullptr) {
		return HIPBLAS_STATUS_INTERNAL_ERROR;
	}

	return HIPBLAS_STATUS_SUCCESS;
}

hipblasStatus_t cuMpSGEMM_destroy(cuMpSGEMM_handle_t handle) {
	delete handle;
	return HIPBLAS_STATUS_SUCCESS;
}
}
