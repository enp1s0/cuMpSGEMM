#include <string>
#include <sstream>
#include <iomanip>
#include <hipblas.h>
#include <unistd.h>
#include <dlfcn.h>
#include <cumpsgemm/cumpsgemm.hpp>
#include <cumpsgemm/hijack_control.hpp>
#include <cutf/memory.hpp>
#include <cugemm_Mx2x2.hpp>
#include "handle.hpp"
#include "exp_stats.hpp"
#include "dynamic_launch.hpp"
#include "dynamic_launch_utils.hpp"
#include "dynamic_scaling.hpp"
#include "culip.hpp"

namespace {
std::string get_XeY_format_string(const double a) {
	std::stringstream ss;

	ss << std::scientific << a;

	return ss.str();
}
cuMpSGEMM_handle_t internal_global_cuMpSGEMM_handle = nullptr;
std::string internal_global_last_called_function_str = "";
bool global_internal_gemm_Mx2x2_enabled = false;
bool restore_AB = true;
cumpsgemm::hijack_control::control_function_t internal_global_control_func;

enum hijack_control_t {
	static_mode,
	dynamic_mode
} hijack_mode = dynamic_mode;
cuMpSGEMM_compute_mode_t internal_global_compute_mode = CUMPSGEMM_CUBLAS;

template <class Func>
void cuMpSGEMM_run_if_env_defined(
		const std::string env_str,
		const Func func
		) {
	const auto env = getenv(env_str.c_str());
	if (env != nullptr && std::string(env) != "0") {
		func();
	}
}

const std::string info_env_name = "CUMPSGEMM_INFO";
void cuMpSGEMM_log(
		const std::string str
		) {
	cuMpSGEMM_run_if_env_defined(
			info_env_name,
			[&](){
				std::fprintf(stdout, "[cuMpSGEMM LOG] %s\n",
						str.c_str());
				std::fflush(stdout);
			});
}

const std::string error_env_name = "CUMPSGEMM_ERROR_LOG";
void cuMpSGEMM_error(
		const std::string str
		) {
	cuMpSGEMM_run_if_env_defined(
			error_env_name,
			[&](){
				std::fprintf(stdout, "[cuMpSGEMM ERROR] %s\n",
						str.c_str());
				std::fflush(stdout);
			});
}

void cuMpSGEMM_warning(
		const std::string str
		) {
	cuMpSGEMM_run_if_env_defined(
			error_env_name,
			[&](){
				std::fprintf(stdout, "[cuMpSGEMM WARNING] %s\n",
						str.c_str());
				std::fflush(stdout);
			});
}

void* cuMpSGEMM_get_function_pointer(const std::string library_name, const std::string function_name) {

	// Open the library
	const auto lib_ptr = dlopen(library_name.c_str(), RTLD_NOW);
	if (lib_ptr == nullptr) {
		cuMpSGEMM_warning("Failed to load " + library_name + ". Default rule will be used.");
		return nullptr;
	}

	// Get function pointer
	void* function_ptr = dlsym(lib_ptr, function_name.c_str());
	if (function_ptr == NULL) {
		cuMpSGEMM_warning("Failed to load a function " + function_name + " during selecting hijacking function. Default rule will be used.");
		return nullptr;
	}

	return function_ptr;
}

std::string get_cublas_op_str(const hipblasOperation_t op) {
	switch (op) {
	case HIPBLAS_OP_C:
		return "C";
	case HIPBLAS_OP_N:
		return "N";
	case HIPBLAS_OP_T:
		return "T";
	default:
		return "?";
	}
}

cuMpSGEMM_handle_t cuMpSGEMM_get_internal_global_handle() {
	if (internal_global_cuMpSGEMM_handle == nullptr) {
		cuMpSGEMM_log("Initialize cuMpSGEMM handle...");
		cuMpSGEMM_create(&internal_global_cuMpSGEMM_handle);


		const auto init_float_by_env = [&](const std::string env_str, const float default_value) {
			const auto env = getenv(env_str.c_str());
			if (env != nullptr) {
				return std::stof(env);
			}
			return default_value;
		};

		const auto init_int_by_env = [&](const std::string env_str, const int default_value) {
			const auto env = getenv(env_str.c_str());
			if (env != nullptr) {
				return std::stoi(env);
			}
			return default_value;
		};

		// AUTO mode configure
		const auto ignore_threshold         = init_float_by_env("CUMPSGEMM_AUTO_IGNORE_THRESHOLD"        , 0);
		const auto underflow_threshold      = init_float_by_env("CUMPSGEMM_AUTO_UNDERFLOW_THRESHOLD"     , 1.f / 32768);
		const auto underflow_tolerance_rate = init_float_by_env("CUMPSGEMM_AUTO_UNDERFLOW_TOLERANCE_RATE", 0);
		const auto restore_AB_scaling       = init_int_by_env  ("CUMPSGEMM_AUTO_RESTORE_AB_SCALING"      , 1);

		cuMpSGEMM_log("AUTO config: ignore_threshold="         + get_XeY_format_string(ignore_threshold)         + " @Init");
		cuMpSGEMM_log("AUTO config: underflow_threshold="      + get_XeY_format_string(underflow_threshold)      + " @Init");
		cuMpSGEMM_log("AUTO config: underflow_tolerance_rate=" + get_XeY_format_string(underflow_tolerance_rate) + " @Init");
		cuMpSGEMM_log("AUTO config: restore_AB_scaling="       + std::to_string(restore_AB_scaling)            + " @Init");

		cumpsgemm::set_exp_stats_params(cuMpSGEMM_get_internal_global_handle(), ignore_threshold, underflow_threshold, underflow_tolerance_rate);
		restore_AB = restore_AB_scaling;
	}

	return internal_global_cuMpSGEMM_handle;
}

const std::string gemm_Mx2x2_env_name = "CUMPSGEMM_CUSTOM_GEMM_MX2X2";
bool is_gemm_Mx2x2_enabled() {
	if (global_internal_gemm_Mx2x2_enabled) {
		return true;
	}

	const auto env = getenv(gemm_Mx2x2_env_name.c_str());
	if (env == nullptr || std::string(env) == "0") {
		return false;
	}

	return true;
}

const std::string rule_lib_name = "libcumpsgemm_rule.so";
const std::string cublas_lib_name = "libcublas.so";
} // noname namespace

extern "C" const char* cuMpSGEMM_get_compute_mode_string (
		const cuMpSGEMM_compute_mode_t mode
		) {
	switch (mode) {
	case CUMPSGEMM_CUBLAS:
		return "CUBLAS";
	case CUMPSGEMM_FP16TC:
		return "FP16TC";
	case CUMPSGEMM_FP16TCEC:
		return "FP16TCEC";
	case CUMPSGEMM_TF32TC:
		return "TF32TC";
	case CUMPSGEMM_TF32TCEC:
		return "TF32TCEC";
	case CUMPSGEMM_CUBLAS_SIMT:
		return "CUBLAS_SIMT";
	case CUMPSGEMM_CUBLAS_FP16TC:
		return "CUBLAS_FP16TC";
	case CUMPSGEMM_CUBLAS_TF32TC:
		return "CUBLAS_TF32TC";
	case CUMPSGEMM_DRY_RUN:
		return "DRY_RUN";
	case CUMPSGEMM_AUTO:
		return "AUTO";
	case CUMPSGEMM_UNDEFINED:
		return "UNDEFINED";
	case CUMPSGEMM_FP16TCEC_SCALING:
		return "FP16TCEC_SCALING";
	default:
		break;
	}
	return "Unknown";
}

extern "C" cuMpSGEMM_compute_mode_t cuMpSGEMM_get_compute_mode_internal (
		const char* const func_name,
		hipblasHandle_t const cublas_handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const unsigned m, const unsigned n, const unsigned k
		) {
	if (hijack_mode == dynamic_mode) {
		if (internal_global_control_func) {
			return internal_global_control_func(op_A, op_B, m, n, k);
		}
		cuMpSGEMM_compute_mode_t (*func)(
				const char* const func_name,
				hipblasHandle_t const cublas_handle,
				const hipblasOperation_t op_A,
				const hipblasOperation_t op_B,
				const unsigned m, const unsigned n, const unsigned k
				);
		*(void**)(&func) = cuMpSGEMM_get_function_pointer(rule_lib_name, __func__);

		if (func == nullptr) {
			return cuMpSGEMM_get_compute_mode(func_name, cublas_handle, op_A, op_B, m, n, k);
		}

		return func(func_name, cublas_handle, op_A, op_B, m, n, k);
	}
	return internal_global_compute_mode;
}

template <class T>
hipblasStatus_t cuMpSGEMM_hijack_core(
		const char* const func_name,
		hipblasHandle_t const cublas_handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const T* alpha,
		const T* const a_dmem_ptr, const uint64_t lda,
		const T* const b_dmem_ptr, const uint64_t ldb,
		const T* beta,
		T* const c_dmem_ptr, const uint64_t ldc
		) {
	hipStream_t hip_stream;
	hipblasGetStream(cublas_handle, &hip_stream);

	if (m == 0 || n == 0 || k == 0 || lda == 0 || ldb == 0 || ldc == 0) {
		return HIPBLAS_STATUS_INVALID_VALUE;
	}

	cumpsgemm::CULiP::profile_result profile_result;
	const auto profiling_flag = cumpsgemm::CULiP::is_profiling_enabled();

	cuMpSGEMM_compute_mode_t compute_mode =
		cuMpSGEMM_get_compute_mode_internal(
				func_name,
				cublas_handle,
				op_A,
				op_B,
				m, n, k
				);

	cuMpSGEMM_log(std::string(func_name) + " op=(" + get_cublas_op_str(op_A) + ", " + get_cublas_op_str(op_B) +
			"), shape=(" + std::to_string(m) + ", " + std::to_string(n) + ", " + std::to_string(k) + "), mode=" + cuMpSGEMM_get_compute_mode_string(compute_mode) +
			"[" + (hijack_mode == dynamic_mode ? "dynamic" : "static") + "][exp_stats:" + (cumpsgemm::hijack_control::get_internal_global_handle()->exp_stats_handle->enabled ? "1" : "0") + "]");
	cumpsgemm::hijack_control::set_last_called_function_str(
			std::string(func_name) + "," +
			get_cublas_op_str(op_A) + "," +
			get_cublas_op_str(op_B) + "," +
			std::to_string(m) + "," +
			std::to_string(n) + "," +
			std::to_string(k) + "," +
			"1," + // batch_size
			cuMpSGEMM_get_compute_mode_string(compute_mode)
			);

	if (compute_mode == CUMPSGEMM_DRY_RUN) {
		return HIPBLAS_STATUS_SUCCESS;
	}

	hipblasStatus_t res;

	// -----------------------------------
	// gemm_Mx2x2
	// -----------------------------------
	if (((m & (m - 1)) == 0) && n == 2 && k == 2 &&
			is_gemm_Mx2x2_enabled()) {

		if (profiling_flag) {
			const std::string func_name = std::string(std::is_same<T, float>::value ? "s" : "c") + "gemm_Mx2x2";
			snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-m%lu-n%lu-k%lu", func_name.c_str(), cumpsgemm::CULiP::get_cublasOperation_t_string(op_A), cumpsgemm::CULiP::get_cublasOperation_t_string(op_B), m, n, k);
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.start_timestamp);
		}
		cuMpSGEMM_log(" +---> gemm_Mx2x2");

		mtk::cugemm::gemm_Mx2x2(
				op_A, op_B,
				m,
				*alpha,
				a_dmem_ptr, lda,
				b_dmem_ptr, ldb,
				*beta,
				c_dmem_ptr, ldc,
				hip_stream
				);

		if (profiling_flag) {
			// Record end rimestamp
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.end_timestamp);

			// Print result
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::print_profile_result, (void*)&profile_result);
		}

		return HIPBLAS_STATUS_SUCCESS;
	}

	// -----------------------------------
	// gemm_2xNx2
	// -----------------------------------
	if (((n & (n - 1)) == 0) && m == 2 && k == 2 &&
			is_gemm_Mx2x2_enabled()) {

		if (profiling_flag) {
			const std::string func_name = std::string(std::is_same<T, float>::value ? "s" : "c") + "gemm_2xNx2";
			snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-m%lu-n%lu-k%lu", func_name.c_str(), cumpsgemm::CULiP::get_cublasOperation_t_string(op_A), cumpsgemm::CULiP::get_cublasOperation_t_string(op_B), m, n, k);
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.start_timestamp);
		}
		cuMpSGEMM_log(" +---> gemm_2xNx2");

		mtk::cugemm::gemm_2xNx2(
				op_A, op_B,
				n,
				*alpha,
				a_dmem_ptr, lda,
				b_dmem_ptr, ldb,
				*beta,
				c_dmem_ptr, ldc,
				hip_stream
				);

		if (profiling_flag) {
			// Record end rimestamp
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.end_timestamp);

			// Print result
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::print_profile_result, (void*)&profile_result);
		}

		return HIPBLAS_STATUS_SUCCESS;
	}

	if (compute_mode == CUMPSGEMM_CUBLAS || compute_mode == CUMPSGEMM_CUBLAS_FP16TC || compute_mode == CUMPSGEMM_CUBLAS_TF32TC || compute_mode == CUMPSGEMM_CUBLAS_SIMT) {
		// -----------------------------------
		// cuBLAS
		// -----------------------------------
		hipblasMath_t math_mode;
		hipblasGetMathMode(cublas_handle, &math_mode);
		if (compute_mode == CUMPSGEMM_CUBLAS_TF32TC) {
			hipblasSetMathMode(cublas_handle, HIPBLAS_TF32_TENSOR_OP_MATH);
		} else if (compute_mode == CUMPSGEMM_CUBLAS_FP16TC) {
			hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);
		} else if (compute_mode == CUMPSGEMM_CUBLAS_SIMT) {
			hipblasSetMathMode(cublas_handle, HIPBLAS_DEFAULT_MATH);
		}

		hipblasStatus_t (*func_ptr)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const T*, const T*, int, const T*, int, const T*, T*, int);
		*(void**)(&func_ptr) = cuMpSGEMM_get_function_pointer(
				cublas_lib_name.c_str(),
				func_name
				);
		if (func_ptr == nullptr) {
			cuMpSGEMM_error(std::string("Could not load cuBLAS function \"") + func_name + "\"");
		}

		if (profiling_flag) {
			snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-m%lu-n%lu-k%lu", func_name, cumpsgemm::CULiP::get_cublasOperation_t_string(op_A), cumpsgemm::CULiP::get_cublasOperation_t_string(op_B), m, n, k);
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.start_timestamp);
		}

		res = (*func_ptr)(cublas_handle, op_A, op_B, m, n, k, alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, beta, c_dmem_ptr, ldc);

		if (profiling_flag) {
			// Record end rimestamp
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.end_timestamp);

			// Print result
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::print_profile_result, (void*)&profile_result);
		}

		// restore math mode
		hipblasSetMathMode(cublas_handle, math_mode);

		if (cumpsgemm::hijack_control::get_internal_global_handle()->exp_stats_handle->enabled) {
			cumpsgemm::exp_stats::exp_stats_ext(
					cumpsgemm::hijack_control::get_internal_global_handle(),
					m, n,
					c_dmem_ptr, ldc,
					1,
					0
					);
		}

	} else {
		// -----------------------------------
		// cuMpSGEMM
		// -----------------------------------
		if (profiling_flag) {
			const std::string func_name = std::string(std::is_same<T, float>::value ? "s" : "c") + "gemm_" + std::string(cuMpSGEMM_get_compute_mode_string(compute_mode));
			snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-m%lu-n%lu-k%lu", func_name.c_str(), cumpsgemm::CULiP::get_cublasOperation_t_string(op_A), cumpsgemm::CULiP::get_cublasOperation_t_string(op_B), m, n, k);
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.start_timestamp);
		}

		unsigned A_exp_stats_id, B_exp_stats_id, dynamic_launch_id;
		if (compute_mode == CUMPSGEMM_AUTO) {
			// Exp stats
			cumpsgemm::exp_stats::exp_stats_ext(cuMpSGEMM_get_internal_global_handle(), (op_A == HIPBLAS_OP_N ? m : k), (op_A == HIPBLAS_OP_N ? k : m), a_dmem_ptr, lda, 1, 0);
			A_exp_stats_id = cumpsgemm::exp_stats::get_current_exp_stats_buffer_id(cuMpSGEMM_get_internal_global_handle());
			cumpsgemm::exp_stats::exp_stats_ext(cuMpSGEMM_get_internal_global_handle(), (op_B == HIPBLAS_OP_N ? k : n), (op_B == HIPBLAS_OP_N ? n : k), b_dmem_ptr, ldb, 1, 0);
			B_exp_stats_id = cumpsgemm::exp_stats::get_current_exp_stats_buffer_id(cuMpSGEMM_get_internal_global_handle());

			// Kernel dicision
			dynamic_launch_id = cumpsgemm::dynamic_launch::get_next_dynamic_launch_flag_buffer_id(cuMpSGEMM_get_internal_global_handle());
			cumpsgemm::dynamic_scaling::set_dynamic_launch_buffer_by_exp_stats(cuMpSGEMM_get_internal_global_handle(), dynamic_launch_id, A_exp_stats_id, B_exp_stats_id);

			cuMpSGEMM_run_if_env_defined(
					info_env_name,
					[&]() {
					int flag;
					cutf::memory::copy(&flag, cuMpSGEMM_get_internal_global_handle()->dynamic_launch_handle->flag_buffer + dynamic_launch_id, 1);
					const auto gemm_mode = cumpsgemm::dynamic_launch::utils::get_gemm_flag(flag);
					const auto scale_A = cumpsgemm::dynamic_launch::utils::get_scale_A_flag(flag);
					const auto scale_B = cumpsgemm::dynamic_launch::utils::get_scale_B_flag(flag);
					const auto loss_rate_A = cumpsgemm::get_exp_stats(cuMpSGEMM_get_internal_global_handle(), A_exp_stats_id);
					const auto loss_rate_B = cumpsgemm::get_exp_stats(cuMpSGEMM_get_internal_global_handle(), B_exp_stats_id);
					cuMpSGEMM_log(std::string("AUTO[ignore<") + get_XeY_format_string(cuMpSGEMM_get_internal_global_handle()->exp_stats_handle->ignore_threshold) + ", uf<"
							+ get_XeY_format_string(cuMpSGEMM_get_internal_global_handle()->exp_stats_handle->underflow_threshold) + ", tolerance="
							+ get_XeY_format_string(cuMpSGEMM_get_internal_global_handle()->exp_stats_handle->underflow_tolerance_rate)
							+ "]: GEMM_MODE=" + cuMpSGEMM_get_compute_mode_string((cuMpSGEMM_compute_mode_t)gemm_mode)
							+ ", loss_A=" + std::to_string(loss_rate_A.first) + "/" + std::to_string(loss_rate_A.second) + "(" + std::to_string(static_cast<double>(loss_rate_A.first) / loss_rate_A.second) + "), scale_A=" + std::to_string(scale_A)
							+ ", loss_B=" + std::to_string(loss_rate_B.first) + "/" + std::to_string(loss_rate_B.second) + "(" + std::to_string(static_cast<double>(loss_rate_B.first) / loss_rate_B.second) + "), scale_B=" + std::to_string(scale_B));
					});

			// Scaling
			cumpsgemm::dynamic_scaling::scale_A(cuMpSGEMM_get_internal_global_handle(), (op_A == HIPBLAS_OP_N ? m : k), (op_A == HIPBLAS_OP_N ? k : m), const_cast<T*>(a_dmem_ptr), lda, 0, 1, A_exp_stats_id, dynamic_launch_id);
			cumpsgemm::dynamic_scaling::scale_B(cuMpSGEMM_get_internal_global_handle(), (op_B == HIPBLAS_OP_N ? k : n), (op_B == HIPBLAS_OP_N ? n : k), const_cast<T*>(b_dmem_ptr), ldb, 0, 1, B_exp_stats_id, dynamic_launch_id);

			// Enable dynamic launch
			cumpsgemm::dynamic_launch::set_dynamic_launch_flag_buffer_id(cuMpSGEMM_get_internal_global_handle(), dynamic_launch_id);
		}

		res = cumpsgemm::gemm<T>(
				cuMpSGEMM_get_internal_global_handle(),
				op_A, op_B,
				m, n, k,
				alpha,
				a_dmem_ptr, lda,
				b_dmem_ptr, ldb,
				beta,
				c_dmem_ptr, ldc,
				compute_mode
				);

		if (compute_mode == CUMPSGEMM_AUTO) {
			cumpsgemm::dynamic_scaling::scale_C(cuMpSGEMM_get_internal_global_handle(), m, n, c_dmem_ptr, ldc, 0, 1, A_exp_stats_id, B_exp_stats_id, dynamic_launch_id);

			// restore A and B
			if (restore_AB) {
				cumpsgemm::dynamic_scaling::reset_scale_A(cuMpSGEMM_get_internal_global_handle(), (op_A == HIPBLAS_OP_N ? m : k), (op_A == HIPBLAS_OP_N ? k : m), const_cast<T*>(a_dmem_ptr), lda, 0, 1, A_exp_stats_id, dynamic_launch_id);
				cumpsgemm::dynamic_scaling::reset_scale_B(cuMpSGEMM_get_internal_global_handle(), (op_B == HIPBLAS_OP_N ? k : n), (op_B == HIPBLAS_OP_N ? n : k), const_cast<T*>(b_dmem_ptr), ldb, 0, 1, B_exp_stats_id, dynamic_launch_id);
			}

			cumpsgemm::dynamic_launch::unset_dynamic_launch_flag_buffer_id(cuMpSGEMM_get_internal_global_handle());
		}

		if (profiling_flag) {
			// Record end rimestamp
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.end_timestamp);

			// Print result
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::print_profile_result, (void*)&profile_result);
		}
	}

	return res;
}

template <class T>
hipblasStatus_t cuMpSGEMM_stridedBatched_hijack_core(
		const char* const func_name,
		hipblasHandle_t const cublas_handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const T* alpha,
		const T* const a_dmem_ptr, const uint64_t lda, const uint64_t stridea,
		const T* const b_dmem_ptr, const uint64_t ldb, const uint64_t strideb,
		const T* beta,
		T* const c_dmem_ptr, const uint64_t ldc, const uint64_t stridec,
		const uint64_t batch_count
		) {
	hipStream_t hip_stream;
	hipblasGetStream(cublas_handle, &hip_stream);

	if (m == 0 || n == 0 || k == 0 || lda == 0 || ldb == 0 || ldc == 0 || batch_count == 0) {
		return HIPBLAS_STATUS_INVALID_VALUE;
	}

	cumpsgemm::CULiP::profile_result profile_result;
	const auto profiling_flag = cumpsgemm::CULiP::is_profiling_enabled();

	cuMpSGEMM_compute_mode_t compute_mode =
		cuMpSGEMM_get_compute_mode_internal(
				func_name,
				cublas_handle,
				op_A,
				op_B,
				m, n, k
				);

	cuMpSGEMM_log(std::string(func_name) + " op=(" + get_cublas_op_str(op_A) + ", " + get_cublas_op_str(op_B) +
			"), shape=(" + std::to_string(m) + ", " + std::to_string(n) + ", " + std::to_string(k) + "), batch=" + std::to_string(batch_count) + ", mode=" + cuMpSGEMM_get_compute_mode_string(compute_mode) +
			"[" + (hijack_mode == dynamic_mode ? "dynamic" : "static") + "][exp_stats:" + (cumpsgemm::hijack_control::get_internal_global_handle()->exp_stats_handle->enabled ? "1" : "0") + "]");

	cumpsgemm::hijack_control::set_last_called_function_str(
			std::string(func_name) + "," +
			get_cublas_op_str(op_A) + "," +
			get_cublas_op_str(op_B) + "," +
			std::to_string(m) + "," +
			std::to_string(n) + "," +
			std::to_string(k) + "," +
			std::to_string(batch_count) + "," +
			cuMpSGEMM_get_compute_mode_string(compute_mode)
			);

	if (compute_mode == CUMPSGEMM_DRY_RUN) {
		return HIPBLAS_STATUS_SUCCESS;
	}

	hipblasStatus_t res;

	// -----------------------------------
	// gemm_Mx2x2
	// -----------------------------------
	if (((m & (m - 1)) == 0) && n == 2 && k == 2 &&
			is_gemm_Mx2x2_enabled()) {

		if (profiling_flag) {
			const std::string func_name = std::string(std::is_same<T, float>::value ? "s" : "c") + "gemm_strided_batch_Mx2x2";
			snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-m%lu-n%lu-k%lu-batchCount%lu",
					func_name.c_str(), cumpsgemm::CULiP::get_cublasOperation_t_string(op_A), cumpsgemm::CULiP::get_cublasOperation_t_string(op_B), m, n, k, batch_count);
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.start_timestamp);
		}
		cuMpSGEMM_log(" +---> gemm_Mx2x2");

		mtk::cugemm::gemm_strided_batch_Mx2x2(
				op_A, op_B,
				m,
				*alpha,
				a_dmem_ptr, lda, stridea,
				b_dmem_ptr, ldb, strideb,
				*beta,
				c_dmem_ptr, ldc, stridec,
				batch_count,
				hip_stream
				);

		if (profiling_flag) {
			// Record end rimestamp
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.end_timestamp);

			// Print result
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::print_profile_result, (void*)&profile_result);
		}

		return HIPBLAS_STATUS_SUCCESS;
	}

	// -----------------------------------
	// gemm_2xNx2
	// -----------------------------------
	if (((n & (n - 1)) == 0) && m == 2 && k == 2 &&
			is_gemm_Mx2x2_enabled()) {

		if (profiling_flag) {
			const std::string func_name = std::string(std::is_same<T, float>::value ? "s" : "c") + "gemm_strided_batch_2xNx2";
			snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-m%lu-n%lu-k%lu-batchCount%lu",
					func_name.c_str(), cumpsgemm::CULiP::get_cublasOperation_t_string(op_A), cumpsgemm::CULiP::get_cublasOperation_t_string(op_B), m, n, k, batch_count);
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.start_timestamp);
		}
		cuMpSGEMM_log(" +---> gemm_2xNx2");

		mtk::cugemm::gemm_strided_batch_2xNx2(
				op_A, op_B,
				n,
				*alpha,
				a_dmem_ptr, lda, stridea,
				b_dmem_ptr, ldb, strideb,
				*beta,
				c_dmem_ptr, ldc, stridec,
				batch_count,
				hip_stream
				);

		if (profiling_flag) {
			// Record end rimestamp
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.end_timestamp);

			// Print result
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::print_profile_result, (void*)&profile_result);
		}

		return HIPBLAS_STATUS_SUCCESS;
	}

	if (compute_mode == CUMPSGEMM_CUBLAS || compute_mode == CUMPSGEMM_CUBLAS_FP16TC || compute_mode == CUMPSGEMM_CUBLAS_TF32TC || compute_mode == CUMPSGEMM_CUBLAS_SIMT) {
		// -----------------------------------
		// cuBLAS
		// -----------------------------------
		hipblasMath_t math_mode;
		hipblasGetMathMode(cublas_handle, &math_mode);
		if (compute_mode == CUMPSGEMM_CUBLAS_TF32TC) {
			hipblasSetMathMode(cublas_handle, HIPBLAS_TF32_TENSOR_OP_MATH);
		} else if (compute_mode == CUMPSGEMM_CUBLAS_FP16TC) {
			hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);
		} else if (compute_mode == CUMPSGEMM_CUBLAS_SIMT) {
			hipblasSetMathMode(cublas_handle, HIPBLAS_DEFAULT_MATH);
		}

		hipblasStatus_t (*func_ptr)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const T*, const T*, int, long long int, const T*, int, long long int, const T*, T*, int, long long int, int);
		*(void**)(&func_ptr) = cuMpSGEMM_get_function_pointer(
				cublas_lib_name.c_str(),
				func_name
				);
		if (func_ptr == nullptr) {
			cuMpSGEMM_error(std::string("Could not load cuBLAS function \"") + func_name + "\"");
		}

		if (profiling_flag) {
			snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-m%lu-n%lu-k%lu-batchCount%lu", func_name, cumpsgemm::CULiP::get_cublasOperation_t_string(op_A), cumpsgemm::CULiP::get_cublasOperation_t_string(op_B), m, n, k, batch_count);
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.start_timestamp);
		}

		res = (*func_ptr)(cublas_handle, op_A, op_B, m, n, k, alpha, a_dmem_ptr, lda, stridea, b_dmem_ptr, ldb, strideb, beta, c_dmem_ptr, ldc, stridec, batch_count);

		if (profiling_flag) {
			// Record end rimestamp
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.end_timestamp);

			// Print result
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::print_profile_result, (void*)&profile_result);
		}

		hipblasSetMathMode(cublas_handle, math_mode);
	} else {
		// -----------------------------------
		// cuMpSGEMM
		// -----------------------------------
		if (profiling_flag) {
			const std::string func_name = std::string(std::is_same<T, float>::value ? "s" : "c") + "gemm_stridedBatch_" + std::string(cuMpSGEMM_get_compute_mode_string(compute_mode));
			snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-m%lu-n%lu-k%lu-batchCount%lu", func_name.c_str(), cumpsgemm::CULiP::get_cublasOperation_t_string(op_A), cumpsgemm::CULiP::get_cublasOperation_t_string(op_B), m, n, k, batch_count);
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.start_timestamp);
		}

		unsigned A_exp_stats_id, B_exp_stats_id, dynamic_launch_id;
		if (compute_mode == CUMPSGEMM_AUTO) {
			// Exp stats
			cumpsgemm::exp_stats::exp_stats_ext(cuMpSGEMM_get_internal_global_handle(), (op_A == HIPBLAS_OP_N ? m : k), (op_A == HIPBLAS_OP_N ? k : m), a_dmem_ptr, lda, batch_count, stridea);
			A_exp_stats_id = cumpsgemm::exp_stats::get_current_exp_stats_buffer_id(cuMpSGEMM_get_internal_global_handle());
			cumpsgemm::exp_stats::exp_stats_ext(cuMpSGEMM_get_internal_global_handle(), (op_B == HIPBLAS_OP_N ? k : n), (op_B == HIPBLAS_OP_N ? n : k), b_dmem_ptr, ldb, batch_count, strideb);
			B_exp_stats_id = cumpsgemm::exp_stats::get_current_exp_stats_buffer_id(cuMpSGEMM_get_internal_global_handle());

			// Kernel dicision
			dynamic_launch_id = cumpsgemm::dynamic_launch::get_next_dynamic_launch_flag_buffer_id(cuMpSGEMM_get_internal_global_handle());
			cumpsgemm::dynamic_scaling::set_dynamic_launch_buffer_by_exp_stats(cuMpSGEMM_get_internal_global_handle(), dynamic_launch_id, A_exp_stats_id, B_exp_stats_id);

			cuMpSGEMM_run_if_env_defined(
					info_env_name,
					[&]() {
					int flag;
					cutf::memory::copy(&flag, cuMpSGEMM_get_internal_global_handle()->dynamic_launch_handle->flag_buffer + dynamic_launch_id, 1);
					const auto gemm_mode = cumpsgemm::dynamic_launch::utils::get_gemm_flag(flag);
					const auto scale_A = cumpsgemm::dynamic_launch::utils::get_scale_A_flag(flag);
					const auto scale_B = cumpsgemm::dynamic_launch::utils::get_scale_B_flag(flag);
					const auto loss_rate_A = cumpsgemm::get_exp_stats(cuMpSGEMM_get_internal_global_handle(), A_exp_stats_id);
					const auto loss_rate_B = cumpsgemm::get_exp_stats(cuMpSGEMM_get_internal_global_handle(), B_exp_stats_id);
					cuMpSGEMM_log(std::string("AUTO[ignore<") + get_XeY_format_string(cuMpSGEMM_get_internal_global_handle()->exp_stats_handle->ignore_threshold) + ", uf<"
							+ get_XeY_format_string(cuMpSGEMM_get_internal_global_handle()->exp_stats_handle->underflow_threshold) + ", tolerance="
							+ get_XeY_format_string(cuMpSGEMM_get_internal_global_handle()->exp_stats_handle->underflow_tolerance_rate)
							+ "]: GEMM_MODE=" + cuMpSGEMM_get_compute_mode_string((cuMpSGEMM_compute_mode_t)gemm_mode)
							+ ", loss_A=" + std::to_string(loss_rate_A.first) + "/" + std::to_string(loss_rate_A.second) + "(" + std::to_string(static_cast<double>(loss_rate_A.first) / loss_rate_A.second) + "), scale_A=" + std::to_string(scale_A)
							+ ", loss_B=" + std::to_string(loss_rate_B.first) + "/" + std::to_string(loss_rate_B.second) + "(" + std::to_string(static_cast<double>(loss_rate_B.first) / loss_rate_B.second) + "), scale_B=" + std::to_string(scale_B));
					});

			// Scaling
			cumpsgemm::dynamic_scaling::scale_A(cuMpSGEMM_get_internal_global_handle(), (op_A == HIPBLAS_OP_N ? m : k), (op_A == HIPBLAS_OP_N ? k : m), const_cast<T*>(a_dmem_ptr), lda, stridea, batch_count, A_exp_stats_id, dynamic_launch_id);
			cumpsgemm::dynamic_scaling::scale_B(cuMpSGEMM_get_internal_global_handle(), (op_B == HIPBLAS_OP_N ? k : n), (op_B == HIPBLAS_OP_N ? n : k), const_cast<T*>(b_dmem_ptr), ldb, strideb, batch_count, B_exp_stats_id, dynamic_launch_id);

			// Enable dynamic launch
			cumpsgemm::dynamic_launch::set_dynamic_launch_flag_buffer_id(cuMpSGEMM_get_internal_global_handle(), dynamic_launch_id);
		}

		res = cumpsgemm::gemm_stridedBatch<T>(
				cuMpSGEMM_get_internal_global_handle(),
				op_A, op_B,
				m, n, k,
				alpha,
				a_dmem_ptr, lda, stridea,
				b_dmem_ptr, ldb, strideb,
				beta,
				c_dmem_ptr, ldc, stridec,
				batch_count,
				compute_mode
				);

		if (compute_mode == CUMPSGEMM_AUTO) {
			cumpsgemm::dynamic_scaling::scale_C(cuMpSGEMM_get_internal_global_handle(), m, n, c_dmem_ptr, ldc, stridec, batch_count, A_exp_stats_id, B_exp_stats_id, dynamic_launch_id);

			// restore A and B
			if (restore_AB) {
				cumpsgemm::dynamic_scaling::reset_scale_A(cuMpSGEMM_get_internal_global_handle(), (op_A == HIPBLAS_OP_N ? m : k), (op_A == HIPBLAS_OP_N ? k : m), const_cast<T*>(a_dmem_ptr), lda, stridea, batch_count, A_exp_stats_id, dynamic_launch_id);
				cumpsgemm::dynamic_scaling::reset_scale_B(cuMpSGEMM_get_internal_global_handle(), (op_B == HIPBLAS_OP_N ? k : n), (op_B == HIPBLAS_OP_N ? n : k), const_cast<T*>(b_dmem_ptr), ldb, strideb, batch_count, B_exp_stats_id, dynamic_launch_id);
			}
		}

		if (profiling_flag) {
			// Record end rimestamp
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.end_timestamp);

			// Print result
			cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::print_profile_result, (void*)&profile_result);
		}
	}
	return res;
}

// cuBLAS functions
extern "C" {
hipblasStatus_t hipblasSgemm (
		hipblasHandle_t cublas_handle,
		hipblasOperation_t op_A,
		hipblasOperation_t op_B,
		int m,
		int n,
		int k,
		const float* alpha,
		const float* a_dmem_ptr, int lda,
		const float* b_dmem_ptr, int ldb,
		const float* beta,
		float* c_dmem_ptr, int ldc
		) {
	hipStream_t hip_stream;
	hipblasGetStream(cublas_handle, &hip_stream);

	return cuMpSGEMM_hijack_core<float>(
			__func__,
			cublas_handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda,
			b_dmem_ptr, ldb,
			beta,
			c_dmem_ptr, ldc
			);
}

hipblasStatus_t hipblasCgemm (
		hipblasHandle_t cublas_handle,
		hipblasOperation_t op_A,
		hipblasOperation_t op_B,
		int m,
		int n,
		int k,
		const hipComplex* alpha,
		const hipComplex* a_dmem_ptr, int lda,
		const hipComplex* b_dmem_ptr, int ldb,
		const hipComplex* beta,
		hipComplex* c_dmem_ptr, int ldc
		) {
	return cuMpSGEMM_hijack_core<hipComplex>(
			__func__,
			cublas_handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda,
			b_dmem_ptr, ldb,
			beta,
			c_dmem_ptr, ldc
			);
}

hipblasStatus_t hipblasSgemmStridedBatched (
		hipblasHandle_t cublas_handle,
		hipblasOperation_t op_A,
		hipblasOperation_t op_B,
		int m,
		int n,
		int k,
		const float* alpha,
		const float* a_dmem_ptr, int lda, long long int stridea,
		const float* b_dmem_ptr, int ldb, long long int strideb,
		const float* beta,
		float* c_dmem_ptr, int ldc, long long int stridec,
		const int batch_count
		) {
	return cuMpSGEMM_stridedBatched_hijack_core<float>(
			__func__,
			cublas_handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda, stridea,
			b_dmem_ptr, ldb, strideb,
			beta,
			c_dmem_ptr, ldc, stridec,
			batch_count
			);
}

hipblasStatus_t hipblasCgemmStridedBatched (
		hipblasHandle_t cublas_handle,
		hipblasOperation_t op_A,
		hipblasOperation_t op_B,
		int m,
		int n,
		int k,
		const hipComplex* alpha,
		const hipComplex* a_dmem_ptr, int lda, const long long int stridea,
		const hipComplex* b_dmem_ptr, int ldb, const long long int strideb,
		const hipComplex* beta,
		hipComplex* c_dmem_ptr, int ldc, const long long int stridec,
		const int batch_count
		) {
	hipStream_t hip_stream;
	hipblasGetStream(cublas_handle, &hip_stream);

	return cuMpSGEMM_stridedBatched_hijack_core<hipComplex>(
			__func__,
			cublas_handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda, stridea,
			b_dmem_ptr, ldb, strideb,
			beta,
			c_dmem_ptr, ldc, stridec,
			batch_count
			);
}

hipblasStatus_t hipblasGemmEx(hipblasHandle_t handle, hipblasOperation_t transa,
                            hipblasOperation_t transb, int m, int n, int k,
                            const void *alpha, const void *A,
                            hipDataType Atype, int lda, const void *B,
                            hipDataType Btype, int ldb, const void *beta,
														void *C, hipDataType Ctype, int ldc,
														hipblasComputeType_t computeType,
														hipblasGemmAlgo_t algo) {
	if (Atype == HIP_R_32F && Btype == HIP_R_32F && Ctype == HIP_R_32F) {
		return hipblasSgemm(
				handle,
				transa, transb,
				m, n, k,
				reinterpret_cast<const float*>(alpha),
				reinterpret_cast<const float*>(A), lda,
				reinterpret_cast<const float*>(B), ldb,
				reinterpret_cast<const float*>(beta),
				reinterpret_cast<float*>(C), ldc
				);
	}
	if (Atype == HIP_C_32F && Btype == HIP_C_32F && Ctype == HIP_C_32F) {
		return hipblasCgemm(
				handle,
				transa, transb,
				m, n, k,
				reinterpret_cast<const hipComplex*>(alpha),
				reinterpret_cast<const hipComplex*>(A), lda,
				reinterpret_cast<const hipComplex*>(B), ldb,
				reinterpret_cast<const hipComplex*>(beta),
				reinterpret_cast<hipComplex*>(C), ldc
				);
	}

	hipStream_t hip_stream;
	hipblasGetStream(handle, &hip_stream);

	cumpsgemm::CULiP::profile_result profile_result;
	const auto profiling_flag = cumpsgemm::CULiP::is_profiling_enabled();

	hipblasStatus_t (*func_ptr)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void*, hipDataType, int, const void*, hipDataType, int, const void*, void*, hipDataType, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&func_ptr) = cuMpSGEMM_get_function_pointer(
			cublas_lib_name.c_str(),
			__func__
			);

	if (profiling_flag) {
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-m%d-n%d-k%d", __func__, cumpsgemm::CULiP::get_cublasOperation_t_string(transa), cumpsgemm::CULiP::get_cublasOperation_t_string(transb), m, n, k);
		cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.start_timestamp);
	}

	const auto res = (*func_ptr)(handle, transa, transb, m, n, k, alpha, A, Atype, lda, B, Btype, ldb, beta, C, Ctype, ldc, computeType, algo);

	if (profiling_flag) {
		// Record end rimestamp
		cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::print_profile_result, (void*)&profile_result);
	}

	return res;
}

hipblasStatus_t hipblasGemmStridedBatchedEx(hipblasHandle_t handle, hipblasOperation_t transa,
                            hipblasOperation_t transb, int m, int n, int k,
                            const void *alpha, const void *A,
                            hipDataType Atype, int lda, long long int strideA, const void *B,
                            hipDataType Btype, int ldb, long long int strideB, const void *beta,
														void *C, hipDataType Ctype, int ldc, long long int strideC,
														int batch_count,
														hipblasComputeType_t computeType,
														hipblasGemmAlgo_t algo) {
	if (Atype == HIP_R_32F && Btype == HIP_R_32F && Ctype == HIP_R_32F) {
		return hipblasSgemmStridedBatched(
				handle,
				transa, transb,
				m, n, k,
				reinterpret_cast<const float*>(alpha),
				reinterpret_cast<const float*>(A), lda, strideA,
				reinterpret_cast<const float*>(B), ldb, strideB,
				reinterpret_cast<const float*>(beta),
				reinterpret_cast<float*>(C), ldc, strideC,
				batch_count
				);
	}
	if (Atype == HIP_C_32F && Btype == HIP_C_32F && Ctype == HIP_C_32F) {
		return hipblasCgemmStridedBatched(
				handle,
				transa, transb,
				m, n, k,
				reinterpret_cast<const hipComplex*>(alpha),
				reinterpret_cast<const hipComplex*>(A), lda, strideA,
				reinterpret_cast<const hipComplex*>(B), ldb, strideB,
				reinterpret_cast<const hipComplex*>(beta),
				reinterpret_cast<hipComplex*>(C), ldc, strideC,
				batch_count
				);
	}

	hipStream_t hip_stream;
	hipblasGetStream(handle, &hip_stream);

	cumpsgemm::CULiP::profile_result profile_result;
	const auto profiling_flag = cumpsgemm::CULiP::is_profiling_enabled();

	hipblasStatus_t (*func_ptr)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void*, hipDataType, int, long long int, const void*, hipDataType, int, long long int, const void*, void*, hipDataType, int, long long int, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&func_ptr) = cuMpSGEMM_get_function_pointer(
			cublas_lib_name.c_str(),
			__func__
			);

	if (profiling_flag) {
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-m%d-n%d-k%d-batch_count%d", __func__, cumpsgemm::CULiP::get_cublasOperation_t_string(transa), cumpsgemm::CULiP::get_cublasOperation_t_string(transb), m, n, k, batch_count);
		cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.start_timestamp);
	}

	const auto res = (*func_ptr)(handle, transa, transb, m, n, k, alpha, A, Atype, lda, strideA, B, Btype, ldb, strideB, beta, C, Ctype, ldc, strideC, batch_count, computeType, algo);

	if (profiling_flag) {
		// Record end rimestamp
		cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		cumpsgemm::CULiP::launch_function(hip_stream, &cumpsgemm::CULiP::print_profile_result, (void*)&profile_result);
	}

	return res;
}
} // extern "C"

cuMpSGEMM_handle* cumpsgemm::hijack_control::get_internal_global_handle() {
	return cuMpSGEMM_get_internal_global_handle();
}

void cumpsgemm::hijack_control::set_compute_mode(const cuMpSGEMM_compute_mode_t mode) {
	internal_global_compute_mode = mode;
	hijack_mode = static_mode;
}

void cumpsgemm::hijack_control::unset_compute_mode() {
	hijack_mode = dynamic_mode;
}

void cumpsgemm::hijack_control::set_exp_stats_params(
		const float ignore_threshold,
		const float underflow_threshold,
		const float underflow_tolerance_rate
		) {
	cuMpSGEMM_log("AUTO config: ignore_threshold="         + get_XeY_format_string(ignore_threshold)         + " @" + std::string(__func__));
	cuMpSGEMM_log("AUTO config: underflow_threshold="      + get_XeY_format_string(underflow_threshold)      + " @" + std::string(__func__));
	cuMpSGEMM_log("AUTO config: underflow_tolerance_rate=" + get_XeY_format_string(underflow_tolerance_rate) + " @" + std::string(__func__));

	cumpsgemm::set_exp_stats_params(get_internal_global_handle(), ignore_threshold, underflow_threshold, underflow_tolerance_rate);
}

void cumpsgemm::hijack_control::reset_exp_stats_buffer_id() {
	cumpsgemm::exp_stats::reset_exp_stats_buffer_id(get_internal_global_handle());
}

std::string cumpsgemm::hijack_control::get_last_called_function_str() {
	return internal_global_last_called_function_str;
}

void cumpsgemm::hijack_control::set_last_called_function_str(
		const std::string func_str
		) {
	internal_global_last_called_function_str = func_str;
}

void cumpsgemm::hijack_control::clear_last_called_function_str() {
	cumpsgemm::hijack_control::set_last_called_function_str("");
}

void cumpsgemm::hijack_control::enable_custom_gemm_Mx2x2() {
	global_internal_gemm_Mx2x2_enabled  = true;
}

void cumpsgemm::hijack_control::disable_custom_gemm_Mx2x2() {
	global_internal_gemm_Mx2x2_enabled  = false;
}

void cumpsgemm::hijack_control::enable_restoring_AB_after_scaling() {
	restore_AB = true;
	cuMpSGEMM_log("AUTO config: restore_AB_scaling=True @" + std::string(__func__));
}

void cumpsgemm::hijack_control::disable_restoring_AB_after_scaling() {
	restore_AB = false;
	cuMpSGEMM_log("AUTO config: restore_AB_scaling=False @" + std::string(__func__));
}

bool cumpsgemm::hijack_control::is_library_loaded() {
	return true;
}

void cumpsgemm::hijack_control::set_control_function(
		const cumpsgemm::hijack_control::control_function_t control_func
		) {
	internal_global_control_func = control_func;
}

void cumpsgemm::hijack_control::unset_control_function() {
	internal_global_control_func = 0;
}
