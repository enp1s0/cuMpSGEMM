#include <string>
#include <hipblas.h>
#include <unistd.h>
#include <dlfcn.h>
#include <cumpsgemm/cumpsgemm.hpp>
#include <cumpsgemm/hijack_control.hpp>
#include "handle.hpp"

namespace {

cuMpSGEMM_handle_t internal_global_cuMpSGEMM_handle = nullptr;

enum hijack_control_t {
	static_mode,
	dynamic_mode
} hijack_mode = dynamic_mode;
cuMpSGEMM_compute_mode_t internal_global_compute_mode = CUMPSGEMM_CUBLAS;

const std::string info_env_name = "CUMPSGEMM_INFO";
void cuMpSGEMM_log(
		const std::string str
		) {
	const auto env = getenv(info_env_name.c_str());
	if (env != nullptr && std::string(env) != "0") {
		std::fprintf(stdout, "[cuMpSGEMM LOG] %s\n",
				str.c_str());
		std::fflush(stdout);
	}
}

const std::string error_env_name = "CUMPSGEMM_ERROR_LOG";
void cuMpSGEMM_error(
		const std::string str
		) {
	const auto env = getenv(error_env_name.c_str());
	if (env != nullptr && std::string(env) != "0") {
		std::fprintf(stdout, "[cuMpSGEMM ERROR] %s\n",
				str.c_str());
		std::fflush(stdout);
	}
}

void cuMpSGEMM_warning(
		const std::string str
		) {
	const auto env = getenv(error_env_name.c_str());
	if (env != nullptr && std::string(env) != "0") {
		std::fprintf(stdout, "[cuMpSGEMM WARNING] %s\n",
				str.c_str());
		std::fflush(stdout);
	}
}

void* cuMpSGEMM_get_function_pointer(const std::string library_name, const std::string function_name) {

	// Open the library
	const auto lib_ptr = dlopen(library_name.c_str(), RTLD_NOW);
	if (lib_ptr == nullptr) {
		cuMpSGEMM_warning("Failed to load " + library_name + ". Default rule will be used.");
		return nullptr;
	}

	// Get function pointer
	void* function_ptr = dlsym(lib_ptr, function_name.c_str());
	if (function_ptr == NULL) {
		cuMpSGEMM_warning("Failed to load a function " + function_name + " during selecting hijacking function. Default rule will be used.");
		return nullptr;
	}

	return function_ptr;
}

std::string get_cublas_op_str(const hipblasOperation_t op) {
	switch (op) {
	case HIPBLAS_OP_C:
		return "C";
	case HIPBLAS_OP_N:
		return "N";
	case HIPBLAS_OP_T:
		return "T";
	default:
		return "?";
	}
}

cuMpSGEMM_handle_t cuMpSGEMM_get_internal_global_handle() {
	if (internal_global_cuMpSGEMM_handle == nullptr) {
		cuMpSGEMM_create(&internal_global_cuMpSGEMM_handle);
	}
	return internal_global_cuMpSGEMM_handle;
}

const std::string rule_lib_name = "libcumpsgemm_rule.so";
const std::string cublas_lib_name = "libcublas.so";
} // noname namespace

extern "C" const char* cuMpSGEMM_get_compute_mode_string (
		const cuMpSGEMM_compute_mode_t mode
		) {
	switch (mode) {
	case CUMPSGEMM_CUBLAS:
		return "CUBLAS";
	case CUMPSGEMM_FP16TC:
		return "FP16TC";
	case CUMPSGEMM_FP16TCEC:
		return "FP16TCEC";
	case CUMPSGEMM_TF32TC:
		return "TF32TC";
	case CUMPSGEMM_TF32TCEC:
		return "TF32TCEC";
	case CUMPSGEMM_CUBLAS_SIMT:
		return "CUBLAS_SIMT";
	case CUMPSGEMM_CUBLAS_FP16TC:
		return "CUBLAS_FP16TC";
	case CUMPSGEMM_CUBLAS_TF32TC:
		return "CUBLAS_TF32TC";
	}
	return "Unknown";
}

extern "C" cuMpSGEMM_compute_mode_t cuMpSGEMM_get_compute_mode_internal (
		const char* const func_name,
		hipblasHandle_t const cublas_handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const unsigned m, const unsigned n, const unsigned k
		) {
	if (hijack_mode == dynamic_mode) {
		cuMpSGEMM_compute_mode_t (*func)(
				const char* const func_name,
				hipblasHandle_t const cublas_handle,
				const hipblasOperation_t op_A,
				const hipblasOperation_t op_B,
				const unsigned m, const unsigned n, const unsigned k
				);
		*(void**)(&func) = cuMpSGEMM_get_function_pointer(rule_lib_name, __func__);

		if (func == nullptr) {
			return cuMpSGEMM_get_compute_mode(func_name, cublas_handle, op_A, op_B, m, n, k);
		}

		return func(func_name, cublas_handle, op_A, op_B, m, n, k);
	}
	return internal_global_compute_mode;
}

template <class T>
hipblasStatus_t cuMpSGEMM_hijack_core(
		const char* const func_name,
		hipblasHandle_t const cublas_handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const T* alpha,
		const T* const a_dmem_ptr, const uint64_t lda,
		const T* const b_dmem_ptr, const uint64_t ldb,
		const T* beta,
		T* const c_dmem_ptr, const uint64_t ldc
		) {
	if (std::is_same<T, float>::value && (op_A == HIPBLAS_OP_C || op_B == HIPBLAS_OP_C)) {
		return HIPBLAS_STATUS_INVALID_VALUE;
	}

	hipStream_t hip_stream;
	hipblasGetStream(cublas_handle, &hip_stream);

	cuMpSGEMM_compute_mode_t compute_mode =
		cuMpSGEMM_get_compute_mode_internal(
				func_name,
				cublas_handle,
				op_A,
				op_B,
				m, n, k
				);

	cuMpSGEMM_log(std::string(func_name) + " op=(" + get_cublas_op_str(op_A) + ", " + get_cublas_op_str(op_B) +
			"), shape=(" + std::to_string(m) + ", " + std::to_string(n) + ", " + std::to_string(k) + "), mode=" + cuMpSGEMM_get_compute_mode_string(compute_mode) + "[" + (hijack_mode == dynamic_mode ? "dynamic" : "static") + "]");

	if (compute_mode == CUMPSGEMM_CUBLAS || compute_mode == CUMPSGEMM_CUBLAS_FP16TC || compute_mode == CUMPSGEMM_CUBLAS_TF32TC || compute_mode == CUMPSGEMM_CUBLAS_SIMT) {
		hipblasMath_t math_mode;
		hipblasGetMathMode(cublas_handle, &math_mode);
		if (compute_mode == CUMPSGEMM_CUBLAS_TF32TC) {
			hipblasSetMathMode(cublas_handle, HIPBLAS_TF32_TENSOR_OP_MATH);
		} else if (compute_mode == CUMPSGEMM_CUBLAS_FP16TC) {
			hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);
		} else if (compute_mode == CUMPSGEMM_CUBLAS_SIMT) {
			hipblasSetMathMode(cublas_handle, HIPBLAS_DEFAULT_MATH);
		}

		hipblasStatus_t (*func_ptr)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const T*, const T*, int, const T*, int, const T*, T*, int);
		*(void**)(&func_ptr) = cuMpSGEMM_get_function_pointer(
				cublas_lib_name.c_str(),
				func_name
				);
		if (func_ptr == nullptr) {
			cuMpSGEMM_error(std::string("Could not load cuBLAS function \"") + func_name + "\"");
		}
		const auto res = (*func_ptr)(cublas_handle, op_A, op_B, m, n, k, alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, beta, c_dmem_ptr, ldc);

		// restore math mode
		hipblasSetMathMode(cublas_handle, math_mode);
		return res;
	}

	return cumpsgemm::gemm<T>(
			cuMpSGEMM_get_internal_global_handle(),
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda,
			b_dmem_ptr, ldb,
			beta,
			c_dmem_ptr, ldc,
			compute_mode
			);
}

template <class T>
hipblasStatus_t cuMpSGEMM_stridedBatched_hijack_core(
		const char* const func_name,
		hipblasHandle_t const cublas_handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const T* alpha,
		const T* const a_dmem_ptr, const uint64_t lda, const uint64_t stridea,
		const T* const b_dmem_ptr, const uint64_t ldb, const uint64_t strideb,
		const T* beta,
		T* const c_dmem_ptr, const uint64_t ldc, const uint64_t stridec,
		const uint64_t batch_count
		) {
	if (std::is_same<T, float>::value && (op_A == HIPBLAS_OP_C || op_B == HIPBLAS_OP_C)) {
		return HIPBLAS_STATUS_INVALID_VALUE;
	}

	hipStream_t hip_stream;
	hipblasGetStream(cublas_handle, &hip_stream);

	cuMpSGEMM_compute_mode_t compute_mode =
		cuMpSGEMM_get_compute_mode_internal(
				func_name,
				cublas_handle,
				op_A,
				op_B,
				m, n, k
				);

	cuMpSGEMM_log(std::string(func_name) + " op=(" + get_cublas_op_str(op_A) + ", " + get_cublas_op_str(op_B) +
			"), shape=(" + std::to_string(m) + ", " + std::to_string(n) + ", " + std::to_string(k) + "), batch=" + std::to_string(batch_count) + ", mode=" + cuMpSGEMM_get_compute_mode_string(compute_mode) + "[" + (hijack_mode == dynamic_mode ? "dynamic" : "static") + "]");

	if (compute_mode == CUMPSGEMM_CUBLAS || compute_mode == CUMPSGEMM_CUBLAS_FP16TC || compute_mode == CUMPSGEMM_CUBLAS_TF32TC || compute_mode == CUMPSGEMM_CUBLAS_SIMT) {
		hipblasMath_t math_mode;
		hipblasGetMathMode(cublas_handle, &math_mode);
		if (compute_mode == CUMPSGEMM_CUBLAS_TF32TC) {
			hipblasSetMathMode(cublas_handle, HIPBLAS_TF32_TENSOR_OP_MATH);
		} else if (compute_mode == CUMPSGEMM_CUBLAS_FP16TC) {
			hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);
		} else if (compute_mode == CUMPSGEMM_CUBLAS_SIMT) {
			hipblasSetMathMode(cublas_handle, HIPBLAS_DEFAULT_MATH);
		}

		hipblasStatus_t (*func_ptr)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const T*, const T*, int, long long int, const T*, int, long long int, const T*, T*, int, long long int, int);
		*(void**)(&func_ptr) = cuMpSGEMM_get_function_pointer(
				cublas_lib_name.c_str(),
				func_name
				);
		if (func_ptr == nullptr) {
			cuMpSGEMM_error(std::string("Could not load cuBLAS function \"") + func_name + "\"");
		}
		const auto res = (*func_ptr)(cublas_handle, op_A, op_B, m, n, k, alpha, a_dmem_ptr, lda, stridea, b_dmem_ptr, ldb, strideb, beta, c_dmem_ptr, ldc, stridec, batch_count);
		hipblasSetMathMode(cublas_handle, math_mode);
		return res;
	}

	return cumpsgemm::gemm_stridedBatch<T>(
			cuMpSGEMM_get_internal_global_handle(),
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda, stridea,
			b_dmem_ptr, ldb, strideb,
			beta,
			c_dmem_ptr, ldc, stridec,
			batch_count,
			compute_mode
			);
}

// cuBLAS functions
extern "C" {
hipblasStatus_t hipblasSgemm (
		hipblasHandle_t cublas_handle,
		hipblasOperation_t op_A,
		hipblasOperation_t op_B,
		int m,
		int n,
		int k,
		const float* alpha,
		const float* a_dmem_ptr, int lda,
		const float* b_dmem_ptr, int ldb,
		const float* beta,
		float* c_dmem_ptr, int ldc
		) {
	hipStream_t hip_stream;
	hipblasGetStream(cublas_handle, &hip_stream);

	return cuMpSGEMM_hijack_core<float>(
			__func__,
			cublas_handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda,
			b_dmem_ptr, ldb,
			beta,
			c_dmem_ptr, ldc
			);
}

hipblasStatus_t hipblasCgemm (
		hipblasHandle_t cublas_handle,
		hipblasOperation_t op_A,
		hipblasOperation_t op_B,
		int m,
		int n,
		int k,
		const hipComplex* alpha,
		const hipComplex* a_dmem_ptr, int lda,
		const hipComplex* b_dmem_ptr, int ldb,
		const hipComplex* beta,
		hipComplex* c_dmem_ptr, int ldc
		) {
	return cuMpSGEMM_hijack_core<hipComplex>(
			__func__,
			cublas_handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda,
			b_dmem_ptr, ldb,
			beta,
			c_dmem_ptr, ldc
			);
}

hipblasStatus_t hipblasSgemmStridedBatched (
		hipblasHandle_t cublas_handle,
		hipblasOperation_t op_A,
		hipblasOperation_t op_B,
		int m,
		int n,
		int k,
		const float* alpha,
		const float* a_dmem_ptr, int lda, long long int stridea,
		const float* b_dmem_ptr, int ldb, long long int strideb,
		const float* beta,
		float* c_dmem_ptr, int ldc, long long int stridec,
		const int batch_count
		) {
	return cuMpSGEMM_stridedBatched_hijack_core<float>(
			__func__,
			cublas_handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda, stridea,
			b_dmem_ptr, ldb, strideb,
			beta,
			c_dmem_ptr, ldc, stridec,
			batch_count
			);
}

hipblasStatus_t hipblasCgemmStridedBatched (
		hipblasHandle_t cublas_handle,
		hipblasOperation_t op_A,
		hipblasOperation_t op_B,
		int m,
		int n,
		int k,
		const hipComplex* alpha,
		const hipComplex* a_dmem_ptr, int lda, const long long int stridea,
		const hipComplex* b_dmem_ptr, int ldb, const long long int strideb,
		const hipComplex* beta,
		hipComplex* c_dmem_ptr, int ldc, const long long int stridec,
		const int batch_count
		) {
	hipStream_t hip_stream;
	hipblasGetStream(cublas_handle, &hip_stream);

	return cuMpSGEMM_stridedBatched_hijack_core<hipComplex>(
			__func__,
			cublas_handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda, stridea,
			b_dmem_ptr, ldb, strideb,
			beta,
			c_dmem_ptr, ldc, stridec,
			batch_count
			);
}

hipblasStatus_t hipblasGemmEx(hipblasHandle_t handle, hipblasOperation_t transa,
                            hipblasOperation_t transb, int m, int n, int k,
                            const void *alpha, const void *A,
                            hipDataType Atype, int lda, const void *B,
                            hipDataType Btype, int ldb, const void *beta,
														void *C, hipDataType Ctype, int ldc,
														hipblasComputeType_t computeType,
														hipblasGemmAlgo_t algo) {
	cuMpSGEMM_compute_mode_t compute_mode =
		cuMpSGEMM_get_compute_mode_internal(
				__func__,
				handle,
				transa,
				transb,
				m, n, k
				);

	if (compute_mode != CUMPSGEMM_CUBLAS) {
		if (Atype == HIP_R_32F && Btype == HIP_R_32F && Ctype == HIP_R_32F) {
			return hipblasSgemm(
					handle,
					transa, transb,
					m, n, k,
					reinterpret_cast<const float*>(alpha),
					reinterpret_cast<const float*>(A), lda,
					reinterpret_cast<const float*>(B), ldb,
					reinterpret_cast<const float*>(beta),
					reinterpret_cast<float*>(C), ldc
					);
		}
		if (Atype == HIP_C_32F && Btype == HIP_C_32F && Ctype == HIP_C_32F) {
			return hipblasCgemm(
					handle,
					transa, transb,
					m, n, k,
					reinterpret_cast<const hipComplex*>(alpha),
					reinterpret_cast<const hipComplex*>(A), lda,
					reinterpret_cast<const hipComplex*>(B), ldb,
					reinterpret_cast<const hipComplex*>(beta),
					reinterpret_cast<hipComplex*>(C), ldc
					);
		}
	}
	hipblasStatus_t (*func_ptr)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void*, hipDataType, int, const void*, hipDataType, int, const void*, void*, hipDataType, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&func_ptr) = cuMpSGEMM_get_function_pointer(
			cublas_lib_name.c_str(),
			__func__
			);
	return (*func_ptr)(handle, transa, transb, m, n, k, alpha, A, Atype, lda, B, Btype, ldb, beta, C, Ctype, ldc, computeType, algo);
}

hipblasStatus_t hipblasGemmStridedBatchedEx(hipblasHandle_t handle, hipblasOperation_t transa,
                            hipblasOperation_t transb, int m, int n, int k,
                            const void *alpha, const void *A,
                            hipDataType Atype, int lda, long long int strideA, const void *B,
                            hipDataType Btype, int ldb, long long int strideB, const void *beta,
														void *C, hipDataType Ctype, int ldc, long long int strideC,
														int batch_count,
														hipblasComputeType_t computeType,
														hipblasGemmAlgo_t algo) {
	cuMpSGEMM_compute_mode_t compute_mode =
		cuMpSGEMM_get_compute_mode_internal(
				__func__,
				handle,
				transa,
				transb,
				m, n, k
				);

	if (compute_mode != CUMPSGEMM_CUBLAS) {
		if (Atype == HIP_R_32F && Btype == HIP_R_32F && Ctype == HIP_R_32F) {
			return hipblasSgemmStridedBatched(
					handle,
					transa, transb,
					m, n, k,
					reinterpret_cast<const float*>(alpha),
					reinterpret_cast<const float*>(A), lda, strideA,
					reinterpret_cast<const float*>(B), ldb, strideB,
					reinterpret_cast<const float*>(beta),
					reinterpret_cast<float*>(C), ldc, strideC,
					batch_count
					);
		}
		if (Atype == HIP_C_32F && Btype == HIP_C_32F && Ctype == HIP_C_32F) {
			return hipblasCgemmStridedBatched(
					handle,
					transa, transb,
					m, n, k,
					reinterpret_cast<const hipComplex*>(alpha),
					reinterpret_cast<const hipComplex*>(A), lda, strideA,
					reinterpret_cast<const hipComplex*>(B), ldb, strideB,
					reinterpret_cast<const hipComplex*>(beta),
					reinterpret_cast<hipComplex*>(C), ldc, strideC,
					batch_count
					);
		}
	}
	hipblasStatus_t (*func_ptr)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void*, hipDataType, int, long long int, const void*, hipDataType, int, long long int, const void*, void*, hipDataType, int, long long int, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&func_ptr) = cuMpSGEMM_get_function_pointer(
			cublas_lib_name.c_str(),
			__func__
			);
	return (*func_ptr)(handle, transa, transb, m, n, k, alpha, A, Atype, lda, strideA, B, Btype, ldb, strideB, beta, C, Ctype, ldc, strideC, batch_count, computeType, algo);
}
} // extern "C"

cuMpSGEMM_handle* cumpsgemm::hijack_control::get_internal_global_handle() {
	if (internal_global_cuMpSGEMM_handle == nullptr) {
		cuMpSGEMM_create(&internal_global_cuMpSGEMM_handle);
	}
	return internal_global_cuMpSGEMM_handle;
}

void cumpsgemm::hijack_control::set_compute_mode(const cuMpSGEMM_compute_mode_t mode) {
	internal_global_compute_mode = mode;
	hijack_mode = static_mode;
}

void cumpsgemm::hijack_control::unset_compute_mode() {
	hijack_mode = dynamic_mode;
}

std::vector<std::pair<std::size_t, std::size_t>> cumpsgemm::hijack_control::get_last_exp_stats() {
	return cumpsgemm::get_last_exp_stats(get_internal_global_handle());
}

void cumpsgemm::hijack_control::enable_exp_stats() {
	cumpsgemm::enable_exp_stats(get_internal_global_handle());
}

void cumpsgemm::hijack_control::disable_exp_stats() {
	cumpsgemm::disable_exp_stats(get_internal_global_handle());
	cumpsgemm::hijack_control::unset_compute_mode();
}

void cumpsgemm::hijack_control::set_exp_stats_params(
		const float ignore_threshold,
		const float lost_threshold
		) {
	cumpsgemm::set_exp_stats_params(get_internal_global_handle(), ignore_threshold, lost_threshold);
}

bool cumpsgemm::hijack_control::is_exp_stats_enabled() {
	return get_internal_global_handle()->exp_stats_enabled;
}
