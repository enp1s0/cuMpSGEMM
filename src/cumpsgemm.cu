#include <iostream>
#include <unistd.h>
#include <cassert>
#include <dlfcn.h>
#include <cumpsgemm/cumpsgemm.hpp>

#include "cumpsgemm_internal.hpp"

extern "C" const char* cuMpSGEMM_get_compute_mode_string (
		const cuMpSGEMM_compute_mode_t mode
		) {
	switch (mode) {
	case CUMPSGEMM_CUBLAS:
		return "CUBLAS";
	case CUMPSGEMM_FP16TC:
		return "FP16TC";
	case CUMPSGEMM_FP16TCEC:
		return "FP16TCEC";
	case CUMPSGEMM_TF32TC:
		return "TF32TC";
	case CUMPSGEMM_TF32TCEC:
		return "TF32TCEC";
	}
	return "Unknown";
}

namespace {
void* cuMpSGEMM_get_function_pointer(const std::string library_name, const std::string function_name) {

	// Open the library
	const auto lib_ptr = dlopen(library_name.c_str(), RTLD_NOW);
	if (lib_ptr == NULL) {
		std::fprintf(stderr, "[cuMpSGEMM ERROR] Failed to load the real library %s\n", library_name.c_str());
		exit(1);
	}

	// Get function pointer
	void* function_ptr = dlsym(lib_ptr, function_name.c_str());
	if (function_ptr == NULL) {
		fprintf(stderr, "[cuMpSGEMM ERROR] Failed to load the function %s\n", __func__);
		exit(1);
	}

	return function_ptr;
}

const std::string rule_lib_name = "libcumpsgemm_rule.so";
} // noname namespace

extern "C" cuMpSGEMM_compute_mode_t cuMpSGEMM_get_compute_mode (
		const char* const func_name,
		hipblasHandle_t const cublas_handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const unsigned m, const unsigned n, const unsigned k
		) {
	cuMpSGEMM_compute_mode_t (*func)(
			const char* const func_name,
			hipblasHandle_t const cublas_handle,
			const hipblasOperation_t op_A,
			const hipblasOperation_t op_B,
			const unsigned m, const unsigned n, const unsigned k
			);
	*(void**)(&func) = cuMpSGEMM_get_function_pointer(rule_lib_name, __func__);

	return func(func_name, cublas_handle, op_A, op_B, m, n, k);
}


// cuMpGEMM implementation

namespace {
template <
	class T,
	unsigned SMEM_M,
	unsigned SMEM_N,
	unsigned SMEM_K,
	unsigned FRAG_M,
	unsigned FRAG_N,
	unsigned FRAG_K,
	unsigned BLOCK_SIZE,
	class TC_T,
	class EC
>
void layout_selector (
			const hipblasOperation_t op_A,
			const hipblasOperation_t op_B,
			const std::size_t m,
			const std::size_t n,
			const std::size_t k,
			const T alpha,
			const T* const a_ptr, const std::size_t lda,
			const T* const b_ptr, const std::size_t ldb,
			const T beta,
			T* const c_ptr, const std::size_t ldc,
			hipStream_t hip_stream
		) {
#define CASE(A, a, B, b) \
	if (op_A == a && op_B == b) {cumpsgemm::launch_kernel<T, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, A, B, TC_T, EC>(m, n, k, alpha, a_ptr, lda, b_ptr, ldb, beta, c_ptr, ldc, hip_stream);return;}

	CASE(cumpsgemm::col_major, HIPBLAS_OP_N, cumpsgemm::col_major, HIPBLAS_OP_N);
	CASE(cumpsgemm::row_major, HIPBLAS_OP_T, cumpsgemm::col_major, HIPBLAS_OP_N);
	CASE(cumpsgemm::conjugate, HIPBLAS_OP_C, cumpsgemm::col_major, HIPBLAS_OP_N);
	CASE(cumpsgemm::col_major, HIPBLAS_OP_N, cumpsgemm::row_major, HIPBLAS_OP_T);
	CASE(cumpsgemm::row_major, HIPBLAS_OP_T, cumpsgemm::row_major, HIPBLAS_OP_T);
	CASE(cumpsgemm::conjugate, HIPBLAS_OP_C, cumpsgemm::row_major, HIPBLAS_OP_T);
	CASE(cumpsgemm::col_major, HIPBLAS_OP_N, cumpsgemm::conjugate, HIPBLAS_OP_C);
	CASE(cumpsgemm::row_major, HIPBLAS_OP_T, cumpsgemm::conjugate, HIPBLAS_OP_C);
	CASE(cumpsgemm::conjugate, HIPBLAS_OP_C, cumpsgemm::conjugate, HIPBLAS_OP_C);
}
} // unnamed namespace

template <class T>
hipblasStatus_t cumpsgemm::gemm(
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const T* alpha,
		const T* const a_dmem_ptr, const uint64_t lda,
		const T* const b_dmem_ptr, const uint64_t ldb,
		const T* beta,
		T* const c_dmem_ptr, const uint64_t ldc,
		const cuMpSGEMM_compute_mode_t compute_mode,
		hipStream_t hip_stream
		) {
	switch (compute_mode) {
	case CUMPSGEMM_FP16TC:   layout_selector<T, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, half                         , mtk::wmma::tcec::without_ec>(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, *beta, c_dmem_ptr, ldc, hip_stream);break;
	case CUMPSGEMM_FP16TCEC: layout_selector<T, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, half                         , mtk::wmma::tcec::with_ec   >(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, *beta, c_dmem_ptr, ldc, hip_stream);break;
	case CUMPSGEMM_TF32TC:   layout_selector<T, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, nvcuda::wmma::precision::tf32, mtk::wmma::tcec::without_ec>(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, *beta, c_dmem_ptr, ldc, hip_stream);break;
	case CUMPSGEMM_TF32TCEC: layout_selector<T, SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, nvcuda::wmma::precision::tf32, mtk::wmma::tcec::with_ec   >(op_A, op_B, m, n, k, *alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, *beta, c_dmem_ptr, ldc, hip_stream);break;
	default:break;
	}

	return HIPBLAS_STATUS_SUCCESS;
}


extern "C" hipblasStatus_t cuMpSGEMM_sgemm(
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const float* alpha,
		const float* const a_dmem_ptr, const uint64_t lda,
		const float* const b_dmem_ptr, const uint64_t ldb,
		const float* beta,
		float* const c_dmem_ptr, const uint64_t ldc,
		const cuMpSGEMM_compute_mode_t compute_mode,
		hipStream_t hip_stream
		) {
	assert(op_A != HIPBLAS_OP_C);
	assert(op_B != HIPBLAS_OP_C);
	return cumpsgemm::gemm<float>(
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda,
			b_dmem_ptr, ldb,
			beta,
			c_dmem_ptr, ldc,
			compute_mode,
			hip_stream
			);
}

extern "C" hipblasStatus_t cuMpSGEMM_cgemm(
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const hipComplex* alpha,
		const hipComplex* const a_dmem_ptr, const uint64_t lda,
		const hipComplex* const b_dmem_ptr, const uint64_t ldb,
		const hipComplex* beta,
		hipComplex* const c_dmem_ptr, const uint64_t ldc,
		const cuMpSGEMM_compute_mode_t compute_mode,
		hipStream_t hip_stream
		) {
	return cumpsgemm::gemm<hipComplex>(
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda,
			b_dmem_ptr, ldb,
			beta,
			c_dmem_ptr, ldc,
			compute_mode,
			hip_stream
			);
}
