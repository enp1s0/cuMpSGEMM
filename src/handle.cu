#include <cstddef>
#include <cumpsgemm/cumpsgemm.h>
#include "handle.hpp"
#include "cumpsgemm_kernel.cuh"

#define SET_GEMM_KERNEL_MODULE(module_list, io_t, tc_t, ec, op_a, op_b, smem_m, smem_n, smem_k, frag_m, frag_n, frag_k, block_size, num_unrollings, num_stages, pipelined, gemm_type, stage) \
	module_list[cumpsgemm::kernel_module_code::tc_t | cumpsgemm::kernel_module_code::ec | cumpsgemm::kernel_module_code::op_a_##op_a | cumpsgemm::kernel_module_code::op_b_##op_b | cumpsgemm::kernel_module_code::gemm_type][stage] =\
	cumpsgemm::generate_gemm_module<io_t,smem_m,smem_n,smem_k,frag_m,frag_n,frag_k,block_size,num_unrollings,num_stages,cumpsgemm::op_a,cumpsgemm::op_b,tc_t,mtk::wmma::tcec::ec, pipelined>();

#define SET_GEMM_STRIDEDBATCH_KERNEL_MODULE(module_list, io_t, tc_t, ec, op_a, op_b, smem_m, smem_n, smem_k, frag_m, frag_n, frag_k, block_size, num_unrollings, num_stages, pipelined, gemm_type, stage) \
	module_list[cumpsgemm::kernel_module_code::tc_t | cumpsgemm::kernel_module_code::ec | cumpsgemm::kernel_module_code::op_a_##op_a | cumpsgemm::kernel_module_code::op_b_##op_b | cumpsgemm::kernel_module_code::gemm_type][stage] =\
	cumpsgemm::generate_gemm_stridedBatch_module<io_t,smem_m,smem_n,smem_k,frag_m,frag_n,frag_k,block_size,num_unrollings,num_stages,cumpsgemm::op_a,cumpsgemm::op_b,tc_t,mtk::wmma::tcec::ec, pipelined>();

#define COMPILE_SGEMM_KERNEL
#define COMPILE_CGEMM_KERNEL
#define COMPILE_SGEMM_STRIDEDBATCH_KERNEL
#define COMPILE_CGEMM_STRIDEDBATCH_KERNEL

#define ENABLE_A100_OPTIMAZED_PARAMETERS

extern "C" {
hipblasStatus_t cuMpSGEMM_create(cuMpSGEMM_handle_t* const handle) {
	if ((*handle = new cuMpSGEMM_handle) == nullptr) {
		return HIPBLAS_STATUS_INTERNAL_ERROR;
	}

	int num_sms;
	hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
	(*handle)->num_sms = num_sms;

	using tf32 = nvcuda::wmma::precision::tf32;

#ifdef ENABLE_A100_OPTIMAZED_PARAMETERS
	// set kernel modules
#ifdef COMPILE_SGEMM_KERNEL
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, col_major,  64, 128,  32,  32,  64,  32, 128,   1,   2, false, s, 0); // N=  16384, p= 47.33 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, col_major,  64, 128,  32,  32,  64,  32, 128,   1,   2, false, s, 1); // N=   4096, p= 46.33 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, col_major,  64, 128,  32,  32,  64,  32, 128,   1,   2, false, s, 2); // N=   1024, p= 21.54 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, col_major,  64, 128,  32,  32,  64,  16, 128,   1,   2, false, s, 0); // N=  16384, p= 29.71 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, col_major,  64, 128,  32,  64,  32,  16, 128,   1,   2, false, s, 1); // N=   4096, p= 29.51 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, col_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, s, 2); // N=   1024, p= 20.13 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, col_major, 128, 128,  32,  64,  64,  16, 128,   1,   2, false, s, 0); // N=  16384, p= 75.69 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, col_major, 128, 128,  32,  64,  64,  32, 128,   1,   2, false, s, 1); // N=   4096, p= 87.86 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, col_major, 128, 128,  64,  64,  64,  32, 128,   2,   2, false, s, 2); // N=   1024, p= 47.52 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, col_major,  64, 128,  32,  64,  32,  16, 128,   2,   2, false, s, 0); // N=  16384, p= 64.93 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, col_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 1); // N=   4096, p= 69.67 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, col_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 2); // N=   1024, p= 37.49 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, row_major, 128,  64,  32,  64,  32,  32, 128,   1,   2, false, s, 0); // N=  16384, p= 39.64 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, row_major, 128,  64,  32,  64,  32,  32, 128,   1,   2, false, s, 1); // N=   4096, p= 40.81 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, row_major, 128,  32,  32,  32,  32,  16, 128,   2,   2, false, s, 2); // N=   1024, p= 24.70 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, row_major,  64, 128,  32,  32,  64,  16, 128,   1,   2, false, s, 0); // N=  16384, p= 30.35 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, row_major,  64, 128,  32,  32,  64,  16, 128,   1,   2, false, s, 1); // N=   4096, p= 30.38 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, row_major,  32, 128,  32,  32,  32,  16, 128,   1,   2, false, s, 2); // N=   1024, p= 18.67 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, row_major, 128, 128,  32,  64,  64,  32, 128,   1,   2, false, s, 0); // N=  16384, p= 69.42 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, row_major, 128, 128,  32,  64,  64,  32, 128,   1,   2, false, s, 1); // N=   4096, p= 76.70 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, row_major, 128, 128,  64,  64,  64,  64, 128,   1,   2, false, s, 2); // N=   1024, p= 43.22 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, row_major, 128, 128,  32,  32,  64,  16, 256,   1,   2, false, s, 0); // N=  16384, p= 65.32 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, row_major, 128,  64,  32,  64,  32,  32, 128,   2,   2, false, s, 1); // N=   4096, p= 71.81 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, row_major,  64,  64,  32,  32,  32,  32, 128,   1,   2, false, s, 2); // N=   1024, p= 41.37 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, col_major,  64, 128,  32,  64,  32,  32, 128,   1,   2, false, s, 0); // N=  16384, p= 48.66 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, col_major,  64, 128,  32,  32,  64,  32, 128,   1,   2, false, s, 1); // N=   4096, p= 51.17 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, col_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=   1024, p= 27.83 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, col_major,  64, 128,  32,  64,  32,  16, 128,   1,   2, false, s, 0); // N=  16384, p= 30.00 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, col_major,  64, 128,  32,  64,  32,  16, 128,   1,   2, false, s, 1); // N=   4096, p= 29.82 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, col_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, s, 2); // N=   1024, p= 18.83 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, col_major,  64, 128,  32,  32,  64,  16, 128,   1,   2, false, s, 0); // N=  16384, p= 73.09 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, col_major, 128, 128,  32,  64,  64,  32, 128,   1,   2, false, s, 1); // N=   4096, p= 95.97 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, col_major, 128, 128,  64,  64,  64,  32, 128,   1,   2, false, s, 2); // N=   1024, p= 51.86 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, col_major,  64, 128,  32,  32,  64,  32, 128,   2,   2, false, s, 0); // N=  16384, p= 61.45 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, col_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 1); // N=   4096, p= 66.13 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, col_major, 128, 128,  32,  64,  64,  32, 128,   1,   2, false, s, 2); // N=   1024, p= 34.48 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, row_major,  64, 128,  32,  64,  32,  32, 128,   1,   2, false, s, 0); // N=  16384, p= 46.80 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, row_major,  64, 128,  32,  64,  32,  32, 128,   1,   2, false, s, 1); // N=   4096, p= 48.76 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, row_major, 128,  32,  32,  32,  32,  16, 128,   1,   2, false, s, 2); // N=   1024, p= 26.49 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, row_major,  64, 128,  32,  32,  64,  16, 128,   1,   2, false, s, 0); // N=  16384, p= 30.49 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, row_major,  64, 128,  32,  32,  64,  16, 128,   1,   2, false, s, 1); // N=   4096, p= 30.29 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, row_major,  32, 128,  32,  32,  32,  16, 128,   1,   2, false, s, 2); // N=   1024, p= 18.95 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, row_major,  64, 128,  32,  64,  32,  32, 128,   1,   2, false, s, 0); // N=  16384, p= 68.53 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, row_major, 128, 128,  32,  64,  64,  32, 128,   1,   2, false, s, 1); // N=   4096, p= 83.53 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, row_major, 128, 128,  64,  64,  64,  32, 128,   1,   2, false, s, 2); // N=   1024, p= 48.39 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, row_major,  64, 128,  32,  32,  64,  32, 128,   1,   2, false, s, 0); // N=  16384, p= 64.50 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, row_major,  64, 128,  32,  32,  64,  32, 128,   2,   2, false, s, 1); // N=   4096, p= 69.98 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, row_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, s, 2); // N=   1024, p= 38.56 [TFlop/s]
#endif
#ifdef COMPILE_CGEMM_KERNEL
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, col_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 46.01 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, col_major,  64, 128,  32,  16,  64,  16, 256,   2,   2, false, c, 1); // N=   2048, p= 33.33 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, col_major,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 17.78 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, col_major,  64, 128,  32,  16,  64,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 31.35 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, col_major,  64,  64,  32,  16,  64,  16, 128,   2,   2, false, c, 1); // N=   2048, p= 30.10 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, col_major,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 18.28 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, col_major, 128,  64,  32,  32,  64,  16, 128,   2,   2, false, c, 0); // N=   8192, p= 96.94 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, col_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 92.30 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, col_major,  64,  64,  64,  16,  64,  32, 128,   2,   2, false, c, 2); // N=    512, p= 34.33 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, col_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 66.51 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, col_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=   2048, p= 62.94 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, col_major,  32,  32,  32,  16,  16,  32, 128,   1,   2, false, c, 2); // N=    512, p= 29.02 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, row_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 0); // N=   8192, p= 40.78 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, row_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 40.68 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, row_major,  32,  32,  32,  16,  16,  32, 128,   1,   2, false, c, 2); // N=    512, p= 19.13 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, row_major, 128,  64,  32,  16,  64,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 31.21 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, row_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 29.72 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, row_major,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 16.74 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, row_major, 128, 128,  32,  32,  64,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 73.16 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, row_major,  64, 128,  32,  32,  64,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 67.79 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, row_major,  64,  64,  64,  32,  32,  32, 128,   1,   2, false, c, 2); // N=    512, p= 26.70 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, row_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 66.45 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, row_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=   2048, p= 61.95 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, row_major,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 27.80 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   8192, p= 39.59 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 1); // N=   2048, p= 35.83 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 17.04 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   8192, p= 28.55 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, conjugate,  64,  32,  32,  16,  32,  32, 128,   1,   2, false, c, 1); // N=   2048, p= 27.16 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 15.40 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, conjugate, 128, 128,  32,  32,  64,  16, 256,   2,   2, false, c, 0); // N=   8192, p= 65.10 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, conjugate, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 49.61 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, conjugate, 128,  32,  32,  32,  32,  16, 128,   2,   2, false, c, 2); // N=    512, p= 20.20 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, conjugate, 128, 128,  32,  32,  64,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 55.12 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, conjugate,  64,  64,  32,  32,  32,  32, 128,   1,   2, false, c, 1); // N=   2048, p= 49.27 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 22.58 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, col_major,  64, 128,  32,  32,  32,  16, 256,   2,   2, false, c, 0); // N=   8192, p= 53.69 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, col_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=   2048, p= 51.44 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, col_major,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 26.57 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, col_major, 128,  64,  32,  16,  64,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 31.06 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, col_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 30.28 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, col_major,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 17.62 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, col_major,  64, 128,  32,  32,  64,  16, 128,   1,   2, false, c, 0); // N=   8192, p=119.95 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, col_major, 128,  64,  32,  64,  32,  16, 128,   1,   2, false, c, 1); // N=   2048, p=120.93 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, col_major,  32,  64,  32,  32,  16,  16, 128,   2,   2, false, c, 2); // N=    512, p= 43.71 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, col_major, 128, 128,  32,  32,  64,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 65.12 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, col_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=   2048, p= 60.01 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, col_major,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 28.26 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, row_major,  64, 128,  32,  32,  32,  16, 256,   2,   2, false, c, 0); // N=   8192, p= 51.36 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, row_major,  64, 128,  32,  32,  32,  16, 256,   2,   2, false, c, 1); // N=   2048, p= 48.21 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, row_major,  64,  64,  64,  32,  32,  16, 128,   2,   2, false, c, 2); // N=    512, p= 22.50 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, row_major, 128,  64,  32,  16,  64,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 31.01 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, row_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 30.05 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, row_major,  32,  32,  32,  16,  16,  16, 128,   2,   2, false, c, 2); // N=    512, p= 16.93 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, row_major,  64, 128,  32,  64,  32,  32, 128,   2,   2, false, c, 0); // N=   8192, p=101.51 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, row_major,  64, 128,  32,  64,  32,  16, 128,   2,   2, false, c, 1); // N=   2048, p= 94.37 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, row_major,  64,  64,  64,  64,  16,  32, 128,   2,   2, false, c, 2); // N=    512, p= 36.75 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, row_major,  64, 128,  32,  32,  32,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 65.21 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, row_major,  64, 128,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=   2048, p= 60.19 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, row_major,  32,  32,  32,  16,  16,  32, 128,   1,   2, false, c, 2); // N=    512, p= 28.83 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 0); // N=   8192, p= 44.04 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, conjugate,  64,  64,  32,  64,  16,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 41.01 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 18.45 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   8192, p= 28.44 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, conjugate,  64,  32,  32,  16,  32,  32, 128,   1,   2, false, c, 1); // N=   2048, p= 27.14 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 15.19 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, conjugate, 128, 128,  32,  64,  32,  16, 256,   2,   2, false, c, 0); // N=   8192, p= 87.88 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, conjugate, 128,  64,  32,  64,  16,  16, 256,   1,   2, false, c, 1); // N=   2048, p= 61.03 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, conjugate, 128,  32,  32,  32,  16,  16, 256,   2,   2, false, c, 2); // N=    512, p= 26.25 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, conjugate, 128, 128,  32,  32,  64,  16, 256,   2,   2, false, c, 0); // N=   8192, p= 53.60 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, conjugate,  64,  64,  32,  32,  32,  32, 128,   2,   2, false, c, 1); // N=   2048, p= 49.07 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 22.12 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, col_major,  64,  64,  32,  64,  16,  16, 128,   1,   2, false, c, 0); // N=   8192, p= 48.00 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, col_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 43.61 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, col_major,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 21.35 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, col_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 0); // N=   8192, p= 29.65 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, col_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 27.16 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, col_major,  32,  32,  32,  16,  16,  16, 128,   2,   2, false, c, 2); // N=    512, p= 15.83 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, col_major, 128, 128,  32,  32,  64,  16, 256,   2,   2, false, c, 0); // N=   8192, p=112.07 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, col_major,  64, 128,  32,  32,  32,  16, 256,   2,   2, false, c, 1); // N=   2048, p= 74.09 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, col_major,  32, 128,  32,  32,  16,  16, 256,   1,   2, false, c, 2); // N=    512, p= 32.85 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, col_major, 128, 128,  32,  32,  64,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 61.07 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, col_major,  64,  64,  32,  32,  32,  32, 128,   1,   2, false, c, 1); // N=   2048, p= 50.24 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, col_major,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 24.56 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, row_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 0); // N=   8192, p= 46.15 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, row_major,  64,  64,  32,  64,  16,  16, 128,   2,   2, false, c, 1); // N=   2048, p= 42.13 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, row_major,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 19.72 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, row_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   8192, p= 29.25 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, row_major,  64,  64,  32,  16,  64,  16, 128,   2,   2, false, c, 1); // N=   2048, p= 26.90 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, row_major,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 15.98 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, row_major, 128, 128,  32,  64,  32,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 91.11 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, row_major,  64,  64,  32,  64,  16,  16, 128,   2,   2, false, c, 1); // N=   2048, p= 65.06 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, row_major,  32, 128,  32,  32,  16,  16, 256,   1,   2, false, c, 2); // N=    512, p= 26.82 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, row_major, 128, 128,  32,  32,  64,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 61.49 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, row_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 50.85 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, row_major,  32,  32,  32,  16,  16,  32, 128,   1,   2, false, c, 2); // N=    512, p= 23.08 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, conjugate,  64,  64,  32,  64,  16,  16, 128,   1,   2, false, c, 0); // N=   8192, p= 42.83 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, conjugate,  64,  64,  32,  64,  16,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 39.64 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, conjugate,  32,  32,  32,  16,  16,  32, 128,   1,   2, false, c, 2); // N=    512, p= 17.38 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, conjugate,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 0); // N=   8192, p= 26.22 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, conjugate,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 23.91 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 14.49 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, conjugate, 128, 128,  32,  64,  32,  16, 256,   1,   2, false, c, 0); // N=   8192, p= 75.10 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, conjugate,  64,  64,  32,  64,  16,  32, 128,   1,   2, false, c, 1); // N=   2048, p= 57.81 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, conjugate,  64,  64,  64,  32,  16,  16, 256,   2,   2, false, c, 2); // N=    512, p= 21.11 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, conjugate,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 0); // N=   8192, p= 48.24 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, conjugate,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=   2048, p= 46.23 [TFlop/s]
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 2); // N=    512, p= 19.53 [TFlop/s]
#endif

#ifdef COMPILE_SGEMM_STRIDEDBATCH_KERNEL
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, col_major, 128,  64,  32,  32,  64,  32, 128,   1,   2, false, s, 0); // N=   1024, p= 44.68 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, col_major, 128,  64,  32,  32,  64,  32, 128,   1,   2, false, s, 1); // N=    256, p= 24.63 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, col_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  6.38 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, col_major, 128,  64,  32,  64,  32,  16, 128,   1,   2, false, s, 0); // N=   1024, p= 27.86 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, col_major, 128,  64,  32,  64,  32,  16, 128,   1,   2, false, s, 1); // N=    256, p= 21.60 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, col_major,  64,  64,  64,  32,  32,  32, 128,   1,   2, false, s, 2); // N=     64, p=  7.33 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, col_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 0); // N=   1024, p= 85.03 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, col_major, 128,  64,  32,  32,  64,  32, 128,   1,   2, false, s, 1); // N=    256, p= 41.12 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, col_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  9.22 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, col_major, 128, 128,  32,  64,  64,  16, 128,   2,   2, false, s, 0); // N=   1024, p= 67.49 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, col_major, 128,  64,  32,  32,  64,  32, 128,   1,   2, false, s, 1); // N=    256, p= 39.24 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, col_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  9.17 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, row_major, 128,  64,  32,  64,  32,  32, 128,   1,   2, false, s, 0); // N=   1024, p= 38.41 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, row_major, 128,  64,  32,  64,  32,  32, 128,   1,   2, false, s, 1); // N=    256, p= 26.91 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, row_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  8.14 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, row_major, 128,  64,  32,  32,  64,  16, 128,   2,   2, false, s, 0); // N=   1024, p= 28.10 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, row_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, s, 1); // N=    256, p= 21.80 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, row_major,  64,  64,  64,  32,  32,  32, 128,   1,   2, false, s, 2); // N=     64, p=  7.39 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, row_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 0); // N=   1024, p= 76.12 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, row_major, 128, 128,  32,  64,  64,  16, 128,   1,   2, false, s, 1); // N=    256, p= 39.12 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, row_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  9.02 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, row_major, 128, 128,  32,  64,  64,  32, 128,   1,   2, false, s, 0); // N=   1024, p= 68.45 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, row_major, 128,  64,  32,  64,  32,  32, 128,   1,   2, false, s, 1); // N=    256, p= 40.95 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, row_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  9.17 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, col_major, 128,  64,  32,  64,  32,  32, 128,   1,   2, false, s, 0); // N=   1024, p= 47.68 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, col_major, 128,  64,  32,  64,  32,  32, 128,   1,   2, false, s, 1); // N=    256, p= 33.01 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, col_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  8.47 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, col_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, s, 0); // N=   1024, p= 26.66 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, col_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, s, 1); // N=    256, p= 21.10 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, col_major,  64,  64,  64,  32,  32,  32, 128,   1,   2, false, s, 2); // N=     64, p=  7.17 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, col_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 0); // N=   1024, p= 90.90 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, col_major, 128,  64,  32,  32,  32,  32, 256,   1,   2, false, s, 1); // N=    256, p= 41.68 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, col_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  9.64 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, col_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 0); // N=   1024, p= 62.73 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, col_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, s, 1); // N=    256, p= 38.33 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, col_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  8.78 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, row_major, 128,  64,  32,  64,  32,  32, 128,   2,   2, false, s, 0); // N=   1024, p= 45.04 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, row_major, 128,  64,  32,  64,  32,  32, 128,   1,   2, false, s, 1); // N=    256, p= 30.68 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, row_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  8.18 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, row_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, s, 0); // N=   1024, p= 27.84 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, row_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, s, 1); // N=    256, p= 21.88 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, row_major,  64,  64,  64,  32,  32,  32, 128,   1,   2, false, s, 2); // N=     64, p=  7.36 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, row_major, 128, 128,  32,  64,  64,  32, 128,   2,   2, false, s, 0); // N=   1024, p= 83.69 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, row_major, 128,  64,  32,  64,  32,  32, 128,   2,   2, false, s, 1); // N=    256, p= 40.49 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, row_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  9.37 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, row_major, 128, 128,  32,  64,  64,  16, 128,   1,   2, false, s, 0); // N=   1024, p= 65.96 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, row_major, 128,  64,  32,  32,  64,  32, 128,   1,   2, false, s, 1); // N=    256, p= 39.27 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, row_major,  64,  64,  64,  32,  32,  64, 128,   1,   2, false, s, 2); // N=     64, p=  9.17 [TFlop/s]
#endif
#ifdef COMPILE_CGEMM_STRIDEDBATCH_KERNEL
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, col_major,  64,  64,  32,  16,  64,  32, 128,   1,   2, false, c, 0); // N=   1024, p= 44.93 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, col_major,  64,  64,  32,  16,  64,  32, 128,   1,   2, false, c, 1); // N=    256, p= 30.66 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, col_major,  64,  32,  32,  16,  32,  32, 128,   2,   2, false, c, 2); // N=     64, p= 14.31 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, col_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 30.79 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, col_major,  64,  32,  32,  16,  32,  32, 128,   1,   2, false, c, 1); // N=    256, p= 24.96 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, col_major,  64,  32,  32,  16,  32,  16, 128,   2,   2, false, c, 2); // N=     64, p= 15.43 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, col_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 88.76 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, col_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, c, 1); // N=    256, p= 64.39 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, col_major,  64,  64,  64,  16,  64,  32, 128,   1,   2, false, c, 2); // N=     64, p= 24.05 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, col_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 0); // N=   1024, p= 61.04 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, col_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=    256, p= 46.78 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, col_major,  64,  64,  64,  32,  16,  32, 256,   1,   2, false, c, 2); // N=     64, p= 20.34 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, row_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   1024, p= 41.29 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, row_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 1); // N=    256, p= 32.49 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, row_major,  64,  64,  64,  32,  32,  16, 128,   1,   2, false, c, 2); // N=     64, p= 17.57 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, row_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 28.77 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, row_major,  64,  32,  32,  16,  32,  32, 128,   2,   2, false, c, 1); // N=    256, p= 24.71 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, row_major,  64,  32,  32,  16,  32,  16, 128,   1,   2, false, c, 2); // N=     64, p= 15.32 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, row_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 67.23 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, row_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, c, 1); // N=    256, p= 50.76 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, row_major,  64,  64,  64,  32,  32,  16, 128,   1,   2, false, c, 2); // N=     64, p= 21.30 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, row_major,  64,  64,  32,  32,  32,  32, 128,   2,   2, false, c, 0); // N=   1024, p= 57.31 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, row_major,  64,  64,  32,  32,  32,  32, 128,   2,   2, false, c, 1); // N=    256, p= 43.23 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, row_major,  64,  64,  64,  32,  16,  32, 256,   1,   2, false, c, 2); // N=     64, p= 20.34 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   1024, p= 38.97 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 1); // N=    256, p= 27.73 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, conjugate,  64,  32,  32,  32,  16,  16, 128,   1,   2, false, c, 2); // N=     64, p= 15.50 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, conjugate,  64,  32,  32,  16,  32,  32, 128,   1,   2, false, c, 0); // N=   1024, p= 27.11 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, conjugate,  64,  32,  32,  16,  32,  32, 128,   1,   2, false, c, 1); // N=    256, p= 22.45 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, conjugate,  64,  64,  64,  16,  32,  16, 256,   2,   2, false, c, 2); // N=     64, p= 14.28 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, conjugate, 128, 128,  32,  32,  64,  16, 256,   1,   2, false, c, 0); // N=   1024, p= 59.50 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, conjugate, 128, 128,  32,  32,  64,  16, 256,   1,   2, false, c, 1); // N=    256, p= 36.40 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, conjugate,  64,  64,  64,  32,  32,  16, 128,   1,   2, false, c, 2); // N=     64, p= 17.21 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, conjugate,  64,  64,  32,  32,  32,  32, 128,   1,   2, false, c, 0); // N=   1024, p= 52.08 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, conjugate,  64,  64,  32,  32,  16,  16, 256,   2,   2, false, c, 1); // N=    256, p= 37.48 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, conjugate,  64,  64,  64,  32,  16,  32, 256,   2,   2, false, c, 2); // N=     64, p= 18.75 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, col_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   1024, p= 51.34 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, col_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=    256, p= 39.90 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, col_major,  64,  32,  32,  32,  16,  32, 128,   2,   2, false, c, 2); // N=     64, p= 19.74 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, col_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 31.33 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, col_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 1); // N=    256, p= 25.08 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, col_major,  64,  64,  64,  16,  32,  16, 256,   2,   2, false, c, 2); // N=     64, p= 15.43 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, col_major, 128,  64,  32,  64,  32,  16, 128,   1,   2, false, c, 0); // N=   1024, p=119.62 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, col_major, 128,  64,  32,  32,  64,  16, 128,   1,   2, false, c, 1); // N=    256, p= 77.28 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, col_major,  64,  64,  64,  64,  16,  32, 128,   2,   2, false, c, 2); // N=     64, p= 25.91 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, col_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 0); // N=   1024, p= 59.02 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, col_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=    256, p= 45.77 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, col_major,  64,  32,  32,  32,  16,  16, 128,   1,   2, false, c, 2); // N=     64, p= 20.46 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, row_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   1024, p= 47.92 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, row_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=    256, p= 38.19 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, row_major,  64,  64,  64,  32,  32,  16, 128,   2,   2, false, c, 2); // N=     64, p= 19.17 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, row_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 31.18 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, row_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 1); // N=    256, p= 24.90 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, row_major,  64,  32,  32,  16,  32,  16, 128,   2,   2, false, c, 2); // N=     64, p= 15.43 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, row_major, 128,  64,  32,  64,  32,  16, 128,   2,   2, false, c, 0); // N=   1024, p= 90.58 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, row_major, 128,  64,  32,  64,  32,  32, 128,   1,   2, false, c, 1); // N=    256, p= 65.40 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, row_major,  64,  64,  64,  64,  16,  32, 128,   2,   2, false, c, 2); // N=     64, p= 24.67 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, row_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 0); // N=   1024, p= 58.91 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, row_major, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=    256, p= 45.61 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, row_major,  64,  32,  32,  32,  16,  16, 128,   1,   2, false, c, 2); // N=     64, p= 20.46 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 44.86 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, conjugate, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=    256, p= 31.60 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, conjugate,  64,  32,  32,  32,  16,  32, 128,   1,   2, false, c, 2); // N=     64, p= 17.03 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, conjugate,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   1024, p= 27.46 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, conjugate,  64,  32,  32,  16,  32,  32, 128,   1,   2, false, c, 1); // N=    256, p= 22.66 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, conjugate,  64,  64,  64,  16,  32,  16, 256,   2,   2, false, c, 2); // N=     64, p= 13.89 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, conjugate, 128, 128,  32,  64,  32,  16, 256,   1,   2, false, c, 0); // N=   1024, p= 76.47 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, conjugate, 128,  64,  32,  64,  16,  16, 256,   1,   2, false, c, 1); // N=    256, p= 46.43 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, conjugate,  64,  64,  64,  32,  16,  16, 256,   2,   2, false, c, 2); // N=     64, p= 20.84 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, conjugate,  64,  64,  32,  32,  32,  32, 128,   1,   2, false, c, 0); // N=   1024, p= 52.63 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, conjugate, 128,  64,  32,  32,  32,  16, 256,   1,   2, false, c, 1); // N=    256, p= 36.62 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, conjugate,  64,  64,  64,  32,  16,  64, 256,   1,   2, false, c, 2); // N=     64, p= 18.44 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, col_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 47.84 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, col_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=    256, p= 32.18 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, col_major,  64,  64,  64,  32,  16,  16, 256,   2,   2, false, c, 2); // N=     64, p= 17.61 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, col_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 29.00 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, col_major,  64,  64,  32,  16,  64,  16, 128,   1,   2, false, c, 1); // N=    256, p= 21.77 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, col_major,  64,  64,  64,  16,  32,  16, 256,   2,   2, false, c, 2); // N=     64, p= 14.19 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, col_major, 128, 128,  32,  64,  32,  16, 256,   2,   2, false, c, 0); // N=   1024, p= 95.88 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, col_major, 128, 128,  32,  64,  32,  16, 256,   1,   2, false, c, 1); // N=    256, p= 47.06 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, col_major,  64,  64,  64,  32,  16,  16, 256,   2,   2, false, c, 2); // N=     64, p= 22.98 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, col_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 53.66 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, col_major,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=    256, p= 34.70 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, col_major,  64,  64,  64,  32,  16,  64, 256,   1,   2, false, c, 2); // N=     64, p= 18.34 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, row_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   1024, p= 45.18 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, row_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 1); // N=    256, p= 30.69 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, row_major,  64,  64,  64,  16,  32,  16, 256,   2,   2, false, c, 2); // N=     64, p= 17.16 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, row_major,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   1024, p= 27.39 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, row_major,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 1); // N=    256, p= 20.94 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, row_major,  64,  64,  64,  16,  32,  16, 256,   1,   2, false, c, 2); // N=     64, p= 14.28 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, row_major, 128, 128,  32,  64,  32,  16, 256,   1,   2, false, c, 0); // N=   1024, p= 78.89 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, row_major, 128, 128,  32,  64,  32,  16, 256,   1,   2, false, c, 1); // N=    256, p= 41.69 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, row_major,  64,  64,  64,  32,  16,  16, 256,   2,   2, false, c, 2); // N=     64, p= 20.84 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, row_major,  64,  64,  32,  32,  32,  32, 128,   1,   2, false, c, 0); // N=   1024, p= 53.13 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, row_major,  64,  64,  32,  32,  32,  32, 128,   1,   2, false, c, 1); // N=    256, p= 34.32 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, row_major,  64,  64,  64,  32,  16,  64, 256,   2,   2, false, c, 2); // N=     64, p= 18.49 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, conjugate,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 41.35 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, conjugate,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 1); // N=    256, p= 26.40 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, conjugate,  64,  64,  64,  32,  16,  32, 256,   2,   2, false, c, 2); // N=     64, p= 15.90 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, conjugate,  64,  64,  32,  32,  32,  16, 128,   1,   2, false, c, 0); // N=   1024, p= 25.68 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 1); // N=    256, p= 18.68 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, conjugate,  64,  64,  64,  16,  32,  16, 256,   1,   2, false, c, 2); // N=     64, p= 13.29 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, conjugate, 128, 128,  32,  64,  32,  16, 256,   1,   2, false, c, 0); // N=   1024, p= 70.08 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, conjugate, 128, 128,  32,  64,  32,  16, 256,   1,   2, false, c, 1); // N=    256, p= 37.41 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, conjugate,  64,  64,  64,  32,  16,  16, 256,   2,   2, false, c, 2); // N=     64, p= 19.07 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, conjugate,  64,  64,  32,  32,  32,  16, 128,   2,   2, false, c, 0); // N=   1024, p= 47.19 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, conjugate,  32,  32,  32,  16,  16,  16, 128,   1,   2, false, c, 1); // N=    256, p= 28.63 [TFlop/s]
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, conjugate,  64,  32,  32,  16,  16,  16, 256,   1,   2, false, c, 2); // N=     64, p= 17.48 [TFlop/s]
#endif

#else // ENABLE_A100_OPTIMAZED_PARAMETERS
#ifdef COMPILE_SGEMM_KERNEL
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, col_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, col_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, with_ec   , row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, with_ec   , row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , half, without_ec, row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, float    , tf32, without_ec, row_major, row_major, 128, 64, 32, 32, 32, 32, 128, 1, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
#endif
#ifdef COMPILE_CGEMM_KERNEL
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, half, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_KERNEL_MODULE((*handle)->gemm_module, hipComplex, tf32, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 32, 128, 1, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
#endif

#ifdef COMPILE_SGEMM_STRIDEDBATCH_KERNEL
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , half, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, float    , tf32, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, s, 2); // Not optimized but works on any Ampere GPUs
#endif
#ifdef COMPILE_CGEMM_STRIDEDBATCH_KERNEL
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, col_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, row_major, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, col_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, row_major, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, with_ec   , conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, half, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 0); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 1); // Not optimized but works on any Ampere GPUs
	SET_GEMM_STRIDEDBATCH_KERNEL_MODULE((*handle)->gemm_stridedBatch_module, hipComplex, tf32, without_ec, conjugate, conjugate, 64, 64, 32, 32, 32, 16, 128, 2, 2, false, c, 2); // Not optimized but works on any Ampere GPUs
#endif
#endif
	return HIPBLAS_STATUS_SUCCESS;
}

hipblasStatus_t cuMpSGEMM_destroy(cuMpSGEMM_handle_t handle) {
	delete handle;
	return HIPBLAS_STATUS_SUCCESS;
}

hipblasStatus_t cuMpSGEMM_set_stream(cuMpSGEMM_handle_t handle, const hipStream_t hip_stream) {
	handle->hip_stream = hip_stream;
	return HIPBLAS_STATUS_SUCCESS;
}
}
