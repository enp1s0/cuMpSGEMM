#include "hip/hip_runtime.h"
#include <cutf/memory.hpp>
#include <cumpsgemm/hijack_control.hpp>
#include "dynamic_launch.hpp"

namespace {
__global__ void init_flag_buffer(
		int* const ptr
		) {
	ptr[0] =             CUMPSGEMM_TF32TCEC;
	ptr[1] = (3u << 5) | CUMPSGEMM_FP16TCEC;
}
} // unnamed namespace

void init_dynamic_launch_flag_buffer(
		cuMpSGEMM_handle* handle
		) {
	handle->dynamic_launch_handle = new cumpsgemm::dynamic_launch::dynamic_launch_handle;

	handle->dynamic_launch_handle->flag_buffer_length = 10000;
	handle->dynamic_launch_handle->current_buffer_id = 1;
	handle->dynamic_launch_handle->enabled = false;
	handle->dynamic_launch_handle->enabled_id = 0;

	handle->dynamic_launch_handle->mode_A = CUMPSGEMM_TF32TCEC;
	handle->dynamic_launch_handle->mode_B = CUMPSGEMM_FP16TCEC;

	CUTF_CHECK_ERROR(hipMalloc(&handle->dynamic_launch_handle->flag_buffer, sizeof(int) * handle->dynamic_launch_handle->flag_buffer_length));
	init_flag_buffer<<<1, 1, 0, handle->hip_stream>>>(handle->dynamic_launch_handle->flag_buffer);
}

void destroy_launch_flag_buffer(
		cuMpSGEMM_handle* handle
		) {
	CUTF_CHECK_ERROR(hipFree(handle->dynamic_launch_handle->flag_buffer));
	delete handle->dynamic_launch_handle;
}

unsigned cumpsgemm::dynamic_launch::get_next_dynamic_launch_flag_buffer_id(
		cuMpSGEMM_handle* handle
		) {
	const auto next = ++(handle->dynamic_launch_handle->current_buffer_id);
	if (next < handle->dynamic_launch_handle->flag_buffer_length) {
		return next;
	}
	handle->dynamic_launch_handle->flag_buffer_length = 2;
	return 2;
}

unsigned cumpsgemm::dynamic_launch::get_current_dynamic_launch_flag_buffer_id(
		cuMpSGEMM_handle* handle
		) {
	return handle->dynamic_launch_handle->current_buffer_id;
}

int cumpsgemm::dynamic_launch::get_dynamic_launch_buffer(cuMpSGEMM_handle* handle, const unsigned buffer_id) {
	int mode;
	cutf::memory::copy(&mode, handle->dynamic_launch_handle->flag_buffer + buffer_id, 1);

	return mode;
}

void cumpsgemm::dynamic_launch::set_dynamic_launch_flag_buffer_id(
		cuMpSGEMM_handle* handle,
		const unsigned id
		) {
	handle->dynamic_launch_handle->enabled_id = id;
	handle->dynamic_launch_handle->enabled = 1;
}

void cumpsgemm::dynamic_launch::unset_dynamic_launch_flag_buffer_id(
		cuMpSGEMM_handle* handle
		) {
	handle->dynamic_launch_handle->enabled_id = 0;
	handle->dynamic_launch_handle->enabled = 0;
}
