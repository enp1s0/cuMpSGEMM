#include "hip/hip_runtime.h"
#include <string>
#include <hipblas.h>
#include <unistd.h>
#include <dlfcn.h>
#include <cumpsgemm/cumpsgemm.hpp>
#include <cumpsgemm/hijack_control.hpp>
#include <cugemm_Mx2x2.hpp>
#include "handle.hpp"
#include "exp_stats.hpp"
#include "dynamic_launch.hpp"

namespace {

cuMpSGEMM_handle_t internal_global_cuMpSGEMM_handle = nullptr;
std::string internal_global_last_called_function_str = "";
bool global_internal_gemm_Mx2x2_enabled = false;

enum hijack_control_t {
	static_mode,
	dynamic_mode
} hijack_mode = dynamic_mode;
cuMpSGEMM_compute_mode_t internal_global_compute_mode = CUMPSGEMM_CUBLAS;

const std::string info_env_name = "CUMPSGEMM_INFO";
void cuMpSGEMM_log(
		const std::string str
		) {
	const auto env = getenv(info_env_name.c_str());
	if (env != nullptr && std::string(env) != "0") {
		std::fprintf(stdout, "[cuMpSGEMM LOG] %s\n",
				str.c_str());
		std::fflush(stdout);
	}
}

const std::string error_env_name = "CUMPSGEMM_ERROR_LOG";
void cuMpSGEMM_error(
		const std::string str
		) {
	const auto env = getenv(error_env_name.c_str());
	if (env != nullptr && std::string(env) != "0") {
		std::fprintf(stdout, "[cuMpSGEMM ERROR] %s\n",
				str.c_str());
		std::fflush(stdout);
	}
}

void cuMpSGEMM_warning(
		const std::string str
		) {
	const auto env = getenv(error_env_name.c_str());
	if (env != nullptr && std::string(env) != "0") {
		std::fprintf(stdout, "[cuMpSGEMM WARNING] %s\n",
				str.c_str());
		std::fflush(stdout);
	}
}

void* cuMpSGEMM_get_function_pointer(const std::string library_name, const std::string function_name) {

	// Open the library
	const auto lib_ptr = dlopen(library_name.c_str(), RTLD_NOW);
	if (lib_ptr == nullptr) {
		cuMpSGEMM_warning("Failed to load " + library_name + ". Default rule will be used.");
		return nullptr;
	}

	// Get function pointer
	void* function_ptr = dlsym(lib_ptr, function_name.c_str());
	if (function_ptr == NULL) {
		cuMpSGEMM_warning("Failed to load a function " + function_name + " during selecting hijacking function. Default rule will be used.");
		return nullptr;
	}

	return function_ptr;
}

std::string get_cublas_op_str(const hipblasOperation_t op) {
	switch (op) {
	case HIPBLAS_OP_C:
		return "C";
	case HIPBLAS_OP_N:
		return "N";
	case HIPBLAS_OP_T:
		return "T";
	default:
		return "?";
	}
}

cuMpSGEMM_handle_t cuMpSGEMM_get_internal_global_handle() {
	if (internal_global_cuMpSGEMM_handle == nullptr) {
		cuMpSGEMM_create(&internal_global_cuMpSGEMM_handle);
	}
	return internal_global_cuMpSGEMM_handle;
}

const std::string rule_lib_name = "libcumpsgemm_rule.so";
const std::string cublas_lib_name = "libcublas.so";
} // noname namespace

extern "C" const char* cuMpSGEMM_get_compute_mode_string (
		const cuMpSGEMM_compute_mode_t mode
		) {
	switch (mode) {
	case CUMPSGEMM_CUBLAS:
		return "CUBLAS";
	case CUMPSGEMM_FP16TC:
		return "FP16TC";
	case CUMPSGEMM_FP16TCEC:
		return "FP16TCEC";
	case CUMPSGEMM_TF32TC:
		return "TF32TC";
	case CUMPSGEMM_TF32TCEC:
		return "TF32TCEC";
	case CUMPSGEMM_CUBLAS_SIMT:
		return "CUBLAS_SIMT";
	case CUMPSGEMM_CUBLAS_FP16TC:
		return "CUBLAS_FP16TC";
	case CUMPSGEMM_CUBLAS_TF32TC:
		return "CUBLAS_TF32TC";
	case CUMPSGEMM_DRY_RUN:
		return "DRY_RUN";
	case CUMPSGEMM_AUTO:
		return "AUTO";
	}
	return "Unknown";
}

extern "C" cuMpSGEMM_compute_mode_t cuMpSGEMM_get_compute_mode_internal (
		const char* const func_name,
		hipblasHandle_t const cublas_handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const unsigned m, const unsigned n, const unsigned k
		) {
	if (hijack_mode == dynamic_mode) {
		cuMpSGEMM_compute_mode_t (*func)(
				const char* const func_name,
				hipblasHandle_t const cublas_handle,
				const hipblasOperation_t op_A,
				const hipblasOperation_t op_B,
				const unsigned m, const unsigned n, const unsigned k
				);
		*(void**)(&func) = cuMpSGEMM_get_function_pointer(rule_lib_name, __func__);

		if (func == nullptr) {
			return cuMpSGEMM_get_compute_mode(func_name, cublas_handle, op_A, op_B, m, n, k);
		}

		return func(func_name, cublas_handle, op_A, op_B, m, n, k);
	}
	return internal_global_compute_mode;
}

template <class T>
hipblasStatus_t cuMpSGEMM_hijack_core(
		const char* const func_name,
		hipblasHandle_t const cublas_handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const T* alpha,
		const T* const a_dmem_ptr, const uint64_t lda,
		const T* const b_dmem_ptr, const uint64_t ldb,
		const T* beta,
		T* const c_dmem_ptr, const uint64_t ldc
		) {
	if (std::is_same<T, float>::value && (op_A == HIPBLAS_OP_C || op_B == HIPBLAS_OP_C)) {
		return HIPBLAS_STATUS_INVALID_VALUE;
	}

	hipStream_t hip_stream;
	hipblasGetStream(cublas_handle, &hip_stream);

	cuMpSGEMM_compute_mode_t compute_mode =
		cuMpSGEMM_get_compute_mode_internal(
				func_name,
				cublas_handle,
				op_A,
				op_B,
				m, n, k
				);

	cuMpSGEMM_log(std::string(func_name) + " op=(" + get_cublas_op_str(op_A) + ", " + get_cublas_op_str(op_B) +
			"), shape=(" + std::to_string(m) + ", " + std::to_string(n) + ", " + std::to_string(k) + "), mode=" + cuMpSGEMM_get_compute_mode_string(compute_mode) +
			"[" + (hijack_mode == dynamic_mode ? "dynamic" : "static") + "][exp_stats:" + (cumpsgemm::hijack_control::get_internal_global_handle()->exp_stats_handle->enabled ? "1" : "0") + "]");
	cumpsgemm::hijack_control::set_last_called_function_str(
			std::string(func_name) + "," +
			get_cublas_op_str(op_A) + "," +
			get_cublas_op_str(op_B) + "," +
			std::to_string(m) + "," +
			std::to_string(n) + "," +
			std::to_string(k) + "," +
			"1," + // batch_size
			cuMpSGEMM_get_compute_mode_string(compute_mode)
			);

	if (compute_mode == CUMPSGEMM_DRY_RUN) {
		return HIPBLAS_STATUS_SUCCESS;
	}

	hipblasStatus_t res;

	if (compute_mode == CUMPSGEMM_CUBLAS &&
			((m & (m - 1)) == 0) && n == 2 && k == 2 &&
			global_internal_gemm_Mx2x2_enabled) {

		mtk::cugemm::gemm_Mx2x2(
				op_A, op_B,
				m,
				*alpha,
				a_dmem_ptr, lda,
				b_dmem_ptr, ldb,
				*beta,
				c_dmem_ptr, ldc
				);

		return HIPBLAS_STATUS_SUCCESS;
	}

	if (compute_mode == CUMPSGEMM_CUBLAS || compute_mode == CUMPSGEMM_CUBLAS_FP16TC || compute_mode == CUMPSGEMM_CUBLAS_TF32TC || compute_mode == CUMPSGEMM_CUBLAS_SIMT) {
		hipblasMath_t math_mode;
		hipblasGetMathMode(cublas_handle, &math_mode);
		if (compute_mode == CUMPSGEMM_CUBLAS_TF32TC) {
			hipblasSetMathMode(cublas_handle, HIPBLAS_TF32_TENSOR_OP_MATH);
		} else if (compute_mode == CUMPSGEMM_CUBLAS_FP16TC) {
			hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);
		} else if (compute_mode == CUMPSGEMM_CUBLAS_SIMT) {
			hipblasSetMathMode(cublas_handle, HIPBLAS_DEFAULT_MATH);
		}

		hipblasStatus_t (*func_ptr)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const T*, const T*, int, const T*, int, const T*, T*, int);
		*(void**)(&func_ptr) = cuMpSGEMM_get_function_pointer(
				cublas_lib_name.c_str(),
				func_name
				);
		if (func_ptr == nullptr) {
			cuMpSGEMM_error(std::string("Could not load cuBLAS function \"") + func_name + "\"");
		}
		res = (*func_ptr)(cublas_handle, op_A, op_B, m, n, k, alpha, a_dmem_ptr, lda, b_dmem_ptr, ldb, beta, c_dmem_ptr, ldc);

		// restore math mode
		hipblasSetMathMode(cublas_handle, math_mode);

		if (cumpsgemm::hijack_control::get_internal_global_handle()->exp_stats_handle->enabled) {
			cumpsgemm::exp_stats::exp_stats_ext(
					cumpsgemm::hijack_control::get_internal_global_handle(),
					m, n,
					c_dmem_ptr, ldc,
					1,
					0
					);
		}

	} else {
		res = cumpsgemm::gemm<T>(
				cuMpSGEMM_get_internal_global_handle(),
				op_A, op_B,
				m, n, k,
				alpha,
				a_dmem_ptr, lda,
				b_dmem_ptr, ldb,
				beta,
				c_dmem_ptr, ldc,
				compute_mode
				);
	}

	return res;
}

template <class T>
hipblasStatus_t cuMpSGEMM_stridedBatched_hijack_core(
		const char* const func_name,
		hipblasHandle_t const cublas_handle,
		const hipblasOperation_t op_A,
		const hipblasOperation_t op_B,
		const uint64_t m,
		const uint64_t n,
		const uint64_t k,
		const T* alpha,
		const T* const a_dmem_ptr, const uint64_t lda, const uint64_t stridea,
		const T* const b_dmem_ptr, const uint64_t ldb, const uint64_t strideb,
		const T* beta,
		T* const c_dmem_ptr, const uint64_t ldc, const uint64_t stridec,
		const uint64_t batch_count
		) {
	if (std::is_same<T, float>::value && (op_A == HIPBLAS_OP_C || op_B == HIPBLAS_OP_C)) {
		return HIPBLAS_STATUS_INVALID_VALUE;
	}

	hipStream_t hip_stream;
	hipblasGetStream(cublas_handle, &hip_stream);

	cuMpSGEMM_compute_mode_t compute_mode =
		cuMpSGEMM_get_compute_mode_internal(
				func_name,
				cublas_handle,
				op_A,
				op_B,
				m, n, k
				);

	cuMpSGEMM_log(std::string(func_name) + " op=(" + get_cublas_op_str(op_A) + ", " + get_cublas_op_str(op_B) +
			"), shape=(" + std::to_string(m) + ", " + std::to_string(n) + ", " + std::to_string(k) + "), batch=" + std::to_string(batch_count) + ", mode=" + cuMpSGEMM_get_compute_mode_string(compute_mode) +
			"[" + (hijack_mode == dynamic_mode ? "dynamic" : "static") + "][exp_stats:" + (cumpsgemm::hijack_control::get_internal_global_handle()->exp_stats_handle->enabled ? "1" : "0") + "]");

	cumpsgemm::hijack_control::set_last_called_function_str(
			std::string(func_name) + "," +
			get_cublas_op_str(op_A) + "," +
			get_cublas_op_str(op_B) + "," +
			std::to_string(m) + "," +
			std::to_string(n) + "," +
			std::to_string(k) + "," +
			std::to_string(batch_count) + "," +
			cuMpSGEMM_get_compute_mode_string(compute_mode)
			);

	if (compute_mode == CUMPSGEMM_DRY_RUN) {
		return HIPBLAS_STATUS_SUCCESS;
	}

	hipblasStatus_t res;

	if (compute_mode == CUMPSGEMM_CUBLAS &&
			((m & (m - 1)) == 0) && n == 2 && k == 2 &&
			global_internal_gemm_Mx2x2_enabled) {

		mtk::cugemm::gemm_strided_batch_Mx2x2(
				op_A, op_B,
				m,
				*alpha,
				a_dmem_ptr, lda, stridea,
				b_dmem_ptr, ldb, strideb,
				*beta,
				c_dmem_ptr, ldc, stridec,
				batch_count
				);

		return HIPBLAS_STATUS_SUCCESS;
	}

	if (compute_mode == CUMPSGEMM_CUBLAS || compute_mode == CUMPSGEMM_CUBLAS_FP16TC || compute_mode == CUMPSGEMM_CUBLAS_TF32TC || compute_mode == CUMPSGEMM_CUBLAS_SIMT) {
		hipblasMath_t math_mode;
		hipblasGetMathMode(cublas_handle, &math_mode);
		if (compute_mode == CUMPSGEMM_CUBLAS_TF32TC) {
			hipblasSetMathMode(cublas_handle, HIPBLAS_TF32_TENSOR_OP_MATH);
		} else if (compute_mode == CUMPSGEMM_CUBLAS_FP16TC) {
			hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);
		} else if (compute_mode == CUMPSGEMM_CUBLAS_SIMT) {
			hipblasSetMathMode(cublas_handle, HIPBLAS_DEFAULT_MATH);
		}

		hipblasStatus_t (*func_ptr)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const T*, const T*, int, long long int, const T*, int, long long int, const T*, T*, int, long long int, int);
		*(void**)(&func_ptr) = cuMpSGEMM_get_function_pointer(
				cublas_lib_name.c_str(),
				func_name
				);
		if (func_ptr == nullptr) {
			cuMpSGEMM_error(std::string("Could not load cuBLAS function \"") + func_name + "\"");
		}
		res = (*func_ptr)(cublas_handle, op_A, op_B, m, n, k, alpha, a_dmem_ptr, lda, stridea, b_dmem_ptr, ldb, strideb, beta, c_dmem_ptr, ldc, stridec, batch_count);
		hipblasSetMathMode(cublas_handle, math_mode);

		if (cumpsgemm::hijack_control::get_internal_global_handle()->exp_stats_handle->enabled) {
			cumpsgemm::exp_stats::exp_stats_ext(
					cumpsgemm::hijack_control::get_internal_global_handle(),
					m, n,
					c_dmem_ptr, ldc,
					batch_count,
					stridec
					);
		}
	} else {

		res = cumpsgemm::gemm_stridedBatch<T>(
				cuMpSGEMM_get_internal_global_handle(),
				op_A, op_B,
				m, n, k,
				alpha,
				a_dmem_ptr, lda, stridea,
				b_dmem_ptr, ldb, strideb,
				beta,
				c_dmem_ptr, ldc, stridec,
				batch_count,
				compute_mode
				);
	}
	return res;
}

// cuBLAS functions
extern "C" {
hipblasStatus_t hipblasSgemm (
		hipblasHandle_t cublas_handle,
		hipblasOperation_t op_A,
		hipblasOperation_t op_B,
		int m,
		int n,
		int k,
		const float* alpha,
		const float* a_dmem_ptr, int lda,
		const float* b_dmem_ptr, int ldb,
		const float* beta,
		float* c_dmem_ptr, int ldc
		) {
	hipStream_t hip_stream;
	hipblasGetStream(cublas_handle, &hip_stream);

	return cuMpSGEMM_hijack_core<float>(
			__func__,
			cublas_handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda,
			b_dmem_ptr, ldb,
			beta,
			c_dmem_ptr, ldc
			);
}

hipblasStatus_t hipblasCgemm (
		hipblasHandle_t cublas_handle,
		hipblasOperation_t op_A,
		hipblasOperation_t op_B,
		int m,
		int n,
		int k,
		const hipComplex* alpha,
		const hipComplex* a_dmem_ptr, int lda,
		const hipComplex* b_dmem_ptr, int ldb,
		const hipComplex* beta,
		hipComplex* c_dmem_ptr, int ldc
		) {
	return cuMpSGEMM_hijack_core<hipComplex>(
			__func__,
			cublas_handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda,
			b_dmem_ptr, ldb,
			beta,
			c_dmem_ptr, ldc
			);
}

hipblasStatus_t hipblasSgemmStridedBatched (
		hipblasHandle_t cublas_handle,
		hipblasOperation_t op_A,
		hipblasOperation_t op_B,
		int m,
		int n,
		int k,
		const float* alpha,
		const float* a_dmem_ptr, int lda, long long int stridea,
		const float* b_dmem_ptr, int ldb, long long int strideb,
		const float* beta,
		float* c_dmem_ptr, int ldc, long long int stridec,
		const int batch_count
		) {
	return cuMpSGEMM_stridedBatched_hijack_core<float>(
			__func__,
			cublas_handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda, stridea,
			b_dmem_ptr, ldb, strideb,
			beta,
			c_dmem_ptr, ldc, stridec,
			batch_count
			);
}

hipblasStatus_t hipblasCgemmStridedBatched (
		hipblasHandle_t cublas_handle,
		hipblasOperation_t op_A,
		hipblasOperation_t op_B,
		int m,
		int n,
		int k,
		const hipComplex* alpha,
		const hipComplex* a_dmem_ptr, int lda, const long long int stridea,
		const hipComplex* b_dmem_ptr, int ldb, const long long int strideb,
		const hipComplex* beta,
		hipComplex* c_dmem_ptr, int ldc, const long long int stridec,
		const int batch_count
		) {
	hipStream_t hip_stream;
	hipblasGetStream(cublas_handle, &hip_stream);

	return cuMpSGEMM_stridedBatched_hijack_core<hipComplex>(
			__func__,
			cublas_handle,
			op_A, op_B,
			m, n, k,
			alpha,
			a_dmem_ptr, lda, stridea,
			b_dmem_ptr, ldb, strideb,
			beta,
			c_dmem_ptr, ldc, stridec,
			batch_count
			);
}

hipblasStatus_t hipblasGemmEx(hipblasHandle_t handle, hipblasOperation_t transa,
                            hipblasOperation_t transb, int m, int n, int k,
                            const void *alpha, const void *A,
                            hipDataType Atype, int lda, const void *B,
                            hipDataType Btype, int ldb, const void *beta,
														void *C, hipDataType Ctype, int ldc,
														hipblasComputeType_t computeType,
														hipblasGemmAlgo_t algo) {
	if (Atype == HIP_R_32F && Btype == HIP_R_32F && Ctype == HIP_R_32F) {
		return hipblasSgemm(
				handle,
				transa, transb,
				m, n, k,
				reinterpret_cast<const float*>(alpha),
				reinterpret_cast<const float*>(A), lda,
				reinterpret_cast<const float*>(B), ldb,
				reinterpret_cast<const float*>(beta),
				reinterpret_cast<float*>(C), ldc
				);
	}
	if (Atype == HIP_C_32F && Btype == HIP_C_32F && Ctype == HIP_C_32F) {
		return hipblasCgemm(
				handle,
				transa, transb,
				m, n, k,
				reinterpret_cast<const hipComplex*>(alpha),
				reinterpret_cast<const hipComplex*>(A), lda,
				reinterpret_cast<const hipComplex*>(B), ldb,
				reinterpret_cast<const hipComplex*>(beta),
				reinterpret_cast<hipComplex*>(C), ldc
				);
	}
	hipblasStatus_t (*func_ptr)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void*, hipDataType, int, const void*, hipDataType, int, const void*, void*, hipDataType, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&func_ptr) = cuMpSGEMM_get_function_pointer(
			cublas_lib_name.c_str(),
			__func__
			);
	const auto res = (*func_ptr)(handle, transa, transb, m, n, k, alpha, A, Atype, lda, B, Btype, ldb, beta, C, Ctype, ldc, computeType, algo);
	return res;
}

hipblasStatus_t hipblasGemmStridedBatchedEx(hipblasHandle_t handle, hipblasOperation_t transa,
                            hipblasOperation_t transb, int m, int n, int k,
                            const void *alpha, const void *A,
                            hipDataType Atype, int lda, long long int strideA, const void *B,
                            hipDataType Btype, int ldb, long long int strideB, const void *beta,
														void *C, hipDataType Ctype, int ldc, long long int strideC,
														int batch_count,
														hipblasComputeType_t computeType,
														hipblasGemmAlgo_t algo) {
	if (Atype == HIP_R_32F && Btype == HIP_R_32F && Ctype == HIP_R_32F) {
		return hipblasSgemmStridedBatched(
				handle,
				transa, transb,
				m, n, k,
				reinterpret_cast<const float*>(alpha),
				reinterpret_cast<const float*>(A), lda, strideA,
				reinterpret_cast<const float*>(B), ldb, strideB,
				reinterpret_cast<const float*>(beta),
				reinterpret_cast<float*>(C), ldc, strideC,
				batch_count
				);
	}
	if (Atype == HIP_C_32F && Btype == HIP_C_32F && Ctype == HIP_C_32F) {
		return hipblasCgemmStridedBatched(
				handle,
				transa, transb,
				m, n, k,
				reinterpret_cast<const hipComplex*>(alpha),
				reinterpret_cast<const hipComplex*>(A), lda, strideA,
				reinterpret_cast<const hipComplex*>(B), ldb, strideB,
				reinterpret_cast<const hipComplex*>(beta),
				reinterpret_cast<hipComplex*>(C), ldc, strideC,
				batch_count
				);
	}
	hipblasStatus_t (*func_ptr)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void*, hipDataType, int, long long int, const void*, hipDataType, int, long long int, const void*, void*, hipDataType, int, long long int, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&func_ptr) = cuMpSGEMM_get_function_pointer(
			cublas_lib_name.c_str(),
			__func__
			);
	const auto res = (*func_ptr)(handle, transa, transb, m, n, k, alpha, A, Atype, lda, strideA, B, Btype, ldb, strideB, beta, C, Ctype, ldc, strideC, batch_count, computeType, algo);
	return res;
}
} // extern "C"

cuMpSGEMM_handle* cumpsgemm::hijack_control::get_internal_global_handle() {
	if (internal_global_cuMpSGEMM_handle == nullptr) {
		cuMpSGEMM_create(&internal_global_cuMpSGEMM_handle);
	}
	return internal_global_cuMpSGEMM_handle;
}

void cumpsgemm::hijack_control::set_compute_mode(const cuMpSGEMM_compute_mode_t mode) {
	internal_global_compute_mode = mode;
	hijack_mode = static_mode;
}

void cumpsgemm::hijack_control::unset_compute_mode() {
	hijack_mode = dynamic_mode;
}

std::pair<std::size_t, std::size_t> cumpsgemm::hijack_control::get_exp_stats(const unsigned buffer_id) {
	return cumpsgemm::exp_stats::get_exp_stats(get_internal_global_handle(), buffer_id);
}

unsigned cumpsgemm::hijack_control::get_current_exp_stats_buffer_id() {
	return cumpsgemm::exp_stats::get_current_exp_stats_buffer_id(cuMpSGEMM_get_internal_global_handle());
}

void cumpsgemm::hijack_control::enable_exp_stats() {
	cumpsgemm::enable_exp_stats(get_internal_global_handle());
}

void cumpsgemm::hijack_control::disable_exp_stats() {
	cumpsgemm::disable_exp_stats(get_internal_global_handle());
	cumpsgemm::hijack_control::unset_compute_mode();
}

void cumpsgemm::hijack_control::set_exp_stats_params(
		const float ignore_threshold,
		const float lost_threshold
		) {
	cumpsgemm::set_exp_stats_params(get_internal_global_handle(), ignore_threshold, lost_threshold);
}

bool cumpsgemm::hijack_control::is_exp_stats_enabled() {
	return get_internal_global_handle()->exp_stats_handle->enabled;
}

void cumpsgemm::hijack_control::reset_exp_stats_buffer_id() {
	cumpsgemm::exp_stats::reset_exp_stats_buffer_id(get_internal_global_handle());
}

void cumpsgemm::hijack_control::exp_stats(
		const unsigned m,
		const unsigned n,
		const float* const ptr,
		const unsigned ld,
		const unsigned batch_size,
		const unsigned stride
		) {
	cumpsgemm::exp_stats::exp_stats_ext(
			get_internal_global_handle(),
			m, n,
			ptr, ld,
			batch_size, stride
			);
}

void cumpsgemm::hijack_control::download_exp_stats_result(const unsigned id) {
	cumpsgemm::exp_stats::download_exp_stats(get_internal_global_handle(), id);
}

std::string cumpsgemm::hijack_control::get_last_called_function_str() {
	return internal_global_last_called_function_str;
}

void cumpsgemm::hijack_control::set_last_called_function_str(
		const std::string func_str
		) {
	internal_global_last_called_function_str = func_str;
}

void cumpsgemm::hijack_control::clear_last_called_function_str() {
	cumpsgemm::hijack_control::set_last_called_function_str("");
}

unsigned cumpsgemm::hijack_control::get_next_dynamic_launch_flag_buffer_id() {
	return cumpsgemm::dynamic_launch::get_next_dynamic_launch_flag_buffer_id(get_internal_global_handle());
}

void cumpsgemm::hijack_control::set_dynamic_launch_flag_buffer_id_use(unsigned id) {
	cumpsgemm::dynamic_launch::set_dynamic_launch_flag_buffer_id(get_internal_global_handle(), id);
}

namespace {
__global__ void dynamic_launch_flag_buffer_id_by_exp_stats_kernel(
		int* const flag_buffer_ptr,
		const cumpsgemm::counter_t* const total_counter_A_ptr,
		const cumpsgemm::counter_t* const lost_counter_A_ptr,
		const cumpsgemm::counter_t* const total_counter_B_ptr,
		const cumpsgemm::counter_t* const lost_counter_B_ptr,
		const float rate_threshold
		) {
	const auto pA = (static_cast<float>(*lost_counter_A_ptr) / *total_counter_A_ptr) < rate_threshold;
	const auto pB = (static_cast<float>(*lost_counter_B_ptr) / *total_counter_B_ptr) < rate_threshold;
	if (pA && pB) {
		*flag_buffer_ptr = CUMPSGEMM_FP16TCEC;
	} else {
		*flag_buffer_ptr = CUMPSGEMM_TF32TCEC;
	}
}
} // unnamed namespace

void cumpsgemm::hijack_control::set_dynamic_launch_flag_buffer_by_exp_stats(
		const unsigned exp_stats_buffer_id_A,
		const unsigned exp_stats_buffer_id_B,
		const unsigned dynamic_launch_flag_buffer_id,
		const float ratio_threshold
		) {
	const auto handle = get_internal_global_handle();
	const auto hip_stream = handle->hip_stream;

	dynamic_launch_flag_buffer_id_by_exp_stats_kernel<<<1, 1, 0, hip_stream>>>(
			handle->dynamic_launch_handle->frag_buffer + dynamic_launch_flag_buffer_id,
			handle->exp_stats_handle->dev_total_counter_buffer + exp_stats_buffer_id_A,
			handle->exp_stats_handle->dev_lost_counter_buffer  + exp_stats_buffer_id_A,
			handle->exp_stats_handle->dev_total_counter_buffer + exp_stats_buffer_id_B,
			handle->exp_stats_handle->dev_lost_counter_buffer  + exp_stats_buffer_id_B,
			ratio_threshold
			);
}

void cumpsgemm::hijack_control::enable_custom_gemm_Mx2x2() {
	global_internal_gemm_Mx2x2_enabled  = true;
}

void cumpsgemm::hijack_control::disable_custom_gemm_Mx2x2() {
	global_internal_gemm_Mx2x2_enabled  = true;
}
