#include <cstddef>
#include <cutf/device.hpp>
#include <cumpsgemm/cumpsgemm.h>
#include "handle.hpp"

#define ENABLE_A100_OPTIMAZED_PARAMETERS

extern "C" {
hipblasStatus_t cuMpSGEMM_create(cuMpSGEMM_handle_t* const handle) {
	if ((*handle = new cuMpSGEMM_handle) == nullptr) {
		return HIPBLAS_STATUS_INTERNAL_ERROR;
	}

	int num_sms;
	CUTF_CHECK_ERROR(hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0));
	(*handle)->num_sms = num_sms;

	int cc_major, cc_minor;
	CUTF_CHECK_ERROR(hipDeviceGetAttribute(&cc_major, hipDeviceAttributeComputeCapabilityMajor, 0));
	CUTF_CHECK_ERROR(hipDeviceGetAttribute(&cc_minor, hipDeviceAttributeComputeCapabilityMinor, 0));

	if (cc_major == 8 && cc_minor == 0) {
		cumpsgemm::configure_instance_sm80((*handle)->gemm_module, (*handle)->gemm_stridedBatch_module);
	} else {
		cumpsgemm::configure_instance_sm86((*handle)->gemm_module, (*handle)->gemm_stridedBatch_module);
	}
	//cumpsgemm::configure_instance_simt((*handle)->gemm_module, (*handle)->gemm_stridedBatch_module);

	init_exp_stats_counter_buffer((*handle));
	init_dynamic_launch_flag_buffer((*handle));

	return HIPBLAS_STATUS_SUCCESS;
}

hipblasStatus_t cuMpSGEMM_destroy(cuMpSGEMM_handle_t handle) {
	destroy_exp_stats_counter_buffer(handle);
	destroy_launch_flag_buffer(handle);

	delete handle;
	return HIPBLAS_STATUS_SUCCESS;
}

hipblasStatus_t cuMpSGEMM_set_stream(cuMpSGEMM_handle_t handle, const hipStream_t hip_stream) {
	handle->hip_stream = hip_stream;
	return HIPBLAS_STATUS_SUCCESS;
}
}
